#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <hip/hip_runtime.h>

extern "C" {
#include "beam_common.h"
#include "form_beam.h"
#include "mycomplex.h"
}

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    /* Wrapper function for GPU/CUDA error handling. Every CUDA call goes through
       this function. It will return a message giving your the error string,
       file name and line of the error. Aborts on error. */

    if (code != 0)
    {
        fprintf(stderr, "GPUAssert:: %s - %s (%d)\n", hipGetErrorString(code), file, line);
        if (abort)
        {
            exit(code);
        }
    }
}

// define a macro for accessing gpuAssert
#define gpuErrchk(ans) {gpuAssert((ans), __FILE__, __LINE__, true);}


// define constants to be used in the kernel
#define NSTATION  128
#define NPOL      2
#define NSTOKES   4


__device__ void CatomicAdd( ComplexDouble *a, const ComplexDouble &b )
{
    double *x = (double *)a;
    double *y = x+1;
    atomicAdd(x, CReald(b));
    atomicAdd(y, CImagd(b));
}


__global__ void beamform_kernel( uint8_t *data,
                                 ComplexDouble *W,
                                 ComplexDouble *J,
                                 double invw,
                                 ComplexDouble *Bd,
                                 float *C,
                                 float *I )
/* Layout for input arrays:
 *   data [nsamples] [nchan] [NPFB] [NREC] [NINC] -- see docs
 *   W    [NSTATION] [nchan] [NPOL]               -- weights array
 *   J    [NSTATION] [nchan] [NPOL] [NPOL]        -- jones matrix
 * Layout for output arrays:
 *   Bd   [nsamples] [nchan]   [NPOL]             -- detected beam
 *   C    [nsamples] [NSTOKES] [nchan]            -- coherent full stokes
 *   I    [nsamples] [nchan]                      -- incoherent
 */
{
    // Translate GPU block/thread numbers into meaningful names
    int s   = blockIdx.x;  /* The (s)ample number */
    int c   = blockIdx.y;  /* The (c)hannel number */
    int nc  = gridDim.y;   /* The (n)umber of (c)hannels (=128) */
    int ant = threadIdx.x; /* The (ant)enna number */

    // Calculate the beam and the noise floor
    __shared__ double Ia[NSTATION];
    __shared__ ComplexDouble Bx[NSTATION], By[NSTATION];
    ComplexDouble Dx, Dy;
    ComplexDouble WDx, WDy;

    __shared__ ComplexDouble Nxx[NSTATION], Nxy[NSTATION],
                             Nyx[NSTATION], Nyy[NSTATION];


    /* Fix from Maceij regarding NaNs in output when running on Athena, 13 April 2018.
       Apparently the different compilers and architectures are treating what were 
       unintialised variables very differently */
    Bx[ant]  = CMaked( 0.0, 0.0 );
    By[ant]  = CMaked( 0.0, 0.0 );

    Dx  = CMaked( 0.0, 0.0 );
    Dy  = CMaked( 0.0, 0.0 );

    WDx = CMaked( 0.0, 0.0 );
    WDy = CMaked( 0.0, 0.0 );

    Nxx[ant] = CMaked( 0.0, 0.0 );
    Nxy[ant] = CMaked( 0.0, 0.0 );
    Nyx[ant] = CMaked( 0.0, 0.0 );
    Nyy[ant] = CMaked( 0.0, 0.0 );

    Ia[ant] = 0.0;

    // Calculate beamform products for each antenna, and then add them together
    // Calculate the coherent beam (B = J*W*D)
    Dx  = UCMPLX4_TO_CMPLX_FLT(data[D_IDX(s,c,ant,0,nc)]);
    Dy  = UCMPLX4_TO_CMPLX_FLT(data[D_IDX(s,c,ant,1,nc)]);

    Ia[ant] = DETECT(Dx) + DETECT(Dy);

    WDx = CMuld( W[W_IDX(c,ant,0,nc)], Dx );
    WDy = CMuld( W[W_IDX(c,ant,1,nc)], Dy );

    Bx[ant] = CAddd( CMuld( J[J_IDX(c,ant,0,0,nc)], WDx ),
                     CMuld( J[J_IDX(c,ant,1,0,nc)], WDy ) );
    By[ant] = CAddd( CMuld( J[J_IDX(c,ant,0,1,nc)], WDx ),
                     CMuld( J[J_IDX(c,ant,1,1,nc)], WDy ) );

    Nxx[ant] = CMuld( Bx[ant], CConjd(Bx[ant]) );
    Nxy[ant] = CMuld( Bx[ant], CConjd(By[ant]) );
    Nyx[ant] = CMuld( By[ant], CConjd(Bx[ant]) );
    Nyy[ant] = CMuld( By[ant], CConjd(By[ant]) );

    // Detect the coherent beam
    __syncthreads();
    if (ant < 64)
    {
        Ia[ant] += Ia[ant+64];
        Bx[ant] = CAddd( Bx[ant], Bx[ant+64] );
        By[ant] = CAddd( By[ant], By[ant+64] );
        Nxx[ant] = CAddd( Nxx[ant], Nxx[ant+64] );
        Nxy[ant] = CAddd( Nxy[ant], Nxy[ant+64] );
        Nyx[ant] = CAddd( Nyx[ant], Nyx[ant+64] );
        Nyy[ant] = CAddd( Nyy[ant], Nyy[ant+64] );
    }
    __syncthreads();
    if (ant < 32)
    {
        Ia[ant] += Ia[ant+32];
        Bx[ant] = CAddd( Bx[ant], Bx[ant+32] );
        By[ant] = CAddd( By[ant], By[ant+32] );
        Nxx[ant] = CAddd( Nxx[ant], Nxx[ant+32] );
        Nxy[ant] = CAddd( Nxy[ant], Nxy[ant+32] );
        Nyx[ant] = CAddd( Nyx[ant], Nyx[ant+32] );
        Nyy[ant] = CAddd( Nyy[ant], Nyy[ant+32] );
    }
    if (ant < 16)
    {
        Ia[ant] += Ia[ant+16];
        Bx[ant] = CAddd( Bx[ant], Bx[ant+16] );
        By[ant] = CAddd( By[ant], By[ant+16] );
        Nxx[ant] = CAddd( Nxx[ant], Nxx[ant+16] );
        Nxy[ant] = CAddd( Nxy[ant], Nxy[ant+16] );
        Nyx[ant] = CAddd( Nyx[ant], Nyx[ant+16] );
        Nyy[ant] = CAddd( Nyy[ant], Nyy[ant+16] );
    }
    if (ant < 8)
    {
        Ia[ant] += Ia[ant+8];
        Bx[ant] = CAddd( Bx[ant], Bx[ant+8] );
        By[ant] = CAddd( By[ant], By[ant+8] );
        Nxx[ant] = CAddd( Nxx[ant], Nxx[ant+8] );
        Nxy[ant] = CAddd( Nxy[ant], Nxy[ant+8] );
        Nyx[ant] = CAddd( Nyx[ant], Nyx[ant+8] );
        Nyy[ant] = CAddd( Nyy[ant], Nyy[ant+8] );
    }
    if (ant < 4)
    {
        Ia[ant] += Ia[ant+4];
        Bx[ant] = CAddd( Bx[ant], Bx[ant+4] );
        By[ant] = CAddd( By[ant], By[ant+4] );
        Nxx[ant] = CAddd( Nxx[ant], Nxx[ant+4] );
        Nxy[ant] = CAddd( Nxy[ant], Nxy[ant+4] );
        Nyx[ant] = CAddd( Nyx[ant], Nyx[ant+4] );
        Nyy[ant] = CAddd( Nyy[ant], Nyy[ant+4] );
    }
    if (ant < 2)
    {
        Ia[ant] += Ia[ant+2];
        Bx[ant] = CAddd( Bx[ant], Bx[ant+2] );
        By[ant] = CAddd( By[ant], By[ant+2] );
        Nxx[ant] = CAddd( Nxx[ant], Nxx[ant+2] );
        Nxy[ant] = CAddd( Nxy[ant], Nxy[ant+2] );
        Nyx[ant] = CAddd( Nyx[ant], Nyx[ant+2] );
        Nyy[ant] = CAddd( Nyy[ant], Nyy[ant+2] );
    }
    if (ant < 1)
    {
        Ia[ant] += Ia[ant+1];
        Bx[ant] = CAddd( Bx[ant], Bx[ant+1] );
        By[ant] = CAddd( By[ant], By[ant+1] );
        Nxx[ant] = CAddd( Nxx[ant], Nxx[ant+1] );
        Nxy[ant] = CAddd( Nxy[ant], Nxy[ant+1] );
        Nyx[ant] = CAddd( Nyx[ant], Nyx[ant+1] );
        Nyy[ant] = CAddd( Nyy[ant], Nyy[ant+1] );
    }
    __syncthreads();

    // Form the stokes parameters for the coherent beam
    if (ant == 0)
    {
        float bnXX = DETECT(Bx[0]) - CReald(Nxx[0]);
        float bnYY = DETECT(By[0]) - CReald(Nyy[0]);
        ComplexDouble bnXY = CSubd(
                                 CMuld( Bx[0], CConjd( By[0] ) ),
                                 Nxy[0] );

        // The incoherent beam
        I[I_IDX(s,c,nc)] = Ia[0];

        // Stokes I, Q, U, V:
        C[C_IDX(s,c,0,nc)] = invw*(bnXX + bnYY);
        C[C_IDX(s,c,1,nc)] = invw*(bnXX - bnYY);
        C[C_IDX(s,c,2,nc)] =  2.0*invw*CReald( bnXY );
        C[C_IDX(s,c,3,nc)] = -2.0*invw*CImagd( bnXY );

        // The beamformed products
        Bd[B_IDX(s,c,0,nc)] = Bx[0];
        Bd[B_IDX(s,c,1,nc)] = By[0];
    }
}

void cu_form_beam( uint8_t *data, struct make_beam_opts *opts,
                   ComplexDouble ***complex_weights_array,
                   ComplexDouble ****invJi, int file_no, int nstation, int nchan,
                   int npol, int outpol_coh, int outpol_incoh, double invw,
                   ComplexDouble ***detected_beam, float *coh, float *incoh )
/* The CPU version of the beamforming operations, using OpenMP for
 * parallelisation.
 *
 * Inputs:
 *   data    = array of 4bit+4bit complex numbers. For data order, refer to the
 *             documentation.
 *   opts    = passed option parameters, containing meta information about the
 *             obs and the data
 *   W       = complex weights array. [nstation][nchan][npol]
 *   J       = inverse Jones matrix. [nstation][nchan][npol][npol]
 *   file_no = number of file we are processing, starting at 0.
 *   nstation     = 128
 *   nchan        = 128
 *   npol         = 2 (X,Y)
 *   outpol_coh   = 4 (I,Q,U,V)
 *   outpol_incoh = 1 (I)
 *   invw         = the reciprocal of the sum of the antenna weights
 *
 * Outputs:
 *   detected_beam = result of beamforming operation, summed over antennas
 *                   [2*nsamples][nchan][npol]
 *   coh           = result in Stokes parameters (minus noise floor)
 *                   [nsamples][nstokes][nchan]
 *   incoh         = result (just Stokes I)
 *                   [nsamples][nchan]
 *
 * Assumes "coh" and "incoh" contain only zeros.
 */
{
    // Calculate array sizes for host and device
    size_t coh_size   = opts->sample_rate * outpol_coh   * nchan * sizeof(float);
    size_t incoh_size = opts->sample_rate * outpol_incoh * nchan * sizeof(float);
    size_t data_size  = opts->sample_rate * nstation * nchan * npol * sizeof(uint8_t);
    size_t Bd_size    = opts->sample_rate * nchan * npol * sizeof(ComplexDouble);
    size_t W_size     = nstation * nchan * npol          * sizeof(ComplexDouble);
    size_t J_size     = nstation * nchan * npol * npol   * sizeof(ComplexDouble);

    // Arrays to be passed to the GPU kernel
    // (We don't need to allocate host memory for data, coh, or incoh -- we
    // assume this is allocated before these pointers were passed into this
    // function)
    ComplexDouble *W, *d_W;
    ComplexDouble *J, *d_J;
    ComplexDouble *Bd, *d_Bd;
    uint8_t *d_data;
    float   *d_coh;
    float   *d_incoh;

    // Allocate host memory
    W  = (ComplexDouble *)malloc( W_size );
    J  = (ComplexDouble *)malloc( J_size );
    Bd = (ComplexDouble *)malloc( Bd_size );


    // Allocate device memory
    gpuErrchk(hipMalloc( (void **)&d_W,     W_size ));
    gpuErrchk(hipMalloc( (void **)&d_J,     J_size ));
    gpuErrchk(hipMalloc( (void **)&d_Bd,    Bd_size ));
    gpuErrchk(hipMalloc( (void **)&d_data,  data_size ));
    gpuErrchk(hipMalloc( (void **)&d_coh,   coh_size ));
    gpuErrchk(hipMalloc( (void **)&d_incoh, incoh_size ));

    // Setup input values (= populate W and J)
    int s, ant, ch, pol, pol2;
    int Wi, Ji;
    for (ant = 0; ant < nstation; ant++)
    for (ch  = 0; ch  < nchan   ; ch++ )
    for (pol = 0; pol < npol    ; pol++)
    {
        Wi = ant * (npol*nchan) +
             ch  * (npol) +
             pol;
        W[Wi] = complex_weights_array[ant][ch][pol];

        for (pol2 = 0; pol2 < npol; pol2++)
        {
            Ji = Wi*npol + pol2;
            J[Ji] = invJi[ant][ch][pol][pol2];
        }
    }

    // Copy the data to the device
    gpuErrchk(hipMemcpy( d_data, data, data_size, hipMemcpyHostToDevice ));
    gpuErrchk(hipMemcpy( d_W,    W,    W_size,    hipMemcpyHostToDevice ));
    gpuErrchk(hipMemcpy( d_J,    J,    J_size,    hipMemcpyHostToDevice ));

    // Call the kernel
    dim3 sc(opts->sample_rate, nchan);
    beamform_kernel<<<sc, NSTATION>>>(
            d_data, d_W, d_J, invw, d_Bd, d_coh, d_incoh );
    hipDeviceSynchronize();

    // Copy the results back into host memory
    gpuErrchk(hipMemcpy( coh,   d_coh,   coh_size,   hipMemcpyDeviceToHost ));
    gpuErrchk(hipMemcpy( incoh, d_incoh, incoh_size, hipMemcpyDeviceToHost ));
    gpuErrchk(hipMemcpy( Bd,    d_Bd,    Bd_size,    hipMemcpyDeviceToHost ));

    // Copy the data back from Bd back into the detected_beam array
    // Make sure we put it back into the correct half of the array, depending
    // on whether this is an even or odd second.
    int offset, i;
    if (file_no % 2 == 0)
        offset = 0;
    else
        offset = opts->sample_rate;

    for (s   = 0; s   < opts->sample_rate; s++  )
    for (ch  = 0; ch  < nchan            ; ch++ )
    for (pol = 0; pol < npol             ; pol++)
    {
        i = s  * (npol*nchan) +
            ch * (npol)       +
            pol;

        detected_beam[s+offset][ch][pol] = Bd[i];
    }

    // Free memory on host and device
    free( W );
    free( J );
    free( Bd );
    hipFree( d_W );
    hipFree( d_J );
    hipFree( d_Bd );
    hipFree( d_data );
    hipFree( d_coh );
    hipFree( d_incoh );
}

