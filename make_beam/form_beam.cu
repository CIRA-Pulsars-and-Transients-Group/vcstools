#include "hip/hip_runtime.h"
/********************************************************
 *                                                      *
 * Licensed under the Academic Free License version 3.0 *
 *                                                      *
 ********************************************************/

#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>

extern "C" {
#include "beam_common.h"
#include "form_beam.h"
#include "mycomplex.h"
}

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    /* Wrapper function for GPU/CUDA error handling. Every CUDA call goes through
       this function. It will return a message giving your the error string,
       file name and line of the error. Aborts on error. */

    if (code != 0)
    {
        fprintf(stderr, "GPUAssert:: %s - %s (%d)\n", hipGetErrorString(code), file, line);
        if (abort)
        {
            exit(code);
        }
    }
}

// define a macro for accessing gpuAssert
#define gpuErrchk(ans) {gpuAssert((ans), __FILE__, __LINE__, true);}


// define constants to be used in the kernel
#define NSTATION  128
#define NPOL      2
#define NSTOKES   4
// maximum number of pointings (currently)
#define NPOINTING 4



__global__ void beamform_kernel( uint8_t *data,
                                 ComplexDouble *W,
                                 ComplexDouble *J,
                                 double invw,
                                 ComplexDouble *Bd,
                                 float *C,
                                 float *I )
/* Layout for input arrays:
 *   data [nsamples] [nchan] [NPFB] [NREC] [NINC] -- see docs
 *   W    [NSTATION] [nchan] [NPOL]               -- weights array
 *   J    [NSTATION] [nchan] [NPOL] [NPOL]        -- jones matrix
 * Layout for output arrays:
 *   Bd   [nsamples] [nchan]   [NPOL]             -- detected beam
 *   C    [nsamples] [NSTOKES] [nchan]            -- coherent full stokes
 *   I    [nsamples] [nchan]                      -- incoherent
 */
{
    // Translate GPU block/thread numbers into meaningful names
    int s   = blockIdx.x;  /* The (s)ample number */
    int ns  = gridDim.x;   /* The (n)umber of (s)amples (=10000)*/
    int c   = blockIdx.y;  /* The (c)hannel number */
    int p   = threadIdx.y; /* The (p)ointing number */
    int nc  = gridDim.y;   /* The (n)umber of (c)hannels (=128) */
    int ant = threadIdx.x; /* The (ant)enna number */

    // GPU profiling
    clock_t start, stop;
    double setup_t, detect_t, sum_t, stokes_t;
    if ((p == 0) && (ant == 0) && (c == 0) && (s == 0)) start = clock();
    
    // Calculate the beam and the noise floor
    __shared__ double Ia[NSTATION];
    __shared__ ComplexDouble Bx[NPOINTING*NSTATION], By[NPOINTING*NSTATION];
    ComplexDouble Dx, Dy;
    ComplexDouble WDx, WDy;

    __shared__ ComplexDouble Nxx[NPOINTING*NSTATION], Nxy[NPOINTING*NSTATION],
                             Nyy[NPOINTING*NSTATION];//Nyx[NPOINTING][NSTATION]


    /* Fix from Maceij regarding NaNs in output when running on Athena, 13 April 2018.
       Apparently the different compilers and architectures are treating what were 
       unintialised variables very differently */
    Bx[BN_IDX(p,ant)]  = CMaked( 0.0, 0.0 );
    By[BN_IDX(p,ant)]  = CMaked( 0.0, 0.0 );

    Dx  = CMaked( 0.0, 0.0 );
    Dy  = CMaked( 0.0, 0.0 );

    WDx = CMaked( 0.0, 0.0 );
    WDy = CMaked( 0.0, 0.0 );

    Nxx[BN_IDX(p,ant)] = CMaked( 0.0, 0.0 );
    Nxy[BN_IDX(p,ant)] = CMaked( 0.0, 0.0 );
    //Nyx[BN_IDX(p,ant)] = CMaked( 0.0, 0.0 );
    Nyy[BN_IDX(p,ant)] = CMaked( 0.0, 0.0 );

    if (p == 0) Ia[ant] = 0.0;

    // Calculate beamform products for each antenna, and then add them together
    // Calculate the coherent beam (B = J*W*D)
    if ((p == 0) && (ant == 0) && (c == 0) && (s == 0))
    {
        stop = clock();
        setup_t = (double)(stop - start) / CLOCKS_PER_SEC * NPOINTING * NANT;
        start = clock();
    }

    Dx  = UCMPLX4_TO_CMPLX_FLT(data[D_IDX(s,c,ant,0,nc)]);
    Dy  = UCMPLX4_TO_CMPLX_FLT(data[D_IDX(s,c,ant,1,nc)]);

    if (p == 0) Ia[ant] = DETECT(Dx) + DETECT(Dy);

    WDx = CMuld( W[W_IDX(p,ant,c,0,nc)], Dx );
    WDy = CMuld( W[W_IDX(p,ant,c,1,nc)], Dy );

    Bx[BN_IDX(p,ant)] = CAddd( CMuld( J[J_IDX(p,ant,c,0,0,nc)], WDx ),
                               CMuld( J[J_IDX(p,ant,c,1,0,nc)], WDy ) );
    By[BN_IDX(p,ant)] = CAddd( CMuld( J[J_IDX(p,ant,c,0,1,nc)], WDx ),
                               CMuld( J[J_IDX(p,ant,c,1,1,nc)], WDy ) );

    Nxx[BN_IDX(p,ant)] = CMuld( Bx[BN_IDX(p,ant)], CConjd(Bx[BN_IDX(p,ant)]) );
    Nxy[BN_IDX(p,ant)] = CMuld( Bx[BN_IDX(p,ant)], CConjd(By[BN_IDX(p,ant)]) );
    //Nyx[BN_IDX(p,ant)] = CMuld( By[BN_IDX(p,ant)], CConjd(Bx[BN_IDX(p,ant)]) );
    Nyy[BN_IDX(p,ant)] = CMuld( By[BN_IDX(p,ant)], CConjd(By[BN_IDX(p,ant)]) );

    // Detect the coherent beam
    // A summation over an array is faster on a GPU if you add half on array 
    // to its other half as than can be done in parallel. Then this is repeated
    // with half of the previous array until the array is down to 1.
    __syncthreads();

    if ((p == 0) && (ant == 0) && (c == 0) && (s == 0))
    {
        stop = clock();
        detect_t = (double)(stop - start) / CLOCKS_PER_SEC * NPOINTING * NANT;
        start = clock();
    }
    if (ant < 64)
    {
        if (p == 0) Ia[ant] += Ia[ant+64];
        Bx[BN_IDX(p,ant)] = CAddd( Bx[BN_IDX(p,ant)], Bx[BN_IDX(p,ant+64)] );
        By[BN_IDX(p,ant)] = CAddd( By[BN_IDX(p,ant)], By[BN_IDX(p,ant+64)] );
        Nxx[BN_IDX(p,ant)] = CAddd( Nxx[BN_IDX(p,ant)], Nxx[BN_IDX(p,ant+64)] );
        Nxy[BN_IDX(p,ant)] = CAddd( Nxy[BN_IDX(p,ant)], Nxy[BN_IDX(p,ant+64)] );
        //Nyx[BN_IDX(p,ant)] = CAddd( Nyx[BN_IDX(p,ant)], Nyx[BN_IDX(p,ant+64)] );
        Nyy[BN_IDX(p,ant)] = CAddd( Nyy[BN_IDX(p,ant)], Nyy[BN_IDX(p,ant+64)] );
    }
    __syncthreads();
    if (ant < 32)
    {
        if (p == 0) Ia[ant] += Ia[ant+32];
        Bx[BN_IDX(p,ant)] = CAddd( Bx[BN_IDX(p,ant)], Bx[BN_IDX(p,ant+32)] );
        By[BN_IDX(p,ant)] = CAddd( By[BN_IDX(p,ant)], By[BN_IDX(p,ant+32)] );
        Nxx[BN_IDX(p,ant)] = CAddd( Nxx[BN_IDX(p,ant)], Nxx[BN_IDX(p,ant+32)] );
        Nxy[BN_IDX(p,ant)] = CAddd( Nxy[BN_IDX(p,ant)], Nxy[BN_IDX(p,ant+32)] );
        //Nyx[BN_IDX(p,ant)] = CAddd( Nyx[BN_IDX(p,ant)], Nyx[BN_IDX(p,ant+32)] );
        Nyy[BN_IDX(p,ant)] = CAddd( Nyy[BN_IDX(p,ant)], Nyy[BN_IDX(p,ant+32)] );
    }
    __syncthreads();
    if (ant < 16)
    {
        if (p == 0) Ia[ant] += Ia[ant+16];
        Bx[BN_IDX(p,ant)] = CAddd( Bx[BN_IDX(p,ant)], Bx[BN_IDX(p,ant+16)] );
        By[BN_IDX(p,ant)] = CAddd( By[BN_IDX(p,ant)], By[BN_IDX(p,ant+16)] );
        Nxx[BN_IDX(p,ant)] = CAddd( Nxx[BN_IDX(p,ant)], Nxx[BN_IDX(p,ant+16)] );
        Nxy[BN_IDX(p,ant)] = CAddd( Nxy[BN_IDX(p,ant)], Nxy[BN_IDX(p,ant+16)] );
        //Nyx[BN_IDX(p,ant)] = CAddd( Nyx[BN_IDX(p,ant)], Nyx[BN_IDX(p,ant+16)] );
        Nyy[BN_IDX(p,ant)] = CAddd( Nyy[BN_IDX(p,ant)], Nyy[BN_IDX(p,ant+16)] );
    }
    __syncthreads();
    if (ant < 8)
    {
        if (p == 0) Ia[ant] += Ia[ant+8];
        Bx[BN_IDX(p,ant)] = CAddd( Bx[BN_IDX(p,ant)], Bx[BN_IDX(p,ant+8)] );
        By[BN_IDX(p,ant)] = CAddd( By[BN_IDX(p,ant)], By[BN_IDX(p,ant+8)] );
        Nxx[BN_IDX(p,ant)] = CAddd( Nxx[BN_IDX(p,ant)], Nxx[BN_IDX(p,ant+8)] );
        Nxy[BN_IDX(p,ant)] = CAddd( Nxy[BN_IDX(p,ant)], Nxy[BN_IDX(p,ant+8)] );
        //Nyx[BN_IDX(p,ant)] = CAddd( Nyx[BN_IDX(p,ant)], Nyx[BN_IDX(p,ant+8)] );
        Nyy[BN_IDX(p,ant)] = CAddd( Nyy[BN_IDX(p,ant)], Nyy[BN_IDX(p,ant+8)] );
    }
    __syncthreads();
    if (ant < 4)
    {
        if (p == 0) Ia[ant] += Ia[ant+4];
        Bx[BN_IDX(p,ant)] = CAddd( Bx[BN_IDX(p,ant)], Bx[BN_IDX(p,ant+4)] );
        By[BN_IDX(p,ant)] = CAddd( By[BN_IDX(p,ant)], By[BN_IDX(p,ant+4)] );
        Nxx[BN_IDX(p,ant)] = CAddd( Nxx[BN_IDX(p,ant)], Nxx[BN_IDX(p,ant+4)] );
        Nxy[BN_IDX(p,ant)] = CAddd( Nxy[BN_IDX(p,ant)], Nxy[BN_IDX(p,ant+4)] );
        //Nyx[BN_IDX(p,ant)] = CAddd( Nyx[BN_IDX(p,ant)], Nyx[BN_IDX(p,ant+4)] );
        Nyy[BN_IDX(p,ant)] = CAddd( Nyy[BN_IDX(p,ant)], Nyy[BN_IDX(p,ant+4)] );
    }
    __syncthreads();
    if (ant < 2)
    {
        if (p == 0) Ia[ant] += Ia[ant+2];
        Bx[BN_IDX(p,ant)] = CAddd( Bx[BN_IDX(p,ant)], Bx[BN_IDX(p,ant+2)] );
        By[BN_IDX(p,ant)] = CAddd( By[BN_IDX(p,ant)], By[BN_IDX(p,ant+2)] );
        Nxx[BN_IDX(p,ant)] = CAddd( Nxx[BN_IDX(p,ant)], Nxx[BN_IDX(p,ant+2)] );
        Nxy[BN_IDX(p,ant)] = CAddd( Nxy[BN_IDX(p,ant)], Nxy[BN_IDX(p,ant+2)] );
        //Nyx[BN_IDX(p,ant)] = CAddd( Nyx[BN_IDX(p,ant)], Nyx[BN_IDX(p,ant+2)] );
        Nyy[BN_IDX(p,ant)] = CAddd( Nyy[BN_IDX(p,ant)], Nyy[BN_IDX(p,ant+2)] );
    }
    __syncthreads();
    if (ant < 1)
    {
        if (p == 0) Ia[ant] += Ia[ant+1];
        Bx[BN_IDX(p,ant)] = CAddd( Bx[BN_IDX(p,ant)], Bx[BN_IDX(p,ant+1)] );
        By[BN_IDX(p,ant)] = CAddd( By[BN_IDX(p,ant)], By[BN_IDX(p,ant+1)] );
        Nxx[BN_IDX(p,ant)] = CAddd( Nxx[BN_IDX(p,ant)], Nxx[BN_IDX(p,ant+1)] );
        Nxy[BN_IDX(p,ant)] = CAddd( Nxy[BN_IDX(p,ant)], Nxy[BN_IDX(p,ant+1)] );
        //Nyx[BN_IDX(p,ant)] = CAddd( Nyx[BN_IDX(p,ant)], Nyx[BN_IDX(p,ant+1)] );
        Nyy[BN_IDX(p,ant)] = CAddd( Nyy[BN_IDX(p,ant)], Nyy[BN_IDX(p,ant+1)] );
    }
    __syncthreads();
    if ((p == 0) && (ant == 0) && (c == 0) && (s == 0))
    {
        stop = clock();
        sum_t = (double)(stop - start) / CLOCKS_PER_SEC * NPOINTING * NANT;
        start = clock();

    }
    
    // Form the stokes parameters for the coherent beam
    // Only doing it for ant 0 so that it only prints once
    if (ant == 0)
    {
        float bnXX = DETECT(Bx[BN_IDX(p,0)]) - CReald(Nxx[BN_IDX(p,0)]);
        float bnYY = DETECT(By[BN_IDX(p,0)]) - CReald(Nyy[BN_IDX(p,0)]);
        ComplexDouble bnXY = CSubd(
                                 CMuld( Bx[BN_IDX(p,0)], CConjd( By[BN_IDX(p,0)] ) ),
                                 Nxy[BN_IDX(p,0)] );

        // The incoherent beam
        I[I_IDX(s,c,nc)] = Ia[0];

        // Stokes I, Q, U, V:
        C[C_IDX(p,s,0,c,ns,nc)] = invw*(bnXX + bnYY);
        C[C_IDX(p,s,1,c,ns,nc)] = invw*(bnXX - bnYY);
        C[C_IDX(p,s,2,c,ns,nc)] =  2.0*invw*CReald( bnXY );
        C[C_IDX(p,s,3,c,ns,nc)] = -2.0*invw*CImagd( bnXY );

        // The beamformed products
        Bd[B_IDX(p,s,c,0,ns,nc)] = Bx[BN_IDX(p,0)];
        Bd[B_IDX(p,s,c,1,ns,nc)] = By[BN_IDX(p,0)];
    }
    if ((p == 0) && (ant == 0) && (c == 0) && (s == 0))
    {
        stop = clock();
        stokes_t = (double)(stop - start) / CLOCKS_PER_SEC * NPOINTING * NANT;
        printf("Time:  setup: % f detect: %f    sum: %f     stokes: %f\n", setup_t, detect_t, sum_t, stokes_t);
    }
    
}

__global__ void flatten_bandpass_I_kernel(float *I,
                                     int nstep)/* uint8_t *Iout ) */
{
    // For just doing stokes I
    // One block
    // 128 threads each thread will do one channel
    // (we have already summed over all ant)

    // For doing the C array (I,Q,U,V)
    // ... figure it out later.

    // Translate GPU block/thread numbers into meaningful names
    int chan = threadIdx.x; /* The (c)hannel number */
    int nchan = blockDim.x; /* The total number of channels */
    float band;

    int new_var = 32; /* magic number */
    int i;

    float *data_ptr;

    // initialise the band 'array'
    band = 0.0;

    // accumulate abs(data) over all time samples and save into band
    data_ptr = I + I_IDX(0, chan, nchan);
    for (i=0;i<nstep;i++) { // time steps
        band += fabsf(*data_ptr);
        data_ptr = I + I_IDX(i,chan,nchan);
    }

    // now normalise the incoherent beam
    data_ptr = I + I_IDX(0, chan, nchan);
    for (i=0;i<nstep;i++) { // time steps
        *data_ptr = (*data_ptr)/( (band/nstep)/new_var );
        data_ptr = I + I_IDX(i,chan,nchan);
    }

}


__global__ void flatten_bandpass_C_kernel(float *C,
                                          int nstep)/* uint8_t *Iout ) */
{
    // For just doing stokes I
    // One block
    // 128 threads each thread will do one channel
    // (we have already summed over all ant)

    // For doing the C array (I,Q,U,V)
    // ... figure it out later.

    // Translate GPU block/thread numbers into meaningful names
    int chan   = threadIdx.x; /* The (c)hannel number */
    int nchan  = blockDim.x; /* The total number of channels */
    int p      = blockIdx.x;
    int stokes = threadIdx.y;
    //int nstokes = blockDim.y;
    float band;

    int new_var = 32; /* magic number */
    int i;

    float *data_ptr;

    // initialise the band 'array'
    band = 0.0;

    // accumulate abs(data) over all time samples and save into band
    //data_ptr = C + C_IDX(0,stokes,chan,nchan);
    for (i=0;i<nstep;i++) { // time steps
        data_ptr = C + C_IDX(p,i,stokes,chan,nstep,nchan);
        band += fabsf(*data_ptr);
    }

    // now normalise the coherent beam
    //data_ptr = C + C_IDX(0,stokes,chan,nchan);
    for (i=0;i<nstep;i++) { // time steps
        data_ptr = C + C_IDX(p,i,stokes,chan,nstep,nchan);
        *data_ptr = (*data_ptr)/( (band/nstep)/new_var );
    }

}


void cu_form_beam( uint8_t *data, struct make_beam_opts *opts,
                   ComplexDouble ****complex_weights_array,
                   ComplexDouble *****invJi, int file_no, 
                   int npointing, int nstation, int nchan,
                   int npol, int outpol_coh, double invw,
                   struct gpu_formbeam_arrays **g,
                   ComplexDouble ****detected_beam, float *coh, float *incoh )
/* The CPU version of the beamforming operations, using OpenMP for
 * parallelisation.
 *
 * Inputs:
 *   data    = array of 4bit+4bit complex numbers. For data order, refer to the
 *             documentation.
 *   opts    = passed option parameters, containing meta information about the
 *             obs and the data
 *   W       = complex weights array. [npointing][nstation][nchan][npol]
 *   J       = inverse Jones matrix. [npointing][nstation][nchan][npol][npol]
 *   file_no = number of file we are processing, starting at 0.
 *   nstation     = 128
 *   nchan        = 128
 *   npol         = 2 (X,Y)
 *   outpol_coh   = 4 (I,Q,U,V)
 *   invw         = the reciprocal of the sum of the antenna weights
 *   g            = struct containing pointers to various arrays on
 *                  both host and device
 *
 * Outputs:
 *   detected_beam = result of beamforming operation, summed over antennas
 *                   [2*nsamples][nchan][npol]
 *   coh           = result in Stokes parameters (minus noise floor)
 *                   [nsamples][nstokes][nchan]
 *   incoh         = result (just Stokes I)
 *                   [nsamples][nchan]
 *
 * Assumes "coh" and "incoh" contain only zeros.
 */
{
    // Setup input values (= populate W and J)
    int p, s, ant, ch, pol, pol2;
    int Wi, Ji;
    for (p   = 0; p   < npointing; p++  )
    for (ant = 0; ant < nstation ; ant++)
    for (ch  = 0; ch  < nchan    ; ch++ )
    for (pol = 0; pol < npol     ; pol++)
    {
        Wi = p   * (npol*nchan*nstation) +
             ant * (npol*nchan) +
             ch  * (npol) +
             pol;
        (*g)->W[Wi] = complex_weights_array[p][ant][ch][pol];

        for (pol2 = 0; pol2 < npol; pol2++)
        {
            Ji = Wi*npol + pol2;
            (*g)->J[Ji] = invJi[p][ant][ch][pol][pol2];
        }
    }

    // Copy the data to the device
    gpuErrchk(hipMemcpy( (*g)->d_data, data,    (*g)->data_size, hipMemcpyHostToDevice ));
    gpuErrchk(hipMemcpy( (*g)->d_W,    (*g)->W, (*g)->W_size,    hipMemcpyHostToDevice ));
    gpuErrchk(hipMemcpy( (*g)->d_J,    (*g)->J, (*g)->J_size,    hipMemcpyHostToDevice ));
    
    // Call the kernels
    dim3 samples_chan(opts->sample_rate, nchan);
    dim3 stat_point(NSTATION, npointing);
    beamform_kernel<<<samples_chan, stat_point>>>(
            (*g)->d_data, (*g)->d_W, (*g)->d_J, invw, (*g)->d_Bd, (*g)->d_coh, (*g)->d_incoh );
    //hipDeviceSynchronize();
    // sync not required between kernel queues since each stream acts like a FIFO queue
    // so all instances of the above kernel will complete before we move to the next
    // we are using the "default" stream since we don't specify any stream id

    // 1 block per pointing direction, hence the 1 for now
    // TODO check if these actually work, can't see them return values.
    // The incoh kernal also takes 40 second for some reason so commenting out
    //flatten_bandpass_I_kernel<<<1, nchan>>>((*g)->d_incoh, opts->sample_rate);
    //hipDeviceSynchronize();

    // now do the same for the coherent beam
    dim3 chan_stokes(nchan, outpol_coh);
    //flatten_bandpass_C_kernel<<<npointing, chan_stokes>>>((*g)->d_coh, opts->sample_rate);
    //hipDeviceSynchronize(); // Memcpy acts as a synchronize step so don't sync here
    
    // Copy the results back into host memory
    gpuErrchk(hipMemcpy( (*g)->Bd, (*g)->d_Bd,    (*g)->Bd_size,    hipMemcpyDeviceToHost ));
    gpuErrchk(hipMemcpy( incoh,    (*g)->d_incoh, (*g)->incoh_size, hipMemcpyDeviceToHost ));
    gpuErrchk(hipMemcpy( coh,      (*g)->d_coh,   (*g)->coh_size,   hipMemcpyDeviceToHost ));
    
    // Copy the data back from Bd back into the detected_beam array
    // Make sure we put it back into the correct half of the array, depending
    // on whether this is an even or odd second.
    int offset, i;
    offset = file_no % 3 * opts->sample_rate;
    
    for (p   = 0; p   < npointing        ; p++  )
    for (s   = 0; s   < opts->sample_rate; s++  )
    for (ch  = 0; ch  < nchan            ; ch++ )
    for (pol = 0; pol < npol             ; pol++)
    {
        i = p  * (npol*nchan*opts->sample_rate) +
            s  * (npol*nchan)                   +
            ch * (npol)                         +
            pol;

        detected_beam[p][s+offset][ch][pol] = (*g)->Bd[i];
    }
}

void malloc_formbeam( struct gpu_formbeam_arrays **g, unsigned int sample_rate,
        int nstation, int nchan, int npol, int outpol_coh, int outpol_incoh, int npointing)
{
    // Calculate array sizes for host and device
    (*g)->coh_size   = npointing * sample_rate * outpol_coh   * nchan * sizeof(float);
    (*g)->incoh_size = sample_rate * outpol_incoh * nchan * sizeof(float);
    (*g)->data_size  = sample_rate * nstation * nchan * npol * sizeof(uint8_t);
    (*g)->Bd_size    = npointing * sample_rate * nchan * npol * sizeof(ComplexDouble);
    (*g)->W_size     = npointing * nstation * nchan * npol * sizeof(ComplexDouble);
    (*g)->J_size     = npointing * nstation * nchan * npol * npol * sizeof(ComplexDouble);

    // Allocate host memory
    (*g)->W  = (ComplexDouble *)malloc( (*g)->W_size );
    (*g)->J  = (ComplexDouble *)malloc( (*g)->J_size );
    (*g)->Bd = (ComplexDouble *)malloc( (*g)->Bd_size );


    // Allocate device memory
    gpuErrchk(hipMalloc( (void **)&(*g)->d_W,     (*g)->W_size ));
    gpuErrchk(hipMalloc( (void **)&(*g)->d_J,     (*g)->J_size ));
    gpuErrchk(hipMalloc( (void **)&(*g)->d_Bd,    (*g)->Bd_size ));
    gpuErrchk(hipMalloc( (void **)&(*g)->d_data,  (*g)->data_size ));
    gpuErrchk(hipMalloc( (void **)&(*g)->d_coh,   (*g)->coh_size ));
    gpuErrchk(hipMalloc( (void **)&(*g)->d_incoh, (*g)->incoh_size ));

    printf("%d GB GPU memory allocated\n", ((*g)->W_size + (*g)->J_size + 
                                            (*g)->Bd_size + (*g)->data_size +
                                            (*g)->coh_size + (*g)->incoh_size)
                                            /1000000000 );
}

void free_formbeam( struct gpu_formbeam_arrays **g )
{
    // Free memory on host and device
    free( (*g)->W );
    free( (*g)->J );
    free( (*g)->Bd );
    hipFree( (*g)->d_W );
    hipFree( (*g)->d_J );
    hipFree( (*g)->d_Bd );
    hipFree( (*g)->d_data );
    hipFree( (*g)->d_coh );
    hipFree( (*g)->d_incoh );
}
