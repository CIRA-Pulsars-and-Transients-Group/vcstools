#include "hip/hip_runtime.h"
/********************************************************
 *                                                      *
 * Licensed under the Academic Free License version 3.0 *
 *                                                      *
 ********************************************************/

#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>

extern "C" {
#include "beam_common.h"
#include "form_beam.h"
#include "mycomplex.h"
}


#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    /* Wrapper function for GPU/CUDA error handling. Every CUDA call goes through
       this function. It will return a message giving your the error string,
       file name and line of the error. Aborts on error. */

    if (code != 0)
    {
        fprintf(stderr, "GPUAssert:: %s - %s (%d)\n", hipGetErrorString(code), file, line);
        if (abort)
        {
            exit(code);
        }
    }
}

// define a macro for accessing gpuAssert
#define gpuErrchk(ans) {gpuAssert((ans), __FILE__, __LINE__, true);}


// define constants to be used in the kernel
#define NSTATION  128
#define NPOL      2
#define NSTOKES   4
// maximum number of pointings (currently)
#define NPOINTING 4



__global__ void beamform_kernel( uint8_t *data,
                                 ComplexDouble *W,
                                 ComplexDouble *J,
                                 double invw,
                                 ComplexDouble *Bd,
                                 float *C,
                                 float *I,
                                 int p)
/* Layout for input arrays:
 *   data [nsamples] [nchan] [NPFB] [NREC] [NINC] -- see docs
 *   W    [NSTATION] [nchan] [NPOL]               -- weights array
 *   J    [NSTATION] [nchan] [NPOL] [NPOL]        -- jones matrix
 * Layout for output arrays:
 *   Bd   [nsamples] [nchan]   [NPOL]             -- detected beam
 *   C    [nsamples] [NSTOKES] [nchan]            -- coherent full stokes
 *   I    [nsamples] [nchan]                      -- incoherent
 */
{
    // Translate GPU block/thread numbers into meaning->l names
    int s    = blockIdx.x;  /* The (s)ample number */
    int ns   = gridDim.x;   /* The (n)umber of (s)amples (=10000)*/
    int c    = blockIdx.y;  /* The (c)hannel number */
    int nc   = gridDim.y;   /* The (n)umber of (c)hannels (=128) */
    
    int ant  = threadIdx.x; /* The (ant)enna number */
    int nant = blockDim.x;  /* The (n_umber of (ant)ennas */

    // GPU profiling
    clock_t start, stop;
    double setup_t, detect_t, sum_t, stokes_t;
    if ((p == 0) && (ant == 0) && (c == 0) && (s == 0)) start = clock();
    
    // Calculate the beam and the noise floor
    __shared__ double Ia[NSTATION];
    __shared__ ComplexDouble Bx[NSTATION], By[NSTATION];
    ComplexDouble Dx, Dy;
    ComplexDouble WDx, WDy;

    __shared__ ComplexDouble Nxx[NSTATION], Nxy[NSTATION],
                             Nyy[NSTATION];//Nyx[NSTATION]


    /* Fix from Maceij regarding NaNs in output when running on Athena, 13 April 2018.
       Apparently the different compilers and architectures are treating what were 
       unintialised variables very differently */
    Bx[ant]  = CMaked( 0.0, 0.0 );
    By[ant]  = CMaked( 0.0, 0.0 );

    Dx  = CMaked( 0.0, 0.0 );
    Dy  = CMaked( 0.0, 0.0 );

    WDx = CMaked( 0.0, 0.0 );
    WDy = CMaked( 0.0, 0.0 );

    Nxx[ant] = CMaked( 0.0, 0.0 );
    Nxy[ant] = CMaked( 0.0, 0.0 );
    //Nyx[ant] = CMaked( 0.0, 0.0 );
    Nyy[ant] = CMaked( 0.0, 0.0 );

    if (p == 0) Ia[ant] = 0.0;

    // Calculate beamform products for each antenna, and then add them together
    // Calculate the coherent beam (B = J*W*D)
    if ((p == 0) && (ant == 0) && (c == 0) && (s == 0))
    {
        stop = clock();
        setup_t = (double)(stop - start) / CLOCKS_PER_SEC * NPOINTING * NANT;
        start = clock();
    }

    Dx  = UCMPLX4_TO_CMPLX_FLT(data[D_IDX(s,c,ant,0,nc)]);
    Dy  = UCMPLX4_TO_CMPLX_FLT(data[D_IDX(s,c,ant,1,nc)]);

    if (p == 0) Ia[ant] = DETECT(Dx) + DETECT(Dy);

    WDx = CMuld( W[W_IDX(p,ant,c,0,nc)], Dx );
    WDy = CMuld( W[W_IDX(p,ant,c,1,nc)], Dy );

    Bx[ant] = CAddd( CMuld( J[J_IDX(p,ant,c,0,0,nc)], WDx ),
                               CMuld( J[J_IDX(p,ant,c,1,0,nc)], WDy ) );
    By[ant] = CAddd( CMuld( J[J_IDX(p,ant,c,0,1,nc)], WDx ),
                               CMuld( J[J_IDX(p,ant,c,1,1,nc)], WDy ) );

    Nxx[ant] = CMuld( Bx[ant], CConjd(Bx[ant]) );
    Nxy[ant] = CMuld( Bx[ant], CConjd(By[ant]) );
    //Nyx[ant] = CMuld( By[ant], CConjd(Bx[ant]) );
    Nyy[ant] = CMuld( By[ant], CConjd(By[ant]) );

    // Detect the coherent beam
    // A summation over an array is faster on a GPU if you add half on array 
    // to its other half as than can be done in parallel. Then this is repeated
    // with half of the previous array until the array is down to 1.
    __syncthreads();

    if ((p == 0) && (ant == 0) && (c == 0) && (s == 0))
    {
        stop = clock();
        detect_t = (double)(stop - start) / CLOCKS_PER_SEC * NPOINTING * NANT;
        start = clock();
    }
    for ( int h_ant = nant / 2; h_ant > 0; h_ant = h_ant / 2 )
    {
        if (ant < h_ant)
        {
            if (p == 0) Ia[ant] += Ia[ant+h_ant];
            Bx[ant]  = CAddd( Bx[ant],  Bx[ant  + h_ant] );
            By[ant]  = CAddd( By[ant],  By[ant  + h_ant] );
            Nxx[ant] = CAddd( Nxx[ant], Nxx[ant + h_ant] );
            Nxy[ant] = CAddd( Nxy[ant], Nxy[ant + h_ant] );
            //Nyx[ant]=CAddd( Nyx[ant], Nyx[ant + h_ant] );
            Nyy[ant] = CAddd( Nyy[ant], Nyy[ant + h_ant] );
        }
        // below makes no difference so removed
        // else return;
        __syncthreads();
    }

    if ((p == 0) && (ant == 0) && (c == 0) && (s == 0))
    {
        stop = clock();
        sum_t = (double)(stop - start) / CLOCKS_PER_SEC * NPOINTING * NANT;
        start = clock();

    }
    
    // Form the stokes parameters for the coherent beam
    // Only doing it for ant 0 so that it only prints once
    if (ant == 0)
    {
        float bnXX = DETECT(Bx[0]) - CReald(Nxx[0]);
        float bnYY = DETECT(By[0]) - CReald(Nyy[0]);
        ComplexDouble bnXY = CSubd(
                                 CMuld( Bx[0], CConjd( By[0] ) ),
                                 Nxy[0] );

        // The incoherent beam
        I[I_IDX(s,c,nc)] = Ia[0];

        // Stokes I, Q, U, V:
        C[C_IDX(p,s,0,c,ns,nc)] = invw*(bnXX + bnYY);
        C[C_IDX(p,s,1,c,ns,nc)] = invw*(bnXX - bnYY);
        C[C_IDX(p,s,2,c,ns,nc)] =  2.0*invw*CReald( bnXY );
        C[C_IDX(p,s,3,c,ns,nc)] = -2.0*invw*CImagd( bnXY );

        // The beamformed products
        Bd[B_IDX(p,s,c,0,ns,nc)] = Bx[0];
        Bd[B_IDX(p,s,c,1,ns,nc)] = By[0];
    }
    if ((p == 0) && (ant == 0) && (c == 0) && (s == 0))
    {
        stop = clock();
        stokes_t = (double)(stop - start) / CLOCKS_PER_SEC * NPOINTING * NANT;
        printf("Time:  setup: % f detect: %f    sum: %f     stokes: %f\n", setup_t, detect_t, sum_t, stokes_t);
    }
    
}

__global__ void flatten_bandpass_I_kernel(float *I,
                                     int nstep)/* uint8_t *Iout ) */
{
    // For just doing stokes I
    // One block
    // 128 threads each thread will do one channel
    // (we have already summed over all ant)

    // For doing the C array (I,Q,U,V)
    // ... figure it out later.

    // Translate GPU block/thread numbers into meaningful names
    int chan = threadIdx.x; /* The (c)hannel number */
    int nchan = blockDim.x; /* The total number of channels */
    float band;

    int new_var = 32; /* magic number */
    int i;

    float *data_ptr;

    // initialise the band 'array'
    band = 0.0;

    // accumulate abs(data) over all time samples and save into band
    data_ptr = I + I_IDX(0, chan, nchan);
    for (i=0;i<nstep;i++) { // time steps
        band += fabsf(*data_ptr);
        data_ptr = I + I_IDX(i,chan,nchan);
    }

    // now normalise the incoherent beam
    data_ptr = I + I_IDX(0, chan, nchan);
    for (i=0;i<nstep;i++) { // time steps
        *data_ptr = (*data_ptr)/( (band/nstep)/new_var );
        data_ptr = I + I_IDX(i,chan,nchan);
    }

}


__global__ void flatten_bandpass_C_kernel(float *C,
                                          int nstep)/* uint8_t *Iout ) */
{
    // For just doing stokes I
    // One block
    // 128 threads each thread will do one channel
    // (we have already summed over all ant)

    // For doing the C array (I,Q,U,V)
    // ... figure it out later.

    // Translate GPU block/thread numbers into meaningful names
    int chan   = threadIdx.x; /* The (c)hannel number */
    int nchan  = blockDim.x;  /* The (n)umber of (c)hannels */
    int stokes = threadIdx.y; /* The (stokes) number */
    
    int p      = blockIdx.x;  /* The (p)ointing number */

    float band;

    int new_var = 32; /* magic number */
    int i;

    float *data_ptr;

    // initialise the band 'array'
    band = 0.0;

    // accumulate abs(data) over all time samples and save into band
    //data_ptr = C + C_IDX(0,stokes,chan,nchan);
    for (i=0;i<nstep;i++) { // time steps
        data_ptr = C + C_IDX(p,i,stokes,chan,nstep,nchan);
        band += fabsf(*data_ptr);
    }

    // now normalise the coherent beam
    //data_ptr = C + C_IDX(0,stokes,chan,nchan);
    for (i=0;i<nstep;i++) { // time steps
        data_ptr = C + C_IDX(p,i,stokes,chan,nstep,nchan);
        *data_ptr = (*data_ptr)/( (band/nstep)/new_var );
    }

}


void cu_form_beam( uint8_t *data, struct make_beam_opts *opts,
                   ComplexDouble ****complex_weights_array,
                   ComplexDouble *****invJi, int file_no, 
                   int npointing, int nstation, int nchan,
                   int npol, int outpol_coh, double invw,
                   struct gpu_formbeam_arrays *g,
                   ComplexDouble ****detected_beam, float *coh, float *incoh,
                   hipStream_t *streams )
/* The CPU version of the beamforming operations, using OpenMP for
 * parallelisation.
 *
 * Inputs:
 *   data    = array of 4bit+4bit complex numbers. For data order, refer to the
 *             documentation.
 *   opts    = passed option parameters, containing meta information about the
 *             obs and the data
 *   W       = complex weights array. [npointing][nstation][nchan][npol]
 *   J       = inverse Jones matrix. [npointing][nstation][nchan][npol][npol]
 *   file_no = number of file we are processing, starting at 0.
 *   nstation     = 128
 *   nchan        = 128
 *   npol         = 2 (X,Y)
 *   outpol_coh   = 4 (I,Q,U,V)
 *   invw         = the reciprocal of the sum of the antenna weights
 *   g            = struct containing pointers to various arrays on
 *                  both host and device
 *
 * Outputs:
 *   detected_beam = result of beamforming operation, summed over antennas
 *                   [2*nsamples][nchan][npol]
 *   coh           = result in Stokes parameters (minus noise floor)
 *                   [nsamples][nstokes][nchan]
 *   incoh         = result (just Stokes I)
 *                   [nsamples][nchan]
 *
 * Assumes "coh" and "incoh" contain only zeros.
 */
{
    // Setup input values (= populate W and J)
    int p, ant, ch, pol, pol2;
    int Wi, Ji;
    for (p   = 0; p   < npointing; p++  )
    for (ant = 0; ant < nstation ; ant++)
    for (ch  = 0; ch  < nchan    ; ch++ )
    for (pol = 0; pol < npol     ; pol++)
    {
        Wi = p   * (npol*nchan*nstation) +
             ant * (npol*nchan) +
             ch  * (npol) +
             pol;
        g->W[Wi] = complex_weights_array[p][ant][ch][pol];

        for (pol2 = 0; pol2 < npol; pol2++)
        {
            Ji = Wi*npol + pol2;
            g->J[Ji] = invJi[p][ant][ch][pol][pol2];
        }
    }
    // Copy the data to the device
    gpuErrchk(hipMemcpyAsync( g->d_data, data, g->data_size, hipMemcpyHostToDevice ));
    gpuErrchk(hipMemcpyAsync( g->d_W,    g->W, g->W_size,    hipMemcpyHostToDevice ));
    gpuErrchk(hipMemcpyAsync( g->d_J,    g->J, g->J_size,    hipMemcpyHostToDevice ));

    // Call the kernels
    // sammples_chan(index=blockIdx.x  size=gridDim.x,
    //               index=blockIdx.y  size=gridDim.y)
    // stat_point   (index=threadIdx.x size=blockDim.x,
    //               index=threadIdx.y size=blockDim.y)
    dim3 samples_chan(opts->sample_rate, nchan);
    dim3 stat(NSTATION);
    // Send off a parrellel cuda stream for each pointing
    for ( p = 0; p < npointing; p++ )
    {    
        beamform_kernel<<<samples_chan, stat, 0, streams[p]>>>( g->d_data,
                            g->d_W, g->d_J, invw, 
                            g->d_Bd, g->d_coh, g->d_incoh, p );
            
        gpuErrchk( hipPeekAtLastError() );
        //hipDeviceSynchronize();
        // sync not required between kernel queues since each stream acts like a FIFO queue
        // so all instances of the above kernel will complete before we move to the next
        // we are using the "default" stream since we don't specify any stream id

        // 1 block per pointing direction, hence the 1 for now
        // TODO check if these actually work, can't see them return values.
        // The incoh kernal also takes 40 second for some reason so commenting out
        if ( p == 0 )
            flatten_bandpass_I_kernel<<<1, nchan, 0, streams[p]>>>(g->d_incoh,
                                                     opts->sample_rate);
        //hipDeviceSynchronize();

        // now do the same for the coherent beam
        dim3 chan_stokes(nchan, outpol_coh);
        // This doesn't seem to change anything some commenting out
        flatten_bandpass_C_kernel<<<npointing, chan_stokes, 0, streams[p]>>>(g->d_coh, 
                                                               opts->sample_rate);
        //hipDeviceSynchronize(); // Memcpy acts as a synchronize step so don't sync here
    }
    gpuErrchk( hipDeviceSynchronize() );
    
    // Copy the results back into host memory
    gpuErrchk(hipMemcpyAsync( g->Bd, g->d_Bd,    g->Bd_size,    hipMemcpyDeviceToHost ));
    gpuErrchk(hipMemcpyAsync( incoh, g->d_incoh, g->incoh_size, hipMemcpyDeviceToHost ));
    gpuErrchk(hipMemcpyAsync( coh,   g->d_coh,   g->coh_size,   hipMemcpyDeviceToHost ));
    
    // Copy the data back from Bd back into the detected_beam array
    // Make sure we put it back into the correct half of the array, depending
    // on whether this is an even or odd second.
    int offset, i;
    offset = file_no % 3 * opts->sample_rate;
    
    for ( int p   = 0; p   < npointing        ; p++  )
    for ( int s   = 0; s   < opts->sample_rate; s++  )
    for ( int ch  = 0; ch  < nchan            ; ch++ )
    for ( int pol = 0; pol < npol             ; pol++)
    {
        i = p  * (npol*nchan*opts->sample_rate) +
            s  * (npol*nchan)                   +
            ch * (npol)                         +
            pol;

        detected_beam[p][s+offset][ch][pol] = g->Bd[i];
    }
}

void malloc_formbeam( struct gpu_formbeam_arrays *g, unsigned int sample_rate,
        int nstation, int nchan, int npol, int outpol_coh, int outpol_incoh, int npointing)
{
    // Calculate array sizes for host and device
    g->coh_size   = npointing * sample_rate * outpol_coh   * nchan * sizeof(float);
    g->incoh_size = sample_rate * outpol_incoh * nchan * sizeof(float);
    g->data_size  = sample_rate * nstation * nchan * npol * sizeof(uint8_t);
    g->Bd_size    = npointing * sample_rate * nchan * npol * sizeof(ComplexDouble);
    g->W_size     = npointing * nstation * nchan * npol * sizeof(ComplexDouble);
    g->J_size     = npointing * nstation * nchan * npol * npol * sizeof(ComplexDouble);

    // Allocate host memory
    //g->W  = (ComplexDouble *)malloc( g->W_size );
    //g->J  = (ComplexDouble *)malloc( g->J_size );
    //g->Bd = (ComplexDouble *)malloc( g->Bd_size );
    hipHostMalloc( &g->W, g->W_size );
    cudaCheckErrors("hipHostMalloc W fail");
    hipHostMalloc( &g->J, g->J_size );
    cudaCheckErrors("hipHostMalloc J fail");
    hipHostMalloc( &g->Bd, g->Bd_size );
    cudaCheckErrors("hipHostMalloc Bd fail");

    // Allocate device memory
    gpuErrchk(hipMalloc( (void **)&g->d_W,     g->W_size ));
    gpuErrchk(hipMalloc( (void **)&g->d_J,     g->J_size ));
    gpuErrchk(hipMalloc( (void **)&g->d_Bd,    g->Bd_size ));
    gpuErrchk(hipMalloc( (void **)&g->d_data,  g->data_size ));
    gpuErrchk(hipMalloc( (void **)&g->d_coh,   g->coh_size ));
    gpuErrchk(hipMalloc( (void **)&g->d_incoh, g->incoh_size ));

    printf("%d GB GPU memory allocated\n", (g->W_size + g->J_size + 
                                            g->Bd_size + g->data_size +
                                            g->coh_size + g->incoh_size)
                                            /1000000000 );
}

void free_formbeam( struct gpu_formbeam_arrays *g )
{
    // Free memory on host and device
    hipHostFree( g->W );
    hipHostFree( g->J );
    hipHostFree( g->Bd );
    hipFree( g->d_W );
    hipFree( g->d_J );
    hipFree( g->d_Bd );
    hipFree( g->d_data );
    hipFree( g->d_coh );
    hipFree( g->d_incoh );
}

float *create_pinned_data_buffer_psrfits( size_t size )
{
    float *ptr;
    hipHostMalloc( &ptr, size * sizeof(float) );
    //hipError_t status = hipHostRegister((void**)&ptr, size * sizeof(float),
    //                                      hipHostRegisterPortable );
    cudaCheckErrors("hipHostMalloc data_buffer_psrfits fail");
    return ptr;
}

float *create_pinned_data_buffer_vdif( size_t size )
{
    float *ptr;
    hipHostMalloc( &ptr, size * sizeof(float) );
    //hipError_t status = hipHostRegister((void**)&ptr, size * sizeof(float),
    //                                      hipHostRegisterPortable );
    cudaCheckErrors("hipHostMalloc data_buffer_vdif fail");
    return ptr;
}

