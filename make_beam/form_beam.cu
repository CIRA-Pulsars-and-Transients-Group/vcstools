#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <hip/hip_runtime.h>

extern "C" {
#include "beam_common.h"
#include "form_beam.h"
#include "mycomplex.h"
}

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    /* Wrapper function for GPU/CUDA error handling. Every CUDA call goes through
       this function. It will return a message giving your the error string,
       file name and line of the error. Aborts on error. */

    if (code != 0)
    {
        fprintf(stderr, "GPUAssert:: %s - %s (%d)\n", hipGetErrorString(code), file, line);
        if (abort)
        {
            exit(code);
        }
    }
}

// define a macro for accessing gpuAssert
#define gpuErrchk(ans) {gpuAssert((ans), __FILE__, __LINE__, true);}


// define constants to be used in the kernel
#define NSTATION  128
#define NPOL      2
#define NSTOKES   4


__global__ void beamform_kernel( uint8_t *data,
                                 ComplexDouble *W,
                                 ComplexDouble *J,
                                 double invw,
                                 ComplexDouble *Bd,
                                 float *C,
                                 float *I )
/* Layout for input arrays:
 *   data [nsamples] [nchan] [NPFB] [NREC] [NINC] -- see docs
 *   W    [NSTATION] [nchan] [NPOL]               -- weights array
 *   J    [NSTATION] [nchan] [NPOL] [NPOL]        -- jones matrix
 * Layout for output arrays:
 *   Bd   [nsamples] [nchan]   [NPOL]             -- detected beam
 *   C    [nsamples] [NSTOKES] [nchan]            -- coherent full stokes
 *   I    [nsamples] [nchan]                      -- incoherent
 */
{
    // Translate GPU block/thread numbers into meaningful names
    int s  = blockIdx.x;  /* The (s)ample number */
    int nc = blockDim.x;  /* The (n)umber of (c)hannels (=128) */
    int c  = threadIdx.x; /* The (c)hannel number */

    int ant;              /* The (ant)enna number */

    // Calculate the beam and the noise floor
    ComplexDouble Bx, By;
    ComplexDouble Dx, Dy;
    ComplexDouble WDx, WDy;
    ComplexDouble Nxx, Nxy, Nyx, Nyy;


    /* Fix from Maceij regarding NaNs in output when running on Athena, 13 April 2018.
       Apparently the different compilers and architectures are treating what were 
       unintialised variables very differently */
    Bx  = CMaked( 0.0, 0.0 );
    By  = CMaked( 0.0, 0.0 );

    Dx  = CMaked( 0.0, 0.0 );
    Dy  = CMaked( 0.0, 0.0 );

    WDx = CMaked( 0.0, 0.0 );
    WDy = CMaked( 0.0, 0.0 );

    Nxx = CMaked( 0.0, 0.0 );
    Nxy = CMaked( 0.0, 0.0 );
    Nyx = CMaked( 0.0, 0.0 );
    Nyy = CMaked( 0.0, 0.0 );

    I[I_IDX(s,c,nc)] = 0.0;
    Bd[B_IDX(s,c,0,nc)] = CMaked( 0.0, 0.0 );
    Bd[B_IDX(s,c,1,nc)] = CMaked( 0.0, 0.0 );


    // Calculate beamform products for each antenna, and then add them together
    for (ant = 0; ant < NSTATION; ant++)
    {
        // Calculate the coherent beam (B = J*W*D)
        Dx  = UCMPLX4_TO_CMPLX_FLT(data[D_IDX(s,c,ant,0,nc)]);
        Dy  = UCMPLX4_TO_CMPLX_FLT(data[D_IDX(s,c,ant,1,nc)]);

        WDx = CMuld( W[W_IDX(c,ant,0,nc)], Dx );
        WDy = CMuld( W[W_IDX(c,ant,1,nc)], Dy );

        // (... and along the way, calculate the incoherent beam...)
        I[I_IDX(s,c,nc)] = DETECT(Dx) + DETECT(Dy);

        Bx = CAddd( CMuld( J[J_IDX(c,ant,0,0,nc)], WDx ),
                    CMuld( J[J_IDX(c,ant,1,0,nc)], WDy ) );
        By = CAddd( CMuld( J[J_IDX(c,ant,0,1,nc)], WDx ),
                    CMuld( J[J_IDX(c,ant,1,1,nc)], WDy ) );

        // Detect the coherent beam
        Bd[B_IDX(s,c,0,nc)] = CAddd( Bd[B_IDX(s,c,0,nc)], Bx );
        Bd[B_IDX(s,c,1,nc)] = CAddd( Bd[B_IDX(s,c,1,nc)], By );

        // Calculate the noise floor (N = B*B')
        Nxx = CAddd( Nxx, CMuld( Bx, CConjd(Bx) ) );
        Nxy = CAddd( Nxy, CMuld( Bx, CConjd(By) ) );
        Nyx = CAddd( Nyx, CMuld( By, CConjd(Bx) ) );
        Nyy = CAddd( Nyy, CMuld( By, CConjd(By) ) );
    }

    // Form the stokes parameters for the coherent beam
    float bnXX = DETECT(Bd[B_IDX(s,c,0,nc)]) - CReald(Nxx);
    float bnYY = DETECT(Bd[B_IDX(s,c,1,nc)]) - CReald(Nyy);
    ComplexDouble bnXY = CSubd(
                             CMuld(
                                 Bd[B_IDX(s,c,0,nc)],
                                 CConjd( Bd[B_IDX(s,c,1,nc)] ) ),
                             Nxy );

    // Stokes I, Q, U, V:
    C[C_IDX(s,c,0,nc)] = invw*(bnXX + bnYY);
    C[C_IDX(s,c,1,nc)] = invw*(bnXX - bnYY);
    C[C_IDX(s,c,2,nc)] =  2.0*invw*CReald( bnXY );
    C[C_IDX(s,c,3,nc)] = -2.0*invw*CImagd( bnXY );

    __syncthreads();
}

void cu_form_beam( uint8_t *data, struct make_beam_opts *opts,
                   ComplexDouble ***complex_weights_array,
                   ComplexDouble ****invJi, int file_no, int nstation, int nchan,
                   int npol, int outpol_coh, int outpol_incoh, double invw,
                   ComplexDouble ***detected_beam, float *coh, float *incoh )
/* The CPU version of the beamforming operations, using OpenMP for
 * parallelisation.
 *
 * Inputs:
 *   data    = array of 4bit+4bit complex numbers. For data order, refer to the
 *             documentation.
 *   opts    = passed option parameters, containing meta information about the
 *             obs and the data
 *   W       = complex weights array. [nstation][nchan][npol]
 *   J       = inverse Jones matrix. [nstation][nchan][npol][npol]
 *   file_no = number of file we are processing, starting at 0.
 *   nstation     = 128
 *   nchan        = 128
 *   npol         = 2 (X,Y)
 *   outpol_coh   = 4 (I,Q,U,V)
 *   outpol_incoh = 1 (I)
 *   invw         = the reciprocal of the sum of the antenna weights
 *
 * Outputs:
 *   detected_beam = result of beamforming operation, summed over antennas
 *                   [2*nsamples][nchan][npol]
 *   coh           = result in Stokes parameters (minus noise floor)
 *                   [nsamples][nstokes][nchan]
 *   incoh         = result (just Stokes I)
 *                   [nsamples][nchan]
 *
 * Assumes "coh" and "incoh" contain only zeros.
 */
{
    // Calculate array sizes for host and device
    size_t coh_size   = opts->sample_rate * outpol_coh   * nchan * sizeof(float);
    size_t incoh_size = opts->sample_rate * outpol_incoh * nchan * sizeof(float);
    size_t data_size  = opts->sample_rate * nstation * nchan * npol * sizeof(uint8_t);
    size_t Bd_size    = opts->sample_rate * nchan * npol * sizeof(ComplexDouble);
    size_t W_size     = nstation * nchan * npol          * sizeof(ComplexDouble);
    size_t J_size     = nstation * nchan * npol * npol   * sizeof(ComplexDouble);

    // Arrays to be passed to the GPU kernel
    // (We don't need to allocate host memory for data, coh, or incoh -- we
    // assume this is allocated before these pointers were passed into this
    // function)
    ComplexDouble *W, *d_W;
    ComplexDouble *J, *d_J;
    ComplexDouble *Bd, *d_Bd;
    uint8_t *d_data;
    float   *d_coh;
    float   *d_incoh;

    // Allocate host memory
    W  = (ComplexDouble *)malloc( W_size );
    J  = (ComplexDouble *)malloc( J_size );
    Bd = (ComplexDouble *)malloc( Bd_size );


    // Allocate device memory
    gpuErrchk(hipMalloc( (void **)&d_W,     W_size ));
    gpuErrchk(hipMalloc( (void **)&d_J,     J_size ));
    gpuErrchk(hipMalloc( (void **)&d_Bd,    Bd_size ));
    gpuErrchk(hipMalloc( (void **)&d_data,  data_size ));
    gpuErrchk(hipMalloc( (void **)&d_coh,   coh_size ));
    gpuErrchk(hipMalloc( (void **)&d_incoh, incoh_size ));

    // Setup input values (= populate W and J)
    int s, ant, ch, pol, pol2;
    int Wi, Ji;
    for (ant = 0; ant < nstation; ant++)
    for (ch  = 0; ch  < nchan   ; ch++ )
    for (pol = 0; pol < npol    ; pol++)
    {
        Wi = ant * (npol*nchan) +
             ch  * (npol) +
             pol;
        W[Wi] = complex_weights_array[ant][ch][pol];

        for (pol2 = 0; pol2 < npol; pol2++)
        {
            Ji = Wi*npol + pol2;
            J[Ji] = invJi[ant][ch][pol][pol2];
        }
    }

    // Copy the data to the device
    gpuErrchk(hipMemcpy( d_data, data, data_size, hipMemcpyHostToDevice ));
    gpuErrchk(hipMemcpy( d_W,    W,    W_size,    hipMemcpyHostToDevice ));
    gpuErrchk(hipMemcpy( d_J,    J,    J_size,    hipMemcpyHostToDevice ));

    // Call the kernel
    beamform_kernel<<<opts->sample_rate, nchan>>>(
            d_data, d_W, d_J, invw, d_Bd, d_coh, d_incoh );
    hipDeviceSynchronize();

    // Copy the results back into host memory
    gpuErrchk(hipMemcpy( coh,   d_coh,   coh_size,   hipMemcpyDeviceToHost ));
    gpuErrchk(hipMemcpy( incoh, d_incoh, incoh_size, hipMemcpyDeviceToHost ));
    gpuErrchk(hipMemcpy( Bd,    d_Bd,    Bd_size,    hipMemcpyDeviceToHost ));

    // Copy the data back from Bd back into the detected_beam array
    // Make sure we put it back into the correct half of the array, depending
    // on whether this is an even or odd second.
    int offset, i;
    if (file_no % 2 == 0)
        offset = 0;
    else
        offset = opts->sample_rate;

    for (s   = 0; s   < opts->sample_rate; s++  )
    for (ch  = 0; ch  < nchan            ; ch++ )
    for (pol = 0; pol < npol             ; pol++)
    {
        i = s  * (npol*nchan) +
            ch * (npol)       +
            pol;

        detected_beam[s+offset][ch][pol] = Bd[i];
    }

    // Free memory on host and device
    free( W );
    free( J );
    free( Bd );
    hipFree( d_W );
    hipFree( d_J );
    hipFree( d_Bd );
    hipFree( d_data );
    hipFree( d_coh );
    hipFree( d_incoh );
}

