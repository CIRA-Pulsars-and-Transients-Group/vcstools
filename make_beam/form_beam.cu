#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <hip/hip_runtime.h>

extern "C" {
#include "beam_common.h"
#include "form_beam.h"
#include "mycomplex.h"
}

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    /* Wrapper function for GPU/CUDA error handling. Every CUDA call goes through
       this function. It will return a message giving your the error string,
       file name and line of the error. Aborts on error. */

    if (code != 0)
    {
        fprintf(stderr, "GPUAssert:: %s - %s (%d)\n", hipGetErrorString(code), file, line);
        if (abort)
        {
            exit(code);
        }
    }
}

// define a macro for accessing gpuAssert
#define gpuErrchk(ans) {gpuAssert((ans), __FILE__, __LINE__, true);}


// define constants to be used in the kernel
#define NSTATION  128
#define NPOL      2
#define NSTOKES   4



__global__ void beamform_kernel( uint8_t *data,
                                 ComplexDouble *W,
                                 ComplexDouble *J,
                                 double invw,
                                 ComplexDouble *Bd,
                                 float *C,
                                 float *I )
/* Layout for input arrays:
 *   data [nsamples] [nchan] [NPFB] [NREC] [NINC] -- see docs
 *   W    [NSTATION] [nchan] [NPOL]               -- weights array
 *   J    [NSTATION] [nchan] [NPOL] [NPOL]        -- jones matrix
 * Layout for output arrays:
 *   Bd   [nsamples] [nchan]   [NPOL]             -- detected beam
 *   C    [nsamples] [NSTOKES] [nchan]            -- coherent full stokes
 *   I    [nsamples] [nchan]                      -- incoherent
 */
{
    // Translate GPU block/thread numbers into meaningful names
    int s   = blockIdx.x;  /* The (s)ample number */
    int c   = blockIdx.y;  /* The (c)hannel number */
    int nc  = gridDim.y;   /* The (n)umber of (c)hannels (=128) */
    int ant = threadIdx.x; /* The (ant)enna number */

    // Calculate the beam and the noise floor
    __shared__ double Ia[NSTATION];
    __shared__ ComplexDouble Bx[NSTATION], By[NSTATION];
    ComplexDouble Dx, Dy;
    ComplexDouble WDx, WDy;

    __shared__ ComplexDouble Nxx[NSTATION], Nxy[NSTATION],
                             Nyx[NSTATION], Nyy[NSTATION];


    /* Fix from Maceij regarding NaNs in output when running on Athena, 13 April 2018.
       Apparently the different compilers and architectures are treating what were 
       unintialised variables very differently */
    Bx[ant]  = CMaked( 0.0, 0.0 );
    By[ant]  = CMaked( 0.0, 0.0 );

    Dx  = CMaked( 0.0, 0.0 );
    Dy  = CMaked( 0.0, 0.0 );

    WDx = CMaked( 0.0, 0.0 );
    WDy = CMaked( 0.0, 0.0 );

    Nxx[ant] = CMaked( 0.0, 0.0 );
    Nxy[ant] = CMaked( 0.0, 0.0 );
    Nyx[ant] = CMaked( 0.0, 0.0 );
    Nyy[ant] = CMaked( 0.0, 0.0 );

    Ia[ant] = 0.0;

    // Calculate beamform products for each antenna, and then add them together
    // Calculate the coherent beam (B = J*W*D)
    Dx  = UCMPLX4_TO_CMPLX_FLT(data[D_IDX(s,c,ant,0,nc)]);
    Dy  = UCMPLX4_TO_CMPLX_FLT(data[D_IDX(s,c,ant,1,nc)]);

    Ia[ant] = DETECT(Dx) + DETECT(Dy);

    if (CReald(W[W_IDX(c,ant,0,nc)]) == 0.0 &&
        CImagd(W[W_IDX(c,ant,0,nc)]) == 0.0 &&
        CReald(W[W_IDX(c,ant,1,nc)]) == 0.0 &&
        CImagd(W[W_IDX(c,ant,1,nc)]) == 0.0)
    {
        Ia[ant] = 0.0;
    }

    WDx = CMuld( W[W_IDX(c,ant,0,nc)], Dx );
    WDy = CMuld( W[W_IDX(c,ant,1,nc)], Dy );

    Bx[ant] = CAddd( CMuld( J[J_IDX(c,ant,0,0,nc)], WDx ),
                     CMuld( J[J_IDX(c,ant,1,0,nc)], WDy ) );
    By[ant] = CAddd( CMuld( J[J_IDX(c,ant,0,1,nc)], WDx ),
                     CMuld( J[J_IDX(c,ant,1,1,nc)], WDy ) );

    Nxx[ant] = CMuld( Bx[ant], CConjd(Bx[ant]) );
    Nxy[ant] = CMuld( Bx[ant], CConjd(By[ant]) );
    Nyx[ant] = CMuld( By[ant], CConjd(Bx[ant]) );
    Nyy[ant] = CMuld( By[ant], CConjd(By[ant]) );

    // Detect the coherent beam
    __syncthreads();
    if (ant < 64)
    {
        Ia[ant] += Ia[ant+64];
        Bx[ant] = CAddd( Bx[ant], Bx[ant+64] );
        By[ant] = CAddd( By[ant], By[ant+64] );
        Nxx[ant] = CAddd( Nxx[ant], Nxx[ant+64] );
        Nxy[ant] = CAddd( Nxy[ant], Nxy[ant+64] );
        Nyx[ant] = CAddd( Nyx[ant], Nyx[ant+64] );
        Nyy[ant] = CAddd( Nyy[ant], Nyy[ant+64] );
    }
    __syncthreads();
    if (ant < 32)
    {
        Ia[ant] += Ia[ant+32];
        Bx[ant] = CAddd( Bx[ant], Bx[ant+32] );
        By[ant] = CAddd( By[ant], By[ant+32] );
        Nxx[ant] = CAddd( Nxx[ant], Nxx[ant+32] );
        Nxy[ant] = CAddd( Nxy[ant], Nxy[ant+32] );
        Nyx[ant] = CAddd( Nyx[ant], Nyx[ant+32] );
        Nyy[ant] = CAddd( Nyy[ant], Nyy[ant+32] );
    }
    if (ant < 16)
    {
        Ia[ant] += Ia[ant+16];
        Bx[ant] = CAddd( Bx[ant], Bx[ant+16] );
        By[ant] = CAddd( By[ant], By[ant+16] );
        Nxx[ant] = CAddd( Nxx[ant], Nxx[ant+16] );
        Nxy[ant] = CAddd( Nxy[ant], Nxy[ant+16] );
        Nyx[ant] = CAddd( Nyx[ant], Nyx[ant+16] );
        Nyy[ant] = CAddd( Nyy[ant], Nyy[ant+16] );
    }
    if (ant < 8)
    {
        Ia[ant] += Ia[ant+8];
        Bx[ant] = CAddd( Bx[ant], Bx[ant+8] );
        By[ant] = CAddd( By[ant], By[ant+8] );
        Nxx[ant] = CAddd( Nxx[ant], Nxx[ant+8] );
        Nxy[ant] = CAddd( Nxy[ant], Nxy[ant+8] );
        Nyx[ant] = CAddd( Nyx[ant], Nyx[ant+8] );
        Nyy[ant] = CAddd( Nyy[ant], Nyy[ant+8] );
    }
    if (ant < 4)
    {
        Ia[ant] += Ia[ant+4];
        Bx[ant] = CAddd( Bx[ant], Bx[ant+4] );
        By[ant] = CAddd( By[ant], By[ant+4] );
        Nxx[ant] = CAddd( Nxx[ant], Nxx[ant+4] );
        Nxy[ant] = CAddd( Nxy[ant], Nxy[ant+4] );
        Nyx[ant] = CAddd( Nyx[ant], Nyx[ant+4] );
        Nyy[ant] = CAddd( Nyy[ant], Nyy[ant+4] );
    }
    if (ant < 2)
    {
        Ia[ant] += Ia[ant+2];
        Bx[ant] = CAddd( Bx[ant], Bx[ant+2] );
        By[ant] = CAddd( By[ant], By[ant+2] );
        Nxx[ant] = CAddd( Nxx[ant], Nxx[ant+2] );
        Nxy[ant] = CAddd( Nxy[ant], Nxy[ant+2] );
        Nyx[ant] = CAddd( Nyx[ant], Nyx[ant+2] );
        Nyy[ant] = CAddd( Nyy[ant], Nyy[ant+2] );
    }
    if (ant < 1)
    {
        Ia[ant] += Ia[ant+1];
        Bx[ant] = CAddd( Bx[ant], Bx[ant+1] );
        By[ant] = CAddd( By[ant], By[ant+1] );
        Nxx[ant] = CAddd( Nxx[ant], Nxx[ant+1] );
        Nxy[ant] = CAddd( Nxy[ant], Nxy[ant+1] );
        Nyx[ant] = CAddd( Nyx[ant], Nyx[ant+1] );
        Nyy[ant] = CAddd( Nyy[ant], Nyy[ant+1] );
    }
    __syncthreads();

    // Form the stokes parameters for the coherent beam
    if (ant == 0)
    {
        float bnXX = DETECT(Bx[0]) - CReald(Nxx[0]);
        float bnYY = DETECT(By[0]) - CReald(Nyy[0]);
        ComplexDouble bnXY = CSubd(
                                 CMuld( Bx[0], CConjd( By[0] ) ),
                                 Nxy[0] );

        // The incoherent beam
        I[I_IDX(s,c,nc)] = Ia[0];

        // Stokes I, Q, U, V:
        C[C_IDX(s,c,0,nc)] = invw*(bnXX + bnYY);
        C[C_IDX(s,c,1,nc)] = invw*(bnXX - bnYY);
        C[C_IDX(s,c,2,nc)] =  2.0*invw*CReald( bnXY );
        C[C_IDX(s,c,3,nc)] = -2.0*invw*CImagd( bnXY );

        // The beamformed products
        Bd[B_IDX(s,c,0,nc)] = Bx[0];
        Bd[B_IDX(s,c,1,nc)] = By[0];
    }
}

__global__ void flatten_bandpass_I_kernel(float *I,
                                     int nstep)/* uint8_t *Iout ) */
{
    // For just doing stokes I
    // One block
    // 128 threads each thread will do one channel
    // (we have already summed over all ant)

    // For doing the C array (I,Q,U,V)
    // ... figure it out later.

    // Translate GPU block/thread numbers into meaningful names
    int chan = threadIdx.x; /* The (c)hannel number */
    int nchan = blockDim.x; /* The total number of channels */
    float band;

    int new_var = 32; /* magic number */
    int i;

    float *data_ptr;

    // initialise the band 'array'
    band = 0.0;

    // accumulate abs(data) over all time samples and save into band
    data_ptr = I + I_IDX(0, chan, nchan);
    for (i=0;i<nstep;i++) { // time steps
        band += fabsf(*data_ptr);
        data_ptr = I + I_IDX(i,chan,nchan);
    }

    // now normalise the incoherent beam
    data_ptr = I + I_IDX(0, chan, nchan);
    for (i=0;i<nstep;i++) { // time steps
        *data_ptr = (*data_ptr)/( (band/nstep)/new_var );
        data_ptr = I + I_IDX(i,chan,nchan);
    }

}


__global__ void flatten_bandpass_C_kernel(float *C,
                                          int nstep)/* uint8_t *Iout ) */
{
    // For just doing stokes I
    // One block
    // 128 threads each thread will do one channel
    // (we have already summed over all ant)

    // For doing the C array (I,Q,U,V)
    // ... figure it out later.

    // Translate GPU block/thread numbers into meaningful names
    int chan = threadIdx.x; /* The (c)hannel number */
    int nchan = blockDim.x; /* The total number of channels */
    int stokes = threadIdx.y;
//    int nstokes = blockDim.y;

    float band;

    int new_var = 32; /* magic number */
    int i;

    float *data_ptr;

    // initialise the band 'array'
    band = 0.0;

    // accumulate abs(data) over all time samples and save into band
    //data_ptr = C + C_IDX(0,chan,stokes,nchan);
    for (i=0;i<nstep;i++) { // time steps
        data_ptr = C + C_IDX(i,chan,stokes,nchan);
        band += fabsf(*data_ptr);
    }

    // now normalise the coherent beam
    //data_ptr = C + C_IDX(0,chan,stokes,nchan);
    for (i=0;i<nstep;i++) { // time steps
        data_ptr = C + C_IDX(i,chan,stokes,nchan);
        *data_ptr = (*data_ptr)/( (band/nstep)/new_var );
    }

}



void cu_form_beam( uint8_t *data, struct make_beam_opts *opts,
                   ComplexDouble ***complex_weights_array,
                   ComplexDouble ****invJi, int file_no, int nstation, int nchan,
                   int npol, int outpol_coh, double invw,
                   struct gpu_formbeam_arrays *g,
                   ComplexDouble ***detected_beam, float *coh, float *incoh )
/* The CPU version of the beamforming operations, using OpenMP for
 * parallelisation.
 *
 * Inputs:
 *   data    = array of 4bit+4bit complex numbers. For data order, refer to the
 *             documentation.
 *   opts    = passed option parameters, containing meta information about the
 *             obs and the data
 *   W       = complex weights array. [nstation][nchan][npol]
 *   J       = inverse Jones matrix. [nstation][nchan][npol][npol]
 *   file_no = number of file we are processing, starting at 0.
 *   nstation     = 128
 *   nchan        = 128
 *   npol         = 2 (X,Y)
 *   outpol_coh   = 4 (I,Q,U,V)
 *   invw         = the reciprocal of the sum of the antenna weights
 *   g            = struct containing pointers to various arrays on
 *                  both host and device
 *
 * Outputs:
 *   detected_beam = result of beamforming operation, summed over antennas
 *                   [2*nsamples][nchan][npol]
 *   coh           = result in Stokes parameters (minus noise floor)
 *                   [nsamples][nstokes][nchan]
 *   incoh         = result (just Stokes I)
 *                   [nsamples][nchan]
 *
 * Assumes "coh" and "incoh" contain only zeros.
 */
{
    // Setup input values (= populate W and J)
    int s, ant, ch, pol, pol2;
    int Wi, Ji;
    for (ant = 0; ant < nstation; ant++)
    for (ch  = 0; ch  < nchan   ; ch++ )
    for (pol = 0; pol < npol    ; pol++)
    {
        Wi = ant * (npol*nchan) +
             ch  * (npol) +
             pol;
        g->W[Wi] = complex_weights_array[ant][ch][pol];

        for (pol2 = 0; pol2 < npol; pol2++)
        {
            Ji = Wi*npol + pol2;
            g->J[Ji] = invJi[ant][ch][pol][pol2];
        }
    }

    // Copy the data to the device
    gpuErrchk(hipMemcpy( g->d_data, data, g->data_size, hipMemcpyHostToDevice ));
    gpuErrchk(hipMemcpy( g->d_W,    g->W, g->W_size,    hipMemcpyHostToDevice ));
    gpuErrchk(hipMemcpy( g->d_J,    g->J, g->J_size,    hipMemcpyHostToDevice ));

    // Call the kernels
    dim3 samples_chan(opts->sample_rate, nchan);
    beamform_kernel<<<samples_chan, NSTATION>>>(
            g->d_data, g->d_W, g->d_J, invw, g->d_Bd, g->d_coh, g->d_incoh );
    //hipDeviceSynchronize();
    // sync not required between kernel queues since each stream acts like a FIFO queue
    // so all instances of the above kernel will complete before we move to the next
    // we are using the "default" stream since we don't specify any stream id

    // 1 block per pointing direction, hence the 1 for now
    flatten_bandpass_I_kernel<<<1, nchan>>>(g->d_incoh, opts->sample_rate);
    //hipDeviceSynchronize();

    // now do the same for the coherent beam
    dim3 chan_stokes(nchan, outpol_coh);
    flatten_bandpass_C_kernel<<<1, chan_stokes>>>(g->d_coh, opts->sample_rate);

    //hipDeviceSynchronize(); // Memcpy acts as a synchronize step so don't sync here
    // Copy the results back into host memory
    gpuErrchk(hipMemcpy( g->Bd, g->d_Bd,    g->Bd_size,    hipMemcpyDeviceToHost ));
    gpuErrchk(hipMemcpy( incoh, g->d_incoh, g->incoh_size, hipMemcpyDeviceToHost ));
    gpuErrchk(hipMemcpy( coh,   g->d_coh,   g->coh_size,   hipMemcpyDeviceToHost ));

    // Copy the data back from Bd back into the detected_beam array
    // Make sure we put it back into the correct half of the array, depending
    // on whether this is an even or odd second.
    int offset, i;
    if (file_no % 2 == 0)
        offset = 0;
    else
        offset = opts->sample_rate;

    for (s   = 0; s   < opts->sample_rate; s++  )
    for (ch  = 0; ch  < nchan            ; ch++ )
    for (pol = 0; pol < npol             ; pol++)
    {
        i = s  * (npol*nchan) +
            ch * (npol)       +
            pol;

        detected_beam[s+offset][ch][pol] = g->Bd[i];
    }

}

void malloc_formbeam( struct gpu_formbeam_arrays *g, unsigned int sample_rate,
        int nstation, int nchan, int npol, int outpol_coh, int outpol_incoh )
{
    // Calculate array sizes for host and device
    g->coh_size   = sample_rate * outpol_coh   * nchan * sizeof(float);
    g->incoh_size = sample_rate * outpol_incoh * nchan * sizeof(float);
    g->data_size  = sample_rate * nstation * nchan * npol * sizeof(uint8_t);
    g->Bd_size    = sample_rate * nchan * npol * sizeof(ComplexDouble);
    g->W_size     = nstation * nchan * npol * sizeof(ComplexDouble);
    g->J_size     = nstation * nchan * npol * npol * sizeof(ComplexDouble);

    // Allocate host memory
    g->W  = (ComplexDouble *)malloc( g->W_size );
    g->J  = (ComplexDouble *)malloc( g->J_size );
    g->Bd = (ComplexDouble *)malloc( g->Bd_size );


    // Allocate device memory
    gpuErrchk(hipMalloc( (void **)&g->d_W,     g->W_size ));
    gpuErrchk(hipMalloc( (void **)&g->d_J,     g->J_size ));
    gpuErrchk(hipMalloc( (void **)&g->d_Bd,    g->Bd_size ));
    gpuErrchk(hipMalloc( (void **)&g->d_data,  g->data_size ));
    gpuErrchk(hipMalloc( (void **)&g->d_coh,   g->coh_size ));
    gpuErrchk(hipMalloc( (void **)&g->d_incoh, g->incoh_size ));
}

void free_formbeam( struct gpu_formbeam_arrays *g )
{
    // Free memory on host and device
    free( g->W );
    free( g->J );
    free( g->Bd );
    hipFree( g->d_W );
    hipFree( g->d_J );
    hipFree( g->d_Bd );
    hipFree( g->d_data );
    hipFree( g->d_coh );
    hipFree( g->d_incoh );
}
