#include "hip/hip_runtime.h"
/********************************************************
 *                                                      *
 * Licensed under the Academic Free License version 3.0 *
 *                                                      *
 ********************************************************/

#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>

extern "C" {
#include "beam_common.h"
#include "form_beam.h"
#include "mycomplex.h"
}


#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    /* Wrapper function for GPU/CUDA error handling. Every CUDA call goes through
      this function. It will return a message giving your the error string,
      file name and line of the error. Aborts on error. */

    if (code != 0)
    {
        fprintf(stderr, "GPUAssert:: %s - %s (%d)\n", hipGetErrorString(code), file, line);
        if (abort)
        {
            exit(code);
        }
    }
}

// define a macro for accessing gpuAssert
#define gpuErrchk(ans) {gpuAssert((ans), __FILE__, __LINE__, true);}


// define constants to be used in the kernel
#define NSTATION  128
#define NPOL      2
#define NSTOKES   4
// maximum number of pointings (currently)
#define NPOINTING 4

__global__ void invj_the_data( uint8_t       *data,
                               ComplexDouble *J,
                               ComplexDouble *W,
                               ComplexDouble *JDx,
                               ComplexDouble *JDy,
                               float         *Ia,
                               int            incoh )
/* Layout for input arrays:
*   data [nsamples] [nchan] [NPFB] [NREC] [NINC] -- see docs
*   J    [NSTATION] [nchan] [NPOL] [NPOL]        -- jones matrix
*   incoh --true if outputing an incoherent beam
* Layout for output arrays:
*   JDx  [nsamples] [nchan] [NPFB] [NREC] [NINC]
*   JDy  [nsamples] [nchan] [NPFB] [NREC] [NINC]
*/
{
    // Translate GPU block/thread numbers into meaning->l names
    int c    = blockIdx.x;  /* The (c)hannel number */
    int nc   = gridDim.x;   /* The (n)umber of (c)hannels (=128) */
    int s    = blockIdx.y;  /* The (s)ample number */

    int ant  = threadIdx.x; /* The (ant)enna number */

    ComplexDouble Dx, Dy;
    // Convert input data to complex float
    Dx  = UCMPLX4_TO_CMPLX_FLT(data[D_IDX(s,c,ant,0,nc)]);
    Dy  = UCMPLX4_TO_CMPLX_FLT(data[D_IDX(s,c,ant,1,nc)]);

    // If tile is flagged in the calibration, flag it in the incoherent beam
    if (incoh)
    {
        if (CReald(W[W_IDX(0,ant,c,0,nc)]) == 0.0 &&
            CImagd(W[W_IDX(0,ant,c,0,nc)]) == 0.0 &&
            CReald(W[W_IDX(0,ant,c,1,nc)]) == 0.0 &&
            CImagd(W[W_IDX(0,ant,c,1,nc)]) == 0.0)
            Ia[JD_IDX(s,c,ant,nc)] = 0.0;
        else
            Ia[JD_IDX(s,c,ant,nc)] = DETECT(Dx) + DETECT(Dy);
    }

    // Calculate the first step (J*D) of the coherent beam (B = J*W*D)
    // Nick: by my math the order should be:
    // JDx = Jxx*Dx + Jxy*Dy
    // JDy = Jyx*Dx + Jyy*Dy
    // But switching yx and xy is the way it was done previously and appears
    // to give higher signal to noise
    JDx[JD_IDX(s,c,ant,nc)] = CAddd( CMuld( J[J_IDX(ant,c,0,0,nc)], Dx ),
                                     CMuld( J[J_IDX(ant,c,1,0,nc)], Dy ) );
    JDy[JD_IDX(s,c,ant,nc)] = CAddd( CMuld( J[J_IDX(ant,c,0,1,nc)], Dx ),
                                     CMuld( J[J_IDX(ant,c,1,1,nc)], Dy ) );


}

__global__ void beamform_kernel( ComplexDouble *JDx,
                                 ComplexDouble *JDy,
                                 ComplexDouble *W,
                                 float *Iin,
                                 double invw,
                                 int p,
                                 int coh_pol,
                                 int incoh,
                                 int soffset,
                                 int nchunk,
                                 ComplexDouble *Bd,
                                 float *C,
                                 float *I )
/* Layout for input arrays:
*   JDx  [nsamples] [nchan] [NPFB] [NREC] [NINC] -- calibrated voltages
*   JDy  [nsamples] [nchan] [NPFB] [NREC] [NINC]
*   W    [NSTATION] [nchan] [NPOL]               -- weights array
*   Iin  [nsamples] [nchan] [nant]               -- detected incoh
*   invw                                         -- inverse atrix
* Layout of input options
*   p                                            -- pointing number
*   coh_pol                                      -- coherent polorisation number
*   incoh                                        -- true if outputing an incoherent beam
*   soffset                                      -- sample offset (10000/nchunk)
*   nchunk                                       -- number of chunks each second is split into
* Layout for output arrays:
*   Bd   [nsamples] [nchan]   [NPOL]             -- detected beam
*   C    [nsamples] [NSTOKES] [nchan]            -- coherent full stokes
*   I    [nsamples] [nchan]                      -- incoherent
*/
{
    // Translate GPU block/thread numbers into meaning->l names
    int c    = blockIdx.x;  /* The (c)hannel number */
    int nc   = gridDim.x;   /* The (n)umber of (c)hannels (=128) */
    int s    = blockIdx.y;  /* The (s)ample number */
    int ns   = gridDim.y*nchunk;   /* The (n)umber of (s)amples (=10000)*/

    int ant  = threadIdx.x; /* The (ant)enna number */
    int nant = blockDim.x;  /* The (n)_umber of (ant)ennas */

    /*// GPU profiling
    clock_t start, stop;
    double setup_t, detect_t, sum_t, stokes_t;
    if ((p == 0) && (ant == 0) && (c == 0) && (s == 0)) start = clock();*/

    // Calculate the beam and the noise floor
    __shared__ double Ia[NSTATION];
    __shared__ ComplexDouble Bx[NSTATION], By[NSTATION];

    __shared__ ComplexDouble Nxx[NSTATION], Nxy[NSTATION],
                            Nyy[NSTATION];//Nyx[NSTATION]


    /* Fix from Maceij regarding NaNs in output when running on Athena, 13 April 2018.
    Apparently the different compilers and architectures are treating what were
    unintialised variables very differently */

    Bx[ant]  = CMaked( 0.0, 0.0 );
    By[ant]  = CMaked( 0.0, 0.0 );

    Nxx[ant] = CMaked( 0.0, 0.0 );
    Nxy[ant] = CMaked( 0.0, 0.0 );
    //Nyx[ant] = CMaked( 0.0, 0.0 );
    Nyy[ant] = CMaked( 0.0, 0.0 );

    if ((p == 0) && (incoh)) Ia[ant] = Iin[JD_IDX(s,c,ant,nc)];

    /*if ((p == 0) && (ant == 0) && (c == 0) && (s == 0))
    {
        stop = clock();
        setup_t = (double)(stop - start) / CLOCKS_PER_SEC * NPOINTING * NANT;
        start = clock();
    }*/

    // Calculate beamform products for each antenna, and then add them together
    // Calculate the coherent beam (B = J*W*D)
    Bx[ant] = CMuld( W[W_IDX(p,ant,c,0,nc)], JDx[JD_IDX(s,c,ant,nc)] );
    By[ant] = CMuld( W[W_IDX(p,ant,c,1,nc)], JDy[JD_IDX(s,c,ant,nc)] );

    Nxx[ant] = CMuld( Bx[ant], CConjd(Bx[ant]) );
    Nxy[ant] = CMuld( Bx[ant], CConjd(By[ant]) );
    //Nyx[ant] = CMuld( By[ant], CConjd(Bx[ant]) );
    Nyy[ant] = CMuld( By[ant], CConjd(By[ant]) );

    /*if ((p == 0) && (ant == 0) && (c == 0) && (s == 0))
    {
        stop = clock();
        detect_t = (double)(stop - start) / CLOCKS_PER_SEC * NPOINTING * NANT;
        start = clock();
    }*/

    // Detect the coherent beam
    // A summation over an array is faster on a GPU if you add half on array
    // to its other half as than can be done in parallel. Then this is repeated
    // with half of the previous array until the array is down to 1.
    __syncthreads();
    for ( int h_ant = nant / 2; h_ant > 0; h_ant = h_ant / 2 )
    {
        if (ant < h_ant)
        {
            if ( (p == 0) && (incoh)) Ia[ant] += Ia[ant+h_ant];
            Bx[ant]  = CAddd( Bx[ant],  Bx[ant  + h_ant] );
            By[ant]  = CAddd( By[ant],  By[ant  + h_ant] );
            Nxx[ant] = CAddd( Nxx[ant], Nxx[ant + h_ant] );
            Nxy[ant] = CAddd( Nxy[ant], Nxy[ant + h_ant] );
            //Nyx[ant]=CAddd( Nyx[ant], Nyx[ant + h_ant] );
            Nyy[ant] = CAddd( Nyy[ant], Nyy[ant + h_ant] );
        }
        // below makes no difference so removed
        // else return;
        __syncthreads();
    }

    /*if ((p == 0) && (ant == 0) && (c == 0) && (s == 0))
    {
        stop = clock();
        sum_t = (double)(stop - start) / CLOCKS_PER_SEC * NPOINTING * NANT;
        start = clock();

    }*/

    // Form the stokes parameters for the coherent beam
    // Only doing it for ant 0 so that it only prints once
    if ( ant == 0 )
    {
        float bnXX = DETECT(Bx[0]) - CReald(Nxx[0]);
        float bnYY = DETECT(By[0]) - CReald(Nyy[0]);
        ComplexDouble bnXY = CSubd( CMuld( Bx[0], CConjd( By[0] ) ),
                                    Nxy[0] );

        // The incoherent beam
        if ( (p == 0) && (incoh)) I[I_IDX(s+soffset,c,nc)] = Ia[0];

        // Stokes I, Q, U, V:
        C[C_IDX(p,s+soffset,0,c,ns,coh_pol,nc)] = invw*(bnXX + bnYY);
        if ( coh_pol == 4 )
        {
            C[C_IDX(p,s+soffset,1,c,ns,coh_pol,nc)] = invw*(bnXX - bnYY);
            C[C_IDX(p,s+soffset,2,c,ns,coh_pol,nc)] =  2.0*invw*CReald( bnXY );
            C[C_IDX(p,s+soffset,3,c,ns,coh_pol,nc)] = -2.0*invw*CImagd( bnXY );
        }

        // The beamformed products
        Bd[B_IDX(p,s+soffset,c,0,ns,nc)] = Bx[0];
        Bd[B_IDX(p,s+soffset,c,1,ns,nc)] = By[0];
    }
    /*if ((p == 0) && (ant == 0) && (c == 0) && (s == 0))
    {
        stop = clock();
        stokes_t = (double)(stop - start) / CLOCKS_PER_SEC * NPOINTING * NANT;
        printf("Time:  setup: % f detect: %f    sum: %f     stokes: %f\n", setup_t, detect_t, sum_t, stokes_t);
    }*/

}

__global__ void flatten_bandpass_I_kernel( float *I,
                                           int nstep )
{
    // For just doing stokes I
    // One block
    // 128 threads each thread will do one channel
    // (we have already summed over all ant)

    // For doing the C array (I,Q,U,V)
    // ... figure it out later.

    // Translate GPU block/thread numbers into meaningful names
    int chan = threadIdx.x; /* The (c)hannel number */
    int nchan = blockDim.x; /* The total number of channels */
    float band;

    int new_var = 32; /* magic number */
    int i;

    float *data_ptr;

    // initialise the band 'array'
    band = 0.0;

    // accumulate abs(data) over all time samples and save into band
    data_ptr = I + I_IDX(0, chan, nchan);
    for (i=0;i<nstep;i++) { // time steps
        band += fabsf(*data_ptr);
        data_ptr = I + I_IDX(i,chan,nchan);
    }

    // now normalise the incoherent beam
    data_ptr = I + I_IDX(0, chan, nchan);
    for (i=0;i<nstep;i++) { // time steps
        *data_ptr = (*data_ptr)/( (band/nstep)/new_var );
        data_ptr = I + I_IDX(i,chan,nchan);
    }

}


__global__ void flatten_bandpass_C_kernel( float *C, int nstep )
{
    // For just doing stokes I
    // One block
    // 128 threads each thread will do one channel
    // (we have already summed over all ant)

    // For doing the C array (I,Q,U,V)
    // ... figure it out later.

    // Translate GPU block/thread numbers into meaningful names
    int chan    = threadIdx.x; /* The (c)hannel number */
    int nchan   = blockDim.x;  /* The (n)umber of (c)hannels */
    int stokes  = threadIdx.y; /* The (stokes) number */
    int nstokes = blockDim.y;  /* The (n)umber of (stokes) */

    int p      = blockIdx.x;  /* The (p)ointing number */

    float band;

    int new_var = 32; /* magic number */
    int i;

    float *data_ptr;

    // initialise the band 'array'
    band = 0.0;

    // accumulate abs(data) over all time samples and save into band
    //data_ptr = C + C_IDX(0,stokes,chan,nchan);
    for (i=0;i<nstep;i++) { // time steps
        data_ptr = C + C_IDX(p,i,stokes,chan,nstep,nstokes,nchan);
        band += fabsf(*data_ptr);
    }

    // now normalise the coherent beam
    //data_ptr = C + C_IDX(0,stokes,chan,nchan);
    for (i=0;i<nstep;i++) { // time steps
        data_ptr = C + C_IDX(p,i,stokes,chan,nstep,nstokes,nchan);
        *data_ptr = (*data_ptr)/( (band/nstep)/new_var );
    }

}



void cu_form_beam( uint8_t *data, struct make_beam_opts *opts,
                   ComplexDouble ****complex_weights_array,
                   ComplexDouble ****invJi, int file_no,
                   int npointing, int nstation, int nchan,
                   int npol, int outpol_coh, double invw,
                   struct gpu_formbeam_arrays *g,
                   ComplexDouble ****detected_beam, float *coh, float *incoh,
                   hipStream_t *streams, int incoh_check, int nchunk )
/* The CPU version of the beamforming operations, using OpenMP for
* parallelisation.
*
* Inputs:
*   data    = array of 4bit+4bit complex numbers. For data order, refer to the
*             documentation.
*   opts    = passed option parameters, containing meta information about the
*             obs and the data
*   W       = complex weights array. [npointing][nstation][nchan][npol]
*   J       = inverse Jones matrix.  [nstation][nchan][npol][npol]
*   file_no = number of file we are processing, starting at 0.
*   nstation     = 128
*   nchan        = 128
*   npol         = 2 (X,Y)
*   outpol_coh   = 4 (I,Q,U,V)
*   invw         = the reciprocal of the sum of the antenna weights
*   g            = struct containing pointers to various arrays on
*                  both host and device
*
* Outputs:
*   detected_beam = result of beamforming operation, summed over antennas
*                   [2*nsamples][nchan][npol]
*   coh           = result in Stokes parameters (minus noise floor)
*                   [nsamples][nstokes][nchan]
*   incoh         = result (just Stokes I)
*                   [nsamples][nchan]
*
* Assumes "coh" and "incoh" contain only zeros.
*/
{
    // Setup input values (= populate W and J)
    int p, ant, ch, pol, pol2;
    int Wi, Ji;
    for (p   = 0; p   < npointing; p++  )
    for (ant = 0; ant < nstation ; ant++)
    for (ch  = 0; ch  < nchan    ; ch++ )
    for (pol = 0; pol < npol     ; pol++)
    {
        Wi = p   * (npol*nchan*nstation) +
             ant * (npol*nchan) +
             ch  * (npol) +
             pol;
        g->W[Wi] = complex_weights_array[p][ant][ch][pol];

        if ( p == 0 )
        for (pol2 = 0; pol2 < npol; pol2++)
        {
            Ji = ant * (npol*npol*nchan) +
                 ch  * (npol*npol) +
                 pol * (npol) +
                 pol2;
            g->J[Ji] = invJi[ant][ch][pol][pol2];
        }
    }
    // Copy the data to the device
    gpuErrchk(hipMemcpyAsync( g->d_W,    g->W, g->W_size,    hipMemcpyHostToDevice ));
    gpuErrchk(hipMemcpyAsync( g->d_J,    g->J, g->J_size,    hipMemcpyHostToDevice ));

    // Divide the gpu calculation into multiple time chunks so there is enough room on the GPU
    for (int ichunk = 0; ichunk < nchunk; ichunk++)
    {
        //int dataoffset = ichunk * g->data_size / sizeof(uint8_t);
        gpuErrchk(hipMemcpyAsync( g->d_data,
                                   data + ichunk * g->data_size / sizeof(uint8_t),
                                   g->data_size, hipMemcpyHostToDevice ));

        // Call the kernels
        // samples_chan(index=blockIdx.x  size=gridDim.x,
        //              index=blockIdx.y  size=gridDim.y)
        // stat_point  (index=threadIdx.x size=blockDim.x,
        //              index=threadIdx.y size=blockDim.y)
        //dim3 samples_chan(opts->sample_rate, nchan);
        dim3 chan_samples( nchan, opts->sample_rate / nchunk );
        dim3 stat( NSTATION );

        // convert the data and multiply it by J
        invj_the_data<<<chan_samples, stat>>>( g->d_data, g->d_J, g->d_W, g->d_JDx, g->d_JDy,
                                               g->d_Ia, incoh_check );

        // Send off a parrellel cuda stream for each pointing
        for ( int p = 0; p < npointing; p++ )
        {
            beamform_kernel<<<chan_samples, stat, 0, streams[p]>>>( g->d_JDx, g->d_JDy,
                            g->d_W, g->d_Ia, invw,
                            p, outpol_coh , incoh_check, ichunk*opts->sample_rate/nchunk, nchunk,
                            g->d_Bd, g->d_coh, g->d_incoh );

            gpuErrchk( hipPeekAtLastError() );
        }
    }
    gpuErrchk( hipDeviceSynchronize() );


    // Flatten the bandpass
    if ( incoh_check )
    {
        flatten_bandpass_I_kernel<<<1, nchan, 0, streams[0]>>>( g->d_incoh,
                                                                opts->sample_rate );
        gpuErrchk( hipPeekAtLastError() );
    }
    for ( int p = 0; p < npointing; p++ )
    {
        // Now do the same for the coherent beam
        dim3 chan_stokes(nchan, outpol_coh);
        flatten_bandpass_C_kernel<<<npointing, chan_stokes, 0, streams[p]>>>( g->d_coh,
                                                                    opts->sample_rate );
        gpuErrchk( hipPeekAtLastError() );
    }
    gpuErrchk( hipDeviceSynchronize() );

    // Copy the results back into host memory
    gpuErrchk(hipMemcpyAsync( g->Bd, g->d_Bd,    g->Bd_size,    hipMemcpyDeviceToHost ));
    gpuErrchk(hipMemcpyAsync( incoh, g->d_incoh, g->incoh_size, hipMemcpyDeviceToHost ));
    gpuErrchk(hipMemcpyAsync( coh,   g->d_coh,   g->coh_size,   hipMemcpyDeviceToHost ));

    // Copy the data back from Bd back into the detected_beam array
    // Make sure we put it back into the correct half of the array, depending
    // on whether this is an even or odd second.
    int offset, i;
    offset = file_no % 2 * opts->sample_rate;

    for ( int p   = 0; p   < npointing        ; p++  )
    for ( int s   = 0; s   < opts->sample_rate; s++  )
    for ( int ch  = 0; ch  < nchan            ; ch++ )
    for ( int pol = 0; pol < npol             ; pol++)
    {
        i = p  * (npol*nchan*opts->sample_rate) +
            s  * (npol*nchan)                   +
            ch * (npol)                         +
            pol;

        detected_beam[p][s+offset][ch][pol] = g->Bd[i];
    }
}

void malloc_formbeam( struct gpu_formbeam_arrays *g, unsigned int sample_rate,
                      int nstation, int nchan, int npol, int nchunk, int outpol_coh,
                      int outpol_incoh, int npointing, double time )
{
    // Calculate array sizes for host and device
    g->coh_size   = npointing * sample_rate * outpol_coh * nchan * sizeof(float);
    g->incoh_size = sample_rate * outpol_incoh * nchan * sizeof(float);
    g->data_size  = sample_rate * nstation * nchan * npol / nchunk * sizeof(uint8_t);
    g->Bd_size    = npointing * sample_rate * nchan * npol * sizeof(ComplexDouble);
    g->W_size     = npointing * nstation * nchan * npol * sizeof(ComplexDouble);
    g->J_size     = nstation * nchan * npol * npol * sizeof(ComplexDouble);
    g->JD_size    = sample_rate * nstation * nchan / nchunk * sizeof(ComplexDouble);

    // Allocate host memory
    //g->W  = (ComplexDouble *)malloc( g->W_size );
    //g->J  = (ComplexDouble *)malloc( g->J_size );
    //g->Bd = (ComplexDouble *)malloc( g->Bd_size );
    hipHostMalloc( &g->W, g->W_size );
    cudaCheckErrors("hipHostMalloc W fail");
    hipHostMalloc( &g->J, g->J_size );
    cudaCheckErrors("hipHostMalloc J fail");
    hipHostMalloc( &g->Bd, g->Bd_size );
    cudaCheckErrors("hipHostMalloc Bd fail");

    fprintf( stderr, "[%f] coh_size   %d  MB GPU mem\n", time, g->coh_size  /1000000 );
    fprintf( stderr, "[%f] incoh_size %d  MB GPU mem\n", time, g->incoh_size/1000000 );
    fprintf( stderr, "[%f] data_size  %d  MB GPU mem\n", time, g->data_size /1000000 );
    fprintf( stderr, "[%f] Bd_size    %d  MB GPU mem\n", time, g->Bd_size   /1000000 );
    fprintf( stderr, "[%f] W_size     %d  MB GPU mem\n", time, g->W_size    /1000000 );
    fprintf( stderr, "[%f] J_size     %d  MB GPU mem\n", time, g->J_size    /1000000 );
    fprintf( stderr, "[%f] JD_size    %d  MB GPU mem\n", time, g->JD_size*3 /1000000 );

    int GPU_mem = (g->W_size + g->J_size + g->Bd_size + g->data_size +
                   g->coh_size + g->incoh_size + 3*g->JD_size) /1000000000;

    fprintf( stderr, "[%f]  %d GB GPU memory allocated\n", time, GPU_mem );

    // Allocate device memory
    gpuErrchk(hipMalloc( (void **)&g->d_W,     g->W_size ));
    gpuErrchk(hipMalloc( (void **)&g->d_J,     g->J_size ));
    gpuErrchk(hipMalloc( (void **)&g->d_JDx,   g->JD_size ));
    gpuErrchk(hipMalloc( (void **)&g->d_JDy,   g->JD_size ));
    gpuErrchk(hipMalloc( (void **)&g->d_Ia,    g->JD_size ));
    gpuErrchk(hipMalloc( (void **)&g->d_Bd,    g->Bd_size ));
    gpuErrchk(hipMalloc( (void **)&g->d_data,  g->data_size ));
    gpuErrchk(hipMalloc( (void **)&g->d_coh,   g->coh_size ));
    gpuErrchk(hipMalloc( (void **)&g->d_incoh, g->incoh_size ));

}

void free_formbeam( struct gpu_formbeam_arrays *g )
{
    // Free memory on host and device
    hipHostFree( g->W );
    hipHostFree( g->J );
    hipHostFree( g->Bd );
    hipFree( g->d_W );
    hipFree( g->d_J );
    hipFree( g->d_Bd );
    hipFree( g->d_data );
    hipFree( g->d_coh );
    hipFree( g->d_incoh );
}

float *create_pinned_data_buffer_psrfits( size_t size )
{
    float *ptr;
    hipHostMalloc( &ptr, size * sizeof(float) );
    //hipError_t status = hipHostRegister((void**)&ptr, size * sizeof(float),
    //                                      hipHostRegisterPortable );
    cudaCheckErrors("hipHostMalloc data_buffer_psrfits fail");
    return ptr;
}

float *create_pinned_data_buffer_vdif( size_t size )
{
    float *ptr;
    hipHostMalloc( &ptr, size * sizeof(float) );
    //hipError_t status = hipHostRegister((void**)&ptr, size * sizeof(float),
    //                                      hipHostRegisterPortable );
    cudaCheckErrors("hipHostMalloc data_buffer_vdif fail");
    return ptr;
}

void populate_weights_johnes( struct gpu_formbeam_arrays *g,
                              ComplexDouble ****complex_weights_array,
                              ComplexDouble *****invJi,
                              int npointing, int nstation, int nchan, int npol )
{
    // Setup input values (= populate W and J)
    int p, ant, ch, pol, pol2;
    int Wi, Ji;
    for (p   = 0; p   < npointing; p++  )
    for (ant = 0; ant < nstation ; ant++)
    for (ch  = 0; ch  < nchan    ; ch++ )
    for (pol = 0; pol < npol     ; pol++)
    {
        Wi = p   * (npol*nchan*nstation) +
             ant * (npol*nchan) +
             ch  * (npol) +
             pol;
        g->W[Wi] = complex_weights_array[p][ant][ch][pol];

        for (pol2 = 0; pol2 < npol; pol2++)
        {
            Ji = Wi*npol + pol2;
            g->J[Ji] = invJi[p][ant][ch][pol][pol2];
        }
    }
    // Copy the data to the device
    gpuErrchk(hipMemcpy( g->d_W, g->W, g->W_size, hipMemcpyHostToDevice ));
    gpuErrchk(hipMemcpy( g->d_J, g->J, g->J_size, hipMemcpyHostToDevice ));
}

