#include "hip/hip_runtime.h"
#include "utils.h"

void getDeviceDimensions(int *nDevices)
{
    /* We need to know how many devices are available and its functionality. */

    printf("Querying system for device information --\n");
    hipGetDeviceCount(nDevices); // get CUDA to count GPUs

    for (int i = 0; i < *nDevices; i++)
    {
        struct hipDeviceProp_t prop; // create struct to store device info
        hipGetDeviceProperties(&prop, i); // populate prop for this device
        printf("    Device number:                       %d\n", *nDevices-1);
        printf("    Device name:                         %s\n", prop.name);
        printf("    Total global memory available (MB):  %f\n", prop.totalGlobalMem/1e6);
        printf("    Max grid size (# blocks):           (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
        printf("    Max number of threads per block:    (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    }
}

void requiredMemory(int size, int ntiles, int *niter, int *blockSize)
{
    /* Estimate how much device/host memory will be required per iteration.

       The device kernal consists of:
        
       az array = size * sizeof(double) - array of azimuth coords
       za array = size * sizeof(double) - array of zenith coords
       xpos = ntiles * sizeof(float) - x position of tiles from array center
       ypos = ntiles * sizeof(float) - y position of tiles from array center
       zpos = ntiles * sizeof(float) - z position of tiles from array center
       af_array = size * sizeof(hipDoubleComplex) - array factor

       plus we also pass:
       2 integers
       1 double

       and compute in the kernal:
       2 integers
       6 doubles
       1 hipDoubleComplex */

    int iters = 1, nDevices = 0;
    double tfrac = 0.9;
    long double azzaMem = 0, tileposMem = 0, afMem = 0, otherMem = 0, reqMem = 0, devMem = 0;
    size_t freeMem, totMem;
    hipError_t res;
    struct hipDeviceProp_t prop; // create struct to store device info

    // get info about avilable devices
    getDeviceDimensions(&nDevices);
    printf("Number of devices on system: %d\n", nDevices);
    printf("Using device: %d\n",                nDevices-1);
    hipGetDeviceProperties(&prop, 0); 

    // check how much FREE memory is available
    res = hipMemGetInfo(&freeMem, &totMem);
    if (res == hipSuccess)
    {
        printf("Free device memory: %.2f MB\n", (double)freeMem/1.0e6);
    }
    else
    {
        printf("%s\n", hipGetErrorString(res));
    }
    
    // get device max. threads per block
    *blockSize = prop.maxThreadsDim[0];

    // define the array sizes that will go onto the device
    azzaMem    = 2 * (size/1.0e6)   * sizeof(double);          // az and za arrays
    tileposMem = 3 * (ntiles/1.0e6) * sizeof(float);           // x,y,z positions of all tiles
    afMem      =     (size/1.0e6)   * sizeof(hipDoubleComplex); // "array factor" array
    // misc. memory requirments (likely inconsequential)

    otherMem   = (7 * sizeof(double) + 
                  4 * sizeof(int) + 
                  sizeof(hipDoubleComplex) + 
                  sizeof(wavenums)) / 1.0e6;
    
    reqMem = azzaMem + tileposMem + afMem + otherMem; // total required memory in MB
    devMem = (double)freeMem/1.0e6; // available memory in MB

    printf("Memory required for:\n");
    printf("    Az,ZA arrays: %Lf MB\n",   azzaMem);
    printf("    tile positions: %Lf MB\n", tileposMem);
    printf("    array fator: %Lf MB\n",    afMem);
    printf("    intermediate: %Lf MB\n",   otherMem);
    printf("Total: %Lf MB\n",              reqMem);

    if (reqMem < 0)
    {
        fprintf(stderr, "Negative required memory (%Lf)!! Aborting.\n", reqMem);
        exit(1);
    }
    else if ((tfrac*devMem) <= reqMem)
    {
        fprintf(stderr, "Arrays will not fit on device!\n");
        fprintf(stderr, "   total device memory required = %Lf MB\n",  reqMem);
        fprintf(stderr, "   useable device memory        = %Lf MB\n",  tfrac*devMem);
        fprintf(stderr, "       (useable fraction = %.2f of total)\n", tfrac);

        iters = (int)ceil(tfrac*reqMem / devMem)+1;
        printf("Will split task into: %d iterations (approx. %.2Lf MB per iteration)\n", iters, (reqMem/iters));
    }
    printf("\n");

    *niter = iters;
}
