#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <math.h>
#include <slalib.h>
#include <fitsio.h>
//#include <complex.h>

// MWA tile beam
#include "FEE2016/beam2016implementation.h"
#include "FEE2016/mwa_beam_interface.h"
#include "FEE2016/system.h"
#include <H5Cpp.h>

// CUDA kernal
#include "pabeam_kernal.h"

#define PI (acos(-1.0))         // Ensures PI is defined on all systems
#define RAD2DEG (180.0 / PI)
#define DEG2RAD (PI / 180.0)
#define SOL (299792458.0)       // Speed of light
#define KB (1.38064852e-23)     // Boltzmann's constant

#define MWA_LAT (-26.703319)    // Array latitude, degrees North
#define MWA_LON (116.67081)     // Array longitude, degrees East
#define MWA_HGT (377.827)       // Array elevation above sea level, in meters

/* struct to hold all the wavenumbers for each (Az,ZA) */
typedef struct wavenums_t
{
    double kx;
    double ky;
    double kz;
} wavenums; // can just refer to this struct as type wavenums

/* struct to hold the target Azimuth and Zenith angle (in radians) */
typedef struct tazza_t
{
    double az;
    double za;
} tazza;


/* Define all the function prototypes */
void usage();

void utc2mjd(char *utc_str, double *intmjd, double *fracmjd);
void mjd2lst(double mjd, double *lst);
void calcWaveNumber(double lambda, double az, double za, wavenums *p_wn);
void calcTargetAZZA(char *ra_hhmmss, char *dec_ddmmss, char *time_utc, tazza *p_tazza);
int getNumTiles(const char *metafits);
void getTilePositions(const char *metafits, int ninput, 
                        float *n_pols, float *e_pols, float *h_pols,
                        float *n_tile, float *e_tile, float *h_tile);
int getFlaggedTiles(const char *badfile, int *badtiles);
void removeFlaggedTiles(float *n_tile, float *e_tile, float *h_tile, 
                        int *badtiles, int nbad, int nelements);
void getDeviceDimensions(int *nDevices);
void requiredMemory(int size, int ntiles, int *niter, int *blockSize);
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort);

// GPU kernal for calculation
//__global__ void calcArrayFactor(int nel, int ntiles, double a,
//                                double *za, double *az,
//                                float *xp, float *yp, float *zp,
//                                wavenums *p_twn,
//                                hipDoubleComplex *af);
//void calcArrayFactorCPU(int nel, int ntiles, double a,
//                        double *za, double *az,
//                        float *xp, float *yp, float *zp,
//                        wavenums *p_twn,
//                        double complex *af);


void usage()
{
    printf("pabeam_gpu --- computes the array factor that represents the naturally weighted synthesised MWA beam (tied-array/coherent beam) for a given configuration\n");
    printf("syntax:\n");
    printf("    pabeam -f <frequency in Hz> -r <ra in hh:mm:ss> -d <dec in dd:mm:ss> -t <UTC in ISOT format> -m <metafits file> -b <RTS flagged_tiles.txt file> [-e] [-x] [-y] [-g]\n\n");
    printf("Options:\n");
    printf("    -f observing frequency, in Hz\n");
    printf("    -r target RA (J2000), in hh:mm:ss.ss format\n");
    printf("    -d target DEC (J2000), in dd:mm:ss.ss format\n");
    printf("    -t UTC time to evaluate, in format YYYY-MM-DDThh:mm:ss\n");
    printf("    -m metafits file for the observation\n");
    printf("    -b RTS flagged_tiles.txt file from calibration\n");
    printf("    -e radiation efficiency (if unsure, use 1.0)\n");
    printf("    -x Azimuth grid resolution element (>= 0.01)\n");
    printf("    -y Zenith angle grid resolution element (>=0.01)\n");
    printf("    -g Calculate and apply the FEE2016 tile beam with the given \"gridpoint\" number\n");
}

void utc2mjd(char *utc_str, double *intmjd, double *fracmjd)
{
    /* Convert a UTC string (YYYY-MM-DDThh:mm:ss.ss) into MJD in radians.
     * Accepts a stc string and pointers to the integer and fractional MJD values. */
    int year, month, day, hour, min, sec, jflag;

    sscanf(utc_str,"%d-%d-%dT%d:%d:%d", &year, &month, &day, &hour, &min, &sec);
    //fprintf(stderr,"Parsed date : yr %d, month %d, day %d, hour %d, min %d, sec %f\n", year, month, day, hour, min, sec);

    slaCaldj(year, month, day, intmjd, &jflag);
    if (jflag != 0) 
    {
        fprintf(stderr,"Failed to calculate MJD\n");
    }
    *fracmjd = (hour + (min/60.0) + (sec/3600.0))/24.0;
}

void mjd2lst(double mjd, double *lst)
{
    /* Greenwich Mean Sidereal Time to LMST
     * east longitude in hours at the epoch of the MJD */
    double lmst;
    lmst = slaRanorm(slaGmst(mjd) + MWA_LON*DEG2RAD);
    *lst = lmst;
}

void calcWaveNumber(double lambda, double az, double za, wavenums *p_wn)
{
    /* Calculate the 3D wavenumbers for a given wavelength (lambda) from the direction (az,za).
     * Accepts wavelength (m), azimuth (rad) and zenith angle (rad) and a pointer to a wavenums struct to populate.*/
    double a, ast, phi;

    a = 2 * PI / lambda;
    ast = a * sin(za);
    phi = PI/2 - az;

    /* 
     * the standard equations are:
     *      a = 2 * pi / lambda
     *      kx = a * sin(theta) * cos(phi)
     *      ky = a * sin(theta) * sin(phi)
     *      kz = a * cos(theta)
     * this is assuming that the coordinates (theta,phi) are defined in 
     * the convention from Sutinjo et al. 2015, where
     *      theta = za
     *      phi = pi/2 - az
     * i.e. the azimuth is measured clockwise from East (standard for antenna theory, offset from astronomy)
     */

    p_wn->kx = ast * cos(phi); 
    p_wn->ky = ast * sin(phi); 
    p_wn->kz = a * cos(za);   
}

void calcTargetAZZA(char *ra_hhmmss, char *dec_ddmmss, char *time_utc, tazza *p_tazza)
{
    int ra_ih, ra_im, ra_j;
    int dec_id, dec_im, dec_j;
    int sign;
    double ra_rad, ra_fs, ha_rad;
    double dec_rad, dec_fs;
    double az, el;
    double mjd, intmjd, fracmjd, lmst;
    double pr=0, pd=0, px=0, rv=0, eq=2000, ra_ap=0, dec_ap=0; // all for conversion to apparent RA/DEC
    char id_str[20];

    // read ra string into hours, minutes and seconds
    sscanf(ra_hhmmss, "%d:%d:%lf", &ra_ih, &ra_im, &ra_fs);

    //read dec string into degrees, arcmin and arsec (extra steps for sign, '+' or '-')
    sscanf(dec_ddmmss, "%s:%d:%lf", id_str, &dec_im, &dec_fs);
    sign = (id_str[0] == '-' ? -1 : 1); // check sign of dec

    sscanf(dec_ddmmss, "%d:%d:%lf", &dec_id, &dec_im, &dec_fs); // assign values
    dec_id = dec_id * sign; // ensure correct sign


    // convert angles to radians
    slaCtf2r(ra_ih, ra_im, ra_fs, &ra_rad, &ra_j); //right ascension
    slaDaf2r(dec_id, dec_im, dec_fs, &dec_rad, &dec_j); //declination

    if (ra_j != 0) 
    {
        fprintf(stderr,"Error parsing %s as hhmmss\nslalib error code: j=%d\n", ra_hhmmss, ra_j);
        fprintf(stderr,"ih = %d, im = %d, fs = %f\n", ra_ih, ra_im, ra_fs);
        exit(-1);
    }
    if (dec_j != 0) 
    {
        fprintf(stderr,"Error parsing %s as ddmmss\nslalib error code: j=%d\n", dec_ddmmss, dec_j);
        fprintf(stderr,"ih = %d, im = %d, fs = %f\n", dec_id, dec_im, dec_fs);
        exit(-1);
    }

    // convert UTC to MJD
    utc2mjd(time_utc, &intmjd, &fracmjd);
    mjd = intmjd + fracmjd;
    mjd2lst(mjd, &lmst);

    // get apparent RA and Dec of target
    slaMap(ra_rad, dec_rad, pr, pd, px, rv, eq, mjd, &ra_ap, &dec_ap);
    printf("RA = %.4f  RA_app = %.4f  DEC = %.4f  DEC_app = %.4f\n", ra_rad, ra_ap, dec_rad, dec_ap);

    // use RA and LST to get HA
    ha_rad = slaRanorm(lmst - ra_ap);

    // convert (HA, Dec) to (az, el)
    slaDe2h(ha_rad, dec_rad, MWA_LAT*DEG2RAD, &az, &el);

    printf("Az = %.4f  ZA = %.4f\n", az, PI/2-el);
    p_tazza->az = az;
    p_tazza->za = (PI/2) - el;
}

int getNumTiles(const char *metafits)
{
    /* Figure out the number of tiles based on the information in the metafits.

       NOTE: we get warnings from this in compilation because the library functions
       expect char characters, but conversion from string literals to chars is bad.
       It works, but we get warnings... */

    fitsfile *fptr=NULL;
    int status=0;
    size_t ninput=0;

    fits_open_file(&fptr, metafits, READONLY, &status); // open metafits file
    fits_movnam_hdu(fptr, BINARY_TBL, "TILEDATA", 0, &status); // move to TILEDATA HDU
    if (status != 0)
    {
        fprintf(stderr,"Error: Failed to move to TILEDATA HDU\n");
        exit(-1);
    }

    fits_read_key(fptr, TINT, "NAXIS2", &ninput, NULL, &status); // read how many tiles are included
    if (status != 0)
    {
        fprintf(stderr,"Error: Failed to read size of binary table in TILEDATA\n");
        exit(-1);
    }
    fits_close_file(fptr, &status);
    return ninput;
}

void getTilePositions(const char *metafits, int ninput,\
        float *n_pols, float *e_pols, float *h_pols,\
        float *n_tile, float *e_tile, float *h_tile)
{
    /* Get the tile positions from the metafits file.
       Accepts the metafits file name, 
       number of items to read (i.e. 2x number of tiles, 1 per polarisation),
       the array to fill with the polarisation locations, and
       the array to fill with the tile locations (every second element of *_pols).
       
       NOTE: we get warnings from this in compilation because the library functions
       expect char characters, but conversion from string literals to chars is bad.
       It works, but we get warnings... */

    fitsfile *fptr=NULL;
    int status=0, anynull=0;
    int colnum=0;


    fits_open_file(&fptr, metafits, READONLY, &status); // open metafits file
    fits_movnam_hdu(fptr, BINARY_TBL, "TILEDATA", 0, &status); // move to TILEDATA HDU
    if (status != 0) 
    {
        fprintf(stderr,"Error: Failed to move to TILEDATA HDU\n");
        exit(-1);
    }

    fits_get_colnum(fptr, 1, "North", &colnum, &status); // get north coordinates of tiles
    fits_read_col_flt(fptr, colnum, 1, 1, ninput, 0.0, n_pols, &anynull, &status);
    if (status != 0)
    {
        fprintf(stderr,"Error: Failed to read  N coord in metafile\n");
        exit(-1);
    }

    fits_get_colnum(fptr, 1, "East", &colnum, &status); // get east coordinates of tiles
    fits_read_col_flt(fptr, colnum, 1, 1, ninput, 0.0, e_pols, &anynull, &status);
    if (status != 0)
    {
        fprintf(stderr,"Error: Failed to read E coord in metafile\n");
        exit(-1);
    }

    fits_get_colnum(fptr, 1, "Height", &colnum, &status); // get height a.s.l. of tiles
    fits_read_col_flt(fptr, colnum, 1, 1, ninput, 0.0, h_pols, &anynull, &status);
    if (status != 0)
    {
        fprintf(stderr,"Error: Failed to read H coord in metafile\n");
        exit(-1);
    }
    fits_close_file(fptr, &status);

    // populate the tile arrays with every second element of the pol arrays
    for (int i = 0; i < ninput; i+=2)
    {
        n_tile[i/2] = n_pols[i];
        e_tile[i/2] = e_pols[i];
        h_tile[i/2] = h_pols[i];
    }

    // convert heights into height above array center
    for (int i = 0; i < ninput/2; i++)
    {
        h_tile[i] = h_tile[i] - MWA_HGT;
    }
}

int getFlaggedTiles(const char *badfile, int *badtiles)
{
    /* Open the flagged tiles file, read into an array and count how many lines are read.
       Update the array pointer and return number of elements to read from that array
       (as it's initialised to be able to hold every tile) */
    
    FILE *fp;
    int tile=0, i=0;
    int nlines=0;

    fp = fopen(badfile,"r");
    if (fp == NULL)
    {
        fprintf(stderr,"Error opening flagged tiles file.\n");
        exit(-1);
    }

    while(fscanf(fp, "%d\n", &tile) > 0)
    {
        printf("    bad tile: %d\n",tile);
        badtiles[i++] = tile;
        nlines++;
    }

    fclose(fp);
    return nlines;
}

void removeFlaggedTiles(float *n_tile, float *e_tile, float *h_tile,\
        int *badtiles, int nbad, int nelements)
{
    /* Get rid of the bad/flagged tiles from the array. Basically just
       shift all the indexes around so that whenever a bad tile is there,
       it's data is over-written. We end up with an array of the same size,
       but the last nbad elements are all identical (and can be ignored). */

    int counter=0,bidx=0;

    for (int b=0; b < nbad; b++)
    {
        // for each bad tile index in badtiles
        bidx = badtiles[b];
        for (int i=(bidx-counter); i < nelements-1; i++)
        {
            // shift each element in tile positions to the left by one
            // excluding the last element
            n_tile[i] = n_tile[i+1];
            e_tile[i] = e_tile[i+1];
            h_tile[i] = h_tile[i+1];
        }
        // array shifted left one, but the bad indexes refer to original tile positions
        // so we need to move the bad index to the left by one, too
        counter++;
    }
}

void getDeviceDimensions(int *nDevices)
{
    /* We need to know how many devices are available and its functionality. */

    printf("Querying system for device information --\n");
    hipGetDeviceCount(nDevices); // get CUDA to count GPUs

    for (int i = 0; i < *nDevices; i++)
    {
        struct hipDeviceProp_t prop; // create struct to store device info
        hipGetDeviceProperties(&prop, i); // populate prop for this device
        printf("    Device number:                       %d\n", *nDevices-1);
        printf("    Device name:                         %s\n", prop.name);
        printf("    Total global memory available (MB):  %f\n", prop.totalGlobalMem/1e6);
        printf("    Max grid size (# blocks):           (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
        printf("    Max number of threads per block:    (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    }
}

void requiredMemory(int size, int ntiles, int *niter, int *blockSize)
{
    /* Estimate how much device/host memory will be required per iteration.

       The device kernal consists of:
        
       az array = size * sizeof(double) - array of azimuth coords
       za array = size * sizeof(double) - array of zenith coords
       xpos = ntiles * sizeof(float) - x position of tiles from array center
       ypos = ntiles * sizeof(float) - y position of tiles from array center
       zpos = ntiles * sizeof(float) - z position of tiles from array center
       af_array = size * sizeof(hipDoubleComplex) - array factor

       plus we also pass:
       2 integers
       1 double

       and compute in the kernal:
       2 integers
       6 doubles
       1 hipDoubleComplex */

    int iters=1;
    int nDevices=0;
    long double azzaMem=0, tileposMem=0, afMem=0, otherMem=0, reqMem=0, devMem=0;
    double tfrac = 0.9;
    size_t freeMem, totMem;
    hipError_t res;
    struct hipDeviceProp_t prop; // create struct to store device info

    // get info about avilable devices
    getDeviceDimensions(&nDevices);
    printf("Number of devices on system: %d\n", nDevices);
    printf("Using device: %d\n", nDevices-1);
    hipGetDeviceProperties(&prop, 0); 

    // check how much FREE memory is available
    res = hipMemGetInfo(&freeMem, &totMem);
    if (res == hipSuccess)
    {
        printf("Free device memory: %.2f MB\n", (double)freeMem/1.0e6);
    }
    else
    {
        printf("%s\n", hipGetErrorString(res));
    }
    
    // get device max. threads per block
    *blockSize = prop.maxThreadsDim[0];

    // define the array sizes that will go onto the device
    azzaMem = 2 * (size/1.0e6) * sizeof(double); // az and za arrays
    tileposMem = 3 * (ntiles/1.0e6) * sizeof(float); // x,y,z positions of all tiles
    afMem = (size/1.0e6) * sizeof(hipDoubleComplex); // "array factor" array
    // misc. memory requirments (likely inconsequential)
    otherMem = (7 * sizeof(double) + 4 * sizeof(int) + sizeof(hipDoubleComplex) + sizeof(wavenums))/1.0e6;
    
    reqMem = azzaMem + tileposMem + afMem + otherMem; // total required memory in MB
    devMem = (double)freeMem/1.0e6; // available memory in MB

    printf("Memory required for:\n");
    printf("    Az,ZA arrays: %Lf MB\n",azzaMem);
    printf("    tile positions: %Lf MB\n",tileposMem);
    printf("    array fator: %Lf MB\n",afMem);
    printf("    intermediate: %Lf MB\n",otherMem);
    printf("Total: %Lf MB\n",reqMem);

    if (reqMem < 0)
    {
        fprintf(stderr, "Negative required memory (%Lf)!! Aborting.\n", reqMem);
        exit(1);
    }
    else if ((tfrac*devMem) <= reqMem)
    {
        fprintf(stderr, "Arrays will not fit on device!\n");
        fprintf(stderr, "   total device memory required = %Lf MB\n", reqMem);
        fprintf(stderr, "   useable device memory        = %Lf MB\n", tfrac*devMem);
        fprintf(stderr, "       (useable fraction = %.2f of total)\n", tfrac);

        iters = (int)ceil(tfrac*reqMem / devMem)+1;
        printf("Will split task into: %d iterations (approx. %.2Lf MB per iteration)\n", iters, (reqMem/iters));
    }
    printf("\n");

    *niter = iters;
}

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    /* Wrapper function for GPU/CUDA error handling. Every CUDA call goes through 
       this function. It will return a message giving your the error string, 
       file name and line of the error. Aborts on error. */

    if (code != 0)
    {
        fprintf(stderr, "GPUAssert:: %s - %s (%d)\n", hipGetErrorString(code), file, line);
        if (abort)
        {
            exit(code);
        }
    }
}
// define a macro for accessing gpuAssert
#define gpuErrchk(ans) {gpuAssert((ans), __FILE__, __LINE__);}


int main(int argc, char *argv[])
{
    char *ra="00:00:00.00";
    char *dec="-26:00:00.00";
    char *time="2017-06-19T11:13:00";
    char *metafits=NULL;
    char *flagfile=NULL;
    int c=0;
    double freq=0.0, lambda=0.0;
    double az_step=1.0, za_step=1.0, eta=1.0;
    int blockSize=1024, numBlocks=1024;
    int use_tile_beam=0, gridpoint=0;

    /* Parse options */
    if (argc > 1)
    {
        while ((c = getopt(argc, argv, "f:e:r:d:t:m:b:x:y:g:")) != -1)
        {
            switch(c)
            {
                case 'f':
                    freq = atof(optarg);
                    lambda = SOL/freq;
                    break;
                case 'e': 
                    eta = atof(optarg);
                    break;
                case 'r':
                    ra = strdup(optarg);
                    break;
                case 'd': 
                    dec = strdup(optarg);
                    break;
                case 't':
                    time = strdup(optarg);
                    break;
                case 'm':
                    metafits = strdup(optarg);
                    break;
                case 'b':
                    flagfile = strdup(optarg);
                    break;
                case 'x':
                    az_step = atof(optarg);
                    if (az_step < 0.01)
                    {
                        printf("error (option -x): can't use smaller than 0.01 deg resolution (hasn't been tested for that case)\n");
                        usage();
                        exit(1);
                    }
                    break;
                case 'y':
                    za_step = atof(optarg);
                    if (az_step < 0.01)
                    {
                        printf("error (option -y): can't use smaller than 0.01 deg resolution (hasn't been tested for that case)\n");
                        usage();
                        exit(1);
                    }
                    break;
                case 'g':
                    use_tile_beam = 1;
                    gridpoint = atoi(optarg);
                    break;
                default:
                    usage();
                    exit(1);
            }
        }
    }

    if (argc == 1)
    {
        usage();
        exit(1);
    }
    
    // let user know that using the FEE2016 tile beam model will slow down the simulation
    if (use_tile_beam == 1)
    {
        printf("Using FEE2016 tile beam model - this will slow down the computation significantly, but you can get antenna temperatures...\n");
        printf("    grid point number provided: %d\n", gridpoint);
    }
    else
    {
        printf("Not using tile beam model - only computing array factor, but you cannot get antenna temperatures from this...\n");
    }

    // calculate target az,za and wavevector
    tazza target;
    wavenums target_wn;

    printf("Getting target (Az,ZA)\n");
    calcTargetAZZA(ra, dec, time, &target);
    printf("Computing wavenumbers towards target\n");
    calcWaveNumber(lambda, target.az, target.za, &target_wn);
    printf("    kx = %f    ky = %f    kz = %f\n", target_wn.kx, target_wn.ky, target_wn.kz); 

    // get the number of tiles in array
    int ntiles=0;

    printf("Determining number of tiles from metafits\n");
    ntiles = getNumTiles(metafits); // returns 2x the number of tiles, 1 per pol.
    ntiles = ntiles / 2;
    printf("    number of tiles: %d\n",ntiles);

    // allocate dynamic memory for intermediate tile position arrays
    // (probably don't need to check as this should be ~MB scale)
    float *N_pols = (float *)malloc(2 * ntiles * sizeof(float));
    float *E_pols = (float *)malloc(2 * ntiles * sizeof(float));
    float *H_pols = (float *)malloc(2 * ntiles * sizeof(float));
    // allocate dynamic memory for tile positions
    float *N_tile = (float *)malloc(ntiles * sizeof(float));
    float *E_tile = (float *)malloc(ntiles * sizeof(float));
    float *H_tile = (float *)malloc(ntiles * sizeof(float));
    printf("Getting tile positions\n");
    getTilePositions(metafits, 2*ntiles,\
            N_pols, E_pols, H_pols,\
            N_tile, E_tile, H_tile);
    free(N_pols);
    free(E_pols);
    free(H_pols);

    // have to remove tiles from the flagged tiles list.
    // each element in the list is the index of a tile that needs to be removed.
    printf("Getting flagged tiles\n");
    int *flagged_tiles = (int *)malloc(ntiles * sizeof(int));
    int ntoread;
    ntoread = getFlaggedTiles(flagfile, flagged_tiles);
    int flagged[ntoread];

    for (int i=0; i < ntoread; i++)
    {
        flagged[i] = flagged_tiles[i];
    }
    printf("Removing %d flagged tiles\n",ntoread);
    printf("Tiles remaining: %d\n",ntiles-ntoread);
    removeFlaggedTiles(N_tile, E_tile, H_tile, flagged, ntoread, ntiles);
    free(flagged_tiles);
    printf("\n");

    // but, the last ntoread elements are pointless 
    // so now we can allocate static memory for the final list of positions
    ntiles = ntiles - ntoread;
    float xpos[ntiles], ypos[ntiles], zpos[ntiles];

    for (int i=0; i<ntiles; i++)
    {
        // x = East, y = North, z = Height
        xpos[i] = E_tile[i];
        ypos[i] = N_tile[i];
        zpos[i] = H_tile[i];
    }
    free(N_tile);
    free(E_tile);
    free(H_tile);


    // determine number of az/za elements from specified pixel size
    int niter=1;
    int n_az, n_za;
    long int size;

    n_az = (int)(360.0/az_step);
    n_za = (int)(90.0/za_step)+1; // +1 because we want to evalute at 90deg too!
    size = n_az * n_za;
    printf("Number of az steps [0,360): %d\n", n_az); // step from [0,360) - 360 will double count the 0 values
    printf("Number of za steps [0,90] : %d\n", n_za); // step from [0,90]
    printf("Total number of elements to compute: %ld\n", size);
    niter = 1; // how many times do I need to split the problem up?
    printf("\n");
 
    // figure out how many iterations are required (being conservative)
    // and the device properties (as a consequence)
    requiredMemory(size, ntiles, &niter, &blockSize);


    /* We now have the relevant array configuration and target source information 
       needed to calculate the array factor. The best way is to split it up into 
       managable chunks (depending on the device capabilities). */

      // construct arrays for computation on host
    double *az_array, *za_array;
    
    // allocate memory on host and check
    // azimuth vector
    az_array = (double *)calloc(size, sizeof(double));
    if (!az_array)
    {
        fprintf(stderr,"Host memory allocation failed (allocate az_array)\n");
        return EXIT_FAILURE;
    }
    // zenith vector
    za_array = (double *)calloc(size, sizeof(double));
    if (!za_array)
    {
        fprintf(stderr,"Host memory allocation failed (allocate za_array)\n");
        return EXIT_FAILURE;
    }

    // populate the host vectors:
    // TODO: this is currently the most memory intensive part on host.
    //       maybe we want to move this initilisation part into the iteration loop
    //       which will then make the arrays smaller --
    //           need to figure out how to populate correctly then...
    printf("Initialising az, za and result matrix\n");
    // want arrays to be something like:
    // az = [0 0 0 0 ... 1 1 1 1 ...]
    // za = [0 1 2 3 ... 0 1 2 3 ...]
    int cc=0, i=0;
    do
    {
        for (int j=0; j<n_za; j++)
        {
            za_array[cc+j] = j * za_step * DEG2RAD;
            az_array[cc+j] = i * az_step * DEG2RAD;
        }
        cc += n_za;
        i++;
    } while(cc < size);
    printf("Done\n");


    // construct arrays for device computation
    double *d_az_array, *d_za_array;
    double *subAz, *subZA;
    hipDoubleComplex *af_array, *d_af_array;
    float *d_xpos, *d_ypos, *d_zpos;
    //wavenums *d_twn;
    int itersize, az_idx1, az_idx2, za_idx1, za_idx2; 
    int iter_n_az = (int)floor(size / niter);
    int iter_n_za = (int)floor(size / niter);
    double af_max = -1, omega_A = 0.0;

    // before we get to the real computation, better open a file ready for writing
    int obsid;
    char output[100];
    sscanf(metafits, "%d%*s", &obsid);
    printf("Will output beam pattern to:\n");
    printf("    %d_%.2fMHz_%s.dat\n", obsid, freq/1.0e6, time);
    sprintf(output, "%d_%.2fMHz_%s.dat", obsid, freq/1.0e6, time);
    
    FILE *fp;
    fp = fopen(output,"w");  // open the file to write
    //fprintf(fp, "Az\tZA\tP\n"); // and write the header info

    /* This is the primary loop which does the calculations */
    printf("%d az , %d za per iteration\n", iter_n_az, iter_n_za);
    for (int iter = 0; iter < niter; iter++)
    {  
        printf("==== Iteration %d ====\n", iter);
        //fprintf(fp, "Iteration %d\n", iter);
        // figure out this iteration size, then allocate memory
        if (iter != niter-1)
        {
            itersize = iter_n_az; // = iter_n_za

            az_idx1 = iter * iter_n_az;
            az_idx2 = (iter+1) * iter_n_az;
            
            za_idx1 = iter * iter_n_za;
            za_idx2 = (iter+1) * iter_n_za;
        }
        else
        {
            /* If this is the last iteration, construct 
               iter_n_az/za such that it includes what ever
               is left over to compute.
               
               Should be ok in terms of memory because we made
               the number of iterations was computed on a 
               conservative estimate of the device memory. */
            
            iter_n_za = size - (iter * iter_n_za);
            iter_n_az = size - (iter * iter_n_az);
            itersize =  iter_n_az; // = iter_n_za

            az_idx1 = iter * iter_n_az;
            az_idx2 = az_idx1 + itersize - 1;

            za_idx1 = iter * iter_n_za;
            za_idx2 = za_idx1 + itersize - 1;
        }

        printf("# az: %d  # za: %d\n", iter_n_az, iter_n_za); 
        
        subAz = (double *)malloc(iter_n_az * sizeof(double));
        if (!subAz)
        {
            fprintf(stderr,"Host memory allocation failed (allocate subAz)\n");
            return EXIT_FAILURE;
        }
        subZA = (double *)malloc(iter_n_za * sizeof(double));
        if (!subZA)
        {
            fprintf(stderr,"Host memory allocation failed (allocate subZA)\n");
            return EXIT_FAILURE;
        }
        af_array = (hipDoubleComplex *)malloc(iter_n_az * sizeof(hipDoubleComplex));
        if (!af_array)
        {
            fprintf(stderr,"Host memory allocation failed (allocate af_array)\n");
            return EXIT_FAILURE;
        }

        // number of blocks required 
        numBlocks = (itersize + blockSize - 1) / blockSize; 


        printf("azimuth idx: %d - %d\n", az_idx1, az_idx2);
        printf("zenith  idx: %d - %d\n", za_idx1, za_idx2);
        printf("Number of GPU blocks used: %d\n", numBlocks);
        
        // place subset of az/za array into subAz/subZA
        for (int i=0; i<itersize; i++)
        {
            subAz[i] = az_array[i+az_idx1];
            subZA[i] = za_array[i+za_idx1];
            af_array[i] = make_hipDoubleComplex(0,0);
        }

        // allocate memory on device
        gpuErrchk( hipMalloc((void **)&d_az_array, itersize * sizeof(*az_array)));
        gpuErrchk( hipMalloc((void **)&d_za_array, itersize * sizeof(*za_array)));
        //gpuErrchk( hipMalloc((void **)&d_twn, sizeof(wavenums)));
        gpuErrchk( hipMalloc((void **)&d_xpos, ntiles * sizeof(*xpos)));
        gpuErrchk( hipMalloc((void **)&d_ypos, ntiles * sizeof(*ypos)));
        gpuErrchk( hipMalloc((void **)&d_zpos, ntiles * sizeof(*zpos)));
        gpuErrchk( hipMalloc((void **)&d_af_array, itersize * sizeof(*af_array)));


        // copy arrays onto device
        gpuErrchk( hipMemcpy(d_az_array, subAz, itersize * sizeof(*subAz), hipMemcpyHostToDevice));
        gpuErrchk( hipMemcpy(d_za_array, subZA, itersize * sizeof(*subZA), hipMemcpyHostToDevice));
        //gpuErrchk( hipMemcpy(d_twn, &target_wn, sizeof(wavenums), hipMemcpyHostToDevice));
        

        // copy the array factor vector to device
        gpuErrchk( hipMemcpy(d_af_array, af_array, itersize * sizeof(*af_array), hipMemcpyHostToDevice));
        
        // copy over tile position arrays to device
        gpuErrchk( hipMemcpy(d_xpos, xpos, ntiles * sizeof(*xpos), hipMemcpyHostToDevice));
        gpuErrchk( hipMemcpy(d_ypos, ypos, ntiles * sizeof(*ypos), hipMemcpyHostToDevice));
        gpuErrchk( hipMemcpy(d_zpos, zpos, ntiles * sizeof(*zpos), hipMemcpyHostToDevice));

        printf("Launching kernal to compute array factor\n");
        calcArrayFactor<<<numBlocks, blockSize>>>(itersize, ntiles, 2*PI/lambda, d_za_array, d_az_array, d_xpos, d_ypos, d_zpos, target_wn.kx, target_wn.ky, target_wn.kz, d_af_array);
        hipDeviceSynchronize();

        // copy relevant memory back to host
        gpuErrchk( hipMemcpy(af_array, d_af_array, itersize * sizeof(*af_array), hipMemcpyDeviceToHost));
        printf("==== Done ====\n");


        // test the CPU equivalent to make sure we get the same numbers
        //printf("    comparing CPU to GPU:\n");
        //printf("    real: %f imag: %f abs: %f\n", af_array[12].x, af_array[12].y, hipCabs(af_array[12]));
        //printf("initialise cpu af array\n");
        //double complex *aftmp;
        //aftmp = (double complex *)malloc(itersize * sizeof(*aftmp));
        //printf("inttialise elements\n");
        //for (int i=0; i<itersize; i++)
        //{
        //    aftmp[i] = 0+I*0;
        //}
        //printf("call function\n");
        //calcArrayFactorCPU(itersize, ntiles, 2*PI/lambda, subZA, subAz, xpos, ypos, zpos, &target_wn, aftmp);
        //printf("    real: %f imag: %f abs: %f\n", creal(aftmp[12]), cimag(aftmp[12]), cabs(aftmp[12]));

        // cool, we're done with the GPU computation
        printf("Freeing device memory\n");
        //gpuErrchk( hipFree(d_twn));
        gpuErrchk( hipFree(d_xpos));
        gpuErrchk( hipFree(d_ypos));
        gpuErrchk( hipFree(d_zpos));
        gpuErrchk( hipFree(d_af_array));
        gpuErrchk( hipFree(d_az_array));
        gpuErrchk( hipFree(d_za_array));

        // test the CPU equivalent to make sure we get the same numbers
        //printf("Calculation array factor power (|af|^2)\n");
        //printf("    comparing CPU to GPU:\n");
        //double tmp1 = pow(hipCabs(af_array[12]), 2);
        //double tmp2 = pow(cabs(aftmp[12]), 2);
        //printf("    gpu power: %f\n", tmp1);
        //printf("    cpu power: %f\n", tmp2);

        
        /* Write the output to a file */
        double af_power = 0.0;
        double tile_power = 1.0;
        printf("Writing to file...\n");
        for (int i=0; i<itersize; i++)
        {
            af_power = pow(hipCabs(af_array[i]), 2); // need to use hipCabs given af_array is of cuComplexDouble type
            //cpu_power = pow(cabs(aftmp[i]), 2);

            if (use_tile_beam == 1)
            {
                // calcaulte the tile beam power at (az,za) for a given frequency and sweet-spot
                tile_power = CalcMWABeam(subAz[i]-PI/2, subZA[i], freq, 'X', gridpoint, 1);
            }
 
            if (i % 10000 == 0) {printf("\rWriting element %d/%d", i, itersize); fflush(stdout);}

            fprintf(fp, "%f\t%f\t%f\n", subAz[i]*RAD2DEG, subZA[i]*RAD2DEG, af_power*tile_power);
            if (af_power > af_max) {af_max = af_power;}
            
            // integrate over sky
            omega_A = omega_A + sin(subZA[i]) * af_power * (za_step*DEG2RAD) * (az_step*DEG2RAD);
        }
        printf("\nDone -- freeing intermediate host memory\n");
        //free(aftmp);
        free(subAz);
        free(subZA);
        free(af_array);
    }
    printf("\n");
    printf("Closing file\n");
    fclose(fp); // close the file
    
    printf("Freeing host memory\n");
    free(az_array);
    free(za_array);

    // compute the gain and effective area from simulation
    double eff_area = 0.0, gain = 0.0;
    printf("Finished -- now computing relevant parameters:\n");
    eff_area = eta * pow(lambda, 2) * (4 * PI / omega_A);
    gain = (1.0e-26) * eff_area / (2 * KB);

    printf("    Array factor max:                 %f\n", af_max);
    printf("    Beam solid angle (sr):            %f\n", omega_A);
    printf("    Radiation efficiency:             %f\n", eta);
    printf("    Effective collecting area (m^2):  %.4f\n", eff_area);
    printf("    Effective array gain (K/Jy):      %.4f\n", gain);

    return 0;
}
