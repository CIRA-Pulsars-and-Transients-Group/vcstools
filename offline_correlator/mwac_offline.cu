#include "hip/hip_runtime.h"
/* --------------------------- header secton ----------------------------*/
#include <iostream>     /* yes we are moving to C++ */
#include <pthread.h>
#include <assert.h>
#include <cstring>
#include <cstdio>
#include <cstdint>
#include <sys/socket.h> /* for socket(), bind(), and connect() */
#include <sys/un.h>
#include <sys/shm.h>
#include <sys/time.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <sys/wait.h>
#include <arpa/inet.h>  /* for sockaddr_in and inet_ntoa() */
#include <errno.h>
#include "acquire_data.h"
#include "ringbuffer.h"
#include "buffer_sizes.h"
#include "packet.h"
#include <complex.h>
#include <unistd.h>
#include "fitsio.h"
#include "fourbit.h"
#include <fcntl.h>

/* -------------------------------- Correlator ------------------------------ */
// #include "xgpu.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
/* -------------------------------- Correlator Utils ------------------------------ */
#include "corr_utils.h"
/* -------------------------------- Beamformer ------------------------------ */
#include "run_beamer.h"
/* -------------------------------- End Beamformer ------------------------------ */

#define checkCudaError() do {                           \
hipError_t error = hipGetLastError();             \
if (error != hipSuccess) {                         \
fprintf(stderr, "(CUDA) %s", hipGetErrorString(error));  \
fprintf(stderr, " (" __FILE__ ":%d)\n", __LINE__);                \
return XGPU_CUDA_ERROR;                                           \
}                                                   \
} while (0)

/*

 IMPORTANT

 Data ordering for input vectors is (running from slowest to fastest)
 [time][channel][station][polarization][complexity]

 Output matrix has ordering
 [channel][station][station][polarization][polarization][complexity]

 We there is a wrinkle in that the station order out of the PFB and thus
 the correlator is *not* the same as the input order.

 Please see antenna_mapping.h


 */

 /*
    PJE: currently need to understand this code. It uses pthreads to run some tasks
    but would be critical to understand what the tasks are, what data is needed per thread,
    what mutex locks are required, etc.
 */

ringbuf_t ring;

/*--------------------------------------------------------------------------*/

void CheckForError(bool iflag, string errormsg, int errorflag=EXIT_FAILURE)
{
    if (!iflag) return;
    std::cerr << string("ERROR:\t")+errormsg << std::endl;
    exit(errorflag);
}

//get command options
struct Options{
    int nbit, coarse_chan, edge, nfrequency, dumps_to_aver, chan_to_aver, dumps_per_second;
    time_t starttime;
    char *in_file, *obsid;
    Options(){
        in_file = NULL;
        obsid = NULL;
        starttime = -1;
        dumps_per_second = 1; //correlator output dumps per second;
        chan_to_aver = 1; // number of channels to combine on output
        dumps_to_aver = 1; // number of correlator dumps to combine on output
        offline = 0;
        edge = 0;
        nbit = 4;
        coarse_chan = -1; // only set in the header if this is >= 0
        nfrequency = 128;
    };
}

void usage()
{
    std::cout << "offline_correlator: a light-weight correlator for the MWA. "
              << "Takes a NCHAN of data from stdin and correlates as per the "
              << "parameters of the linked xGPU library" << std::endl;
    std::cout << "offline_correlator from VCS Tools " << VERSION_BEAMFORMER
              << std::endl << std::endl;
    std::cout << "offline_correlator: -o <obsid> -s <time_t> -f nchan"
              << std::endl;
    std::cout << "Options:" << std::endl;
    std::cout << " -r <dump_rate> how many correlator dumps per second [1]"
              << std::endl;
    std::cout << " -n <number of channels to average> how many adjacent "
              << "channels to average " << std::endl;
    std::cout << " -i <number of correlator dumps to average> how many "
              << "correlator dumps to average " << std::endl;
    std::cout << "It will take data from stdin. In this case you need to "
              << "give it the start second of the dataset and the associated "
              << "obsid." << std::endl;
}

Options GetArgs(int argc, char *argv[]) {
    if (argc == 1)
    {
        usage();
        exit(EXIT_FAILURE);
    }
    Options opt;
    int arg = 0;
    while ((arg = getopt(argc, argv, "b:c:d:e:f:hi:n:o:r:s:V")) != -1) {

        switch (arg) {
            case 'b':
                opt.nbit = atoi(optarg);
                break;
            case 'c':
                opt.coarse_chan = atoi(optarg);
                break;
            case 'd':
                opt.in_file = strdup(optarg);
                break;
            case 'e':
                opt.edge = atoi(optarg);
                break;
            case 'f':
                // number of channels to correlate per coarse
                opt.nfrequency = atoi(optarg);
                break;
            case 'h':
                usage();
                exit(EXIT_SUCCESS);
            case 'i':
                // correlator dumps to sum
                opt.dumps_to_aver=atoi(optarg);
                break;
            case 'n':
                // number of channels to sum
                opt.chan_to_aver=atoi(optarg);
                break;
            case 'o':
                opt.offline = 1;
                opt.obsid = strdup(optarg);
                break;
            case 'r':
                // correlator dump rate
                opt.dumps_per_second = atoi(optarg);
                break;
            case 's':
                opt.starttime = (time_t) atol(optarg);
                break;
            case 'V':
                std::cout << "offline_correlator from VCS Tools "
                          << VERSION_BEAMFORMER << std::endl;
                exit(EXIT_SUCCESS);
                break;
        }
    }
    if (opt.starttime < 0)
    {
        usage();
        CheckForError(true, "offline mode selected but no starttime on command line");
    }

    if (opt.obsid < 0)
    {
        usage();
        CheckForError(true, "offline mode selected but no obsid on command line");
    }
    return opt;
}

// run the shutdown procedure
int Shutdown(int ring_nbufs,
    char **cuda_buffers,
    XGPUContext &context,
#ifdef RUN_BEAMER
    uint8_t *beam_d,
    int8_t *data_d,
#endif
    Complex *full_matrix_h,
    Complex *baseline_h,
    int xgpu_error
)
{
    /* ------------------------ Do the Book-keeping ---------------------- */
    //why do we sleep before shutting down?
    sleep(2);
    for (int i = 0; i < ring_nbufs; i++) {
        hipHostFree(cuda_buffers[i]);
        hipHostUnregister(cuda_buffers[i]); // device zero only
    }
    xgpuFree(&context);
    #ifdef RUN_BEAMER
    hipFree(beam_d); //
    hipFree(data_d); //
    #endif
    free(full_matrix_h);
    free(baseline_h);
    return xgpu_error;
}


//manages the ingest data.
//called upon thread creation
void *manager(void *context) {

    // this just manages the data ingest
    volatile manager_t *config = (manager_t *) context;
    char *raw_buffer = (char *) malloc(config->ring->bufsize);
    FILE *input = stdin;

    CheckForError((raw_buffer == NULL), "error: raw data buffer on start");

    std::cout << "Building lookup... ";
    build_eight_bit_lookup();
    std::cout << "Ready" << std::endl;;

    // each thread reads a file or stdin
    if (config->infile)
    {
        input= fdopen( config->infile, "r" );
    }

    while(true)
    {
        // we are getting data from stdin
        // lets just fill up a buffer
        int ninputs = config->nstation * config->npol;
        int nchan = config->nfrequency;
        int ntime = config->ntime;
        int ndim = config->ndim;
        int edge = config->edge;
        int nbit = config->nbit;
        int dumps_per_second = config->dumps_per_sec;

        char *buf = NULL;
        //gets the current buffer
        while (buf == NULL) {
            buf = get_buffer_to_fill_sync(config->ring);
        }
        get_buffer_status(config->ring);

        // now need to fill it with 1 seconds worth of input data
        // which is now a variable size as it depends on the number of edge channels that were removed.
        // becuase it is stored on disk as fourbit numbers
        size_t nread = 0;
        size_t to_read = (ntime * (nchan-2*edge) * ninputs * ndim * nbit)/8;
        std::cout << "Attempting to read in "<< to_read << " bytes" << std::endl;
        char *raw_buffer_ptr = raw_buffer;

        if (nbit == 4)
        {
            nread = fread(raw_buffer_ptr, 1, to_read, input);
            // check fred return status ... just in case
            if (nread != to_read)
            {
                std::cerr << "error: incomplete read on STDIN (" << nread << " of " << to_read
                          << "). Likely EOD" << std::endl;
                config->ring->EOD = 1;
                break;
            }
            /* four to eight bit expansion */
            size_t samps = 0;
            size_t timestep = 0;
            size_t chanstep = 0;

            int16_t *current_raw_ptr = (int16_t *) raw_buffer;
            int8_t *current_out_ptr = (int8_t *) buf;
            size_t samps_per_chan =ninputs*ndim; //NINPUTS*NDIM

            // NCHAN*NINPUTS*NDIM*NTIME
            while (timestep < ntime)
            {
                chanstep = 0;
                while (chanstep < nchan) {
                    samps = 0;
                    while (samps < samps_per_chan) {
                        if ((chanstep < edge ) || chanstep >= (nchan-edge)) {

                            current_out_ptr[0] = 0;
                            current_out_ptr[1] = 0;
                            current_out_ptr[2] = 0;
                            current_out_ptr[3] = 0;
                        }
                        else {
                            expand_4bit((uint16_t *) current_raw_ptr, (int8_t *) current_out_ptr);
                            current_raw_ptr++; // move 16 bits or 4 samples
                        }
                        current_out_ptr = current_out_ptr + 4; // mover 4 samples
                        samps=samps+4;
                    }
                    chanstep=chanstep+1;
                }
                timestep = timestep+1;
            }
            /* done 4 to 8bit expansion -- and droppped in the edges*/
        }
        else
        {
            //PJE: confused here, why does the code not try read from raw_buffer_ptr ?
            //seems to imply that nbit=4 always when reading from file
            //and the buff from stdnin
            nread = fread(buf, 1, to_read, stdin);
            // check fred return status ... just in case
            if (nread != to_read)
            {
                std::cerr << "error: incomplete read on STDIN (" << nread << " of " << to_read
                          << "). Likely EOD" << std::endl;
                //PJE: why doesn't this break the loop here? code was missing EOD=1 and break
                //I've added it here but commented
                // config->ring->EOD = 1;
                // break;
            }
        }
        mark_buffer_filled(config->ring); // this marks the buffer full
    }
    if (config->infile) fclose(input);
    free(raw_buffer);
    return NULL;
}

int main(int argc, char **argv) {

    XGPUInfo xgpu_info;
    int xgpu_error = 0;

    char *buf = 0x0;

    pthread_t buffer_handler;
    extern int buffer_handler_arg;

    // char *obsid=NULL;
    // char *in_file=NULL;

    /* picked up from the inbound header */

    struct tm start_utctime;
    struct tm current_utctime;
    /* picked up from the commandline */

    // time_t starttime = -1;

    /* constructed from the tm struct */

    time_t start_time_t = 0;
    time_t current_time_t = 0;
    time_t incremented_time_t = 0;

    timeval clock1,clock2,clock3;
    double elapsed = 0.0;


    char file_time[128];
    char dump_filename[128];

    unsigned int npol = 2, nstation = 128, ntime = 10000;
    // int dumps_per_second = 1; //correlator output dumps per second;
    // int chan_to_aver = 1; // number of channels to combine on output
    // int dumps_to_aver = 1; // number of correlator dumps to combine on output
    //
    // int offline = 0;
    //
    // int edge = 0;
    // int nbit = 4;
    // int coarse_chan = -1; // only set in the header if this is >= 0

    /*
    int arg = 0;

    while ((arg = getopt(argc, argv, "b:c:d:e:f:hi:n:o:r:s:V")) != -1) {

        switch (arg) {
            case 'b':
                nbit = atoi(optarg);
                break;
            case 'c':
                coarse_chan = atoi(optarg);
                break;
            case 'd':
                in_file = strdup(optarg);
                break;
            case 'e':
                edge = atoi(optarg);
                break;
            case 'f':
                // number of channels to correlate per coarse
                nfrequency = atoi(optarg);
                break;
            case 'h':
                usage();
                exit(EXIT_SUCCESS);
            case 'i':
                // correlator dumps to sum
                dumps_to_aver=atoi(optarg);
                break;
            case 'n':
                // number of channels to sum
                chan_to_aver=atoi(optarg);
                break;
            case 'o':
                offline = 1;
                obsid = strdup(optarg);
                break;
            case 'r':
                // correlator dump rate
                dumps_per_second = atoi(optarg);
                break;
            case 's':
                starttime = (time_t) atol(optarg);
                break;
            case 'V':
                std::cout << "offline_correlator from VCS Tools "
                          << VERSION_BEAMFORMER << std::endl;
                exit(EXIT_SUCCESS);
                break;
        }
    }

    if (argc == 1)
    {
        usage();
        exit(EXIT_FAILURE);
    }
    */
    // get command line options
    Options opt = GetArgs(argc, argv);

    manager_t the_manager; // dropped the volatile

    /// PJE: I'm quite confused as to why there is the nstation
    /// variable when manager_t instance does not use the variable
    /// and this variable is just set to 128. What's it's purpose?
    /// that also applies to npol, ndim,
    the_manager.shutdown=0;
    the_manager.offline = opt.offline;
    the_manager.integrate = opt.dumps_to_aver;
    the_manager.chan_to_aver = opt.chan_to_aver;
    the_manager.edge = opt.edge;
    the_manager.nbit = opt.nbit;
    the_manager.coarse_chan = opt.coarse_chan;
    the_manager.nstation = 128;
    the_manager.nfrequency = opt.nfrequency;
    the_manager.ndim = 2;
    the_manager.npol = 2;
    the_manager.dumps_per_sec = opt.dumps_per_second;
    the_manager.infile = 0;

    if (opt.in_file != NULL) {
        // we have an input file
        CheckForError(((the_manager.infile = open(opt.in_file,O_RDONLY)) == -1),
            string("input (") + to_string(opt.in_file) + ") file selected but cannot be opened");
    }

    /*
     First define the input ring buffers, as a thow back to earlier code this
     ringbuffer technology is the same technologu employed by some parts of the media conversion code
     it is not the same buffer technology employed for the output
     a buffer should be the size of an input buffer
     */

    /* lets allocate some buffers
     * need pinned memory
     */
    // Get sizing info from library

    xgpuInfo(&xgpu_info);
    CheckForError((npol != xgpu_info.npol),
        string("fatal missmatch between XGPU library and requested npol XGPU: ")+to_string(xgpu_info.npol)
        + string(", REQUESTED: ") + to_string(npol));
    CheckForError((nstation != xgpu_info.nstation),
        string("fatal missmatch between XGPU library and requested nstation XGPU: ")
        +to_string(xgpu_info.nstation)+string(", REQUESTED: ")+to_string(nstation));
    CheckForError((opt.nfrequency != xgpu_info.nfrequency),
        string("fatal missmatch between XGPU library and requested channels XGPU: ")
        +to_string(xgpu_info.nfrequency)+string(", REQUESTED: ") +to_string(opt.nfrequency));
    ntime = xgpu_info.ntime;

    the_manager.ntime = ntime;
    size_t full_matLength = opt.nfrequency * nstation * nstation * npol * npol;
    size_t full_size = opt.dumps_per_second * full_matLength * sizeof(Complex);
    size_t baseLength = opt.nfrequency;

    size_t ring_bufsz = xgpu_info.vecLength * sizeof(ComplexInput);

    char **cuda_buffers = (char **) calloc ((RING_NBUFS+1),sizeof(char*));

    size_t numbytes =(((ring_bufsz)+4095)/4096)*4096; // page size and page aligned

    // allocate cuda buffers on host and pin to device
    //PJE: why have i=0 and i = RING_NBUFS live just on host?
    for (int i = 0; i <= RING_NBUFS; i++)
    {
        std::cout << "allocating buffer " << i << " of " << numbytes << std::endl;
        cuda_buffers[i] = (char *) valloc(numbytes);
        CheckForError((cuda_buffers[i] == NULL),
            string("failed to allocate buffer ")+to_string(i));
        if ((i>0) && (i<RING_NBUFS)) {
            // xgpu_Init will register the first buffer but not the rest
            hipHostRegister(cuda_buffers[i],numbytes,0);
            checkCudaError();
        }
    }

    //init the ring buffer and assign buffers.
    std::cout << "assigning buffers" << std::endl;
    CheckForError((assign_ring_buffers(RING_NBUFS,ring_bufsz,cuda_buffers,&ring) < 0),
        string("Failed to ASSIGN ringbuffer"));

    Complex *full_matrix_h = NULL;
    Complex *baseline_h    = NULL;

    /*
     * the beamformer results. Format ... 8 bit int

     rember this is a total intensity sum.

     */
#ifdef RUN_BEAMER
    //PJE: would it not be easier to define a beamer class?
    uint8_t *beam_h = NULL;
    uint8_t *beam_d = NULL;
    int8_t *data_d = NULL;
    size_t timesteps_per_call = TIMESTEPS_PER_CALL;
    // how much data is processed per beamformer call
    size_t step_data_size = xgpu_info.nfrequency * xgpu_info.nstation * xgpu_info.npol * 2 * timesteps_per_call; // complex

    size_t step_results_size = xgpu_info.nfrequency*timesteps_per_call*sizeof(uint8_t);
    size_t beam_size = xgpu_info.nfrequency*xgpu_info.ntime*dumps_per_second*sizeof(uint8_t);
    /* FIXME: please check return codes here */

    /* ntime is the number of timesamples per GPU call for the correlator */
    /* dumps_per_second is the number of correlator dumps there are every second */

    /* full seconds worth of output beam */

    beam_h = (uint8_t *) malloc(beam_size);

    /* the input data for the beamformer will simple be offset into the buffer */
    /* But we will need to assign the device memory - this only need to be a number of timesteps equal to a pipelength
     */

    hipMalloc(&beam_d,step_results_size*sizeof(uint8_t)); //
    hipMalloc(&data_d,step_data_size*sizeof(int8_t)); //
#endif

    full_matrix_h = (Complex *) malloc(full_size);
    baseline_h = (Complex *) malloc(baseLength * sizeof(Complex));

    the_manager.ring = &ring;
    the_manager.nfrequency = opt.nfrequency;
    the_manager.nstation = nstation;
    the_manager.npol = npol;

    // create some threads calling the manager function,
    // passing manager_t instance (see corr_utils.h for struct)
    ///PJE: This pthread usage is odd since there is a pthread_create
    ///but the main program does not contain pthread_exit(NULL)
    ///so that it waits till all the threads it has created terminate
    ///There is also no pthread_join nor does the code
    ///try to explicitly set pthread attributes. why?
    CheckForError((pthread_create(&buffer_handler, NULL, manager,
        (void *) &the_manager)),
        string("could not launch manager thread"));

    std::cout << "Launched manager thread" << std::endl;

    uint64_t blockSize = 0;
    int hdu_num = 0;

    // The PRIMARY header + image + padding
    uint64_t n_visibilities = ((uint64_t) xgpu_info.nbaseline *4) ;

    while (hdu_num < dumps_per_second) {
        blockSize = blockSize + 2880; // header
        blockSize = blockSize +  (n_visibilities * (uint64_t) xgpu_info.nfrequency * sizeof(Complex));; // sizeof a data cube

        int remainder = (blockSize%2880); // pad out to the end of the HDU
        blockSize = blockSize + (2880 - remainder);
        hdu_num++; // hdu increment
    }

    assert(!(blockSize%2880));

    std::cout << "Correlating " << nstation << " stations, with " << npol
              << " signals, with " << opt.nfrequency << " channels" << std::endl;

    // allocate the GPU X-engine memory
    XGPUContext context;
    context.array_h = (ComplexInput *) cuda_buffers[0]; // already asssigned this above
    context.matrix_h = NULL; // we are letting the xgpu library configure tis memory - should make sure it is big enough
    context.array_len = xgpu_info.vecLength;
    context.matrix_len = xgpu_info.matLength;

    xgpu_error = xgpuInit(&context,0);

    if(xgpu_error)
    {
        std::cerr << "error: xgpuInit returned error code " << xgpu_error << std::endl;
        xgpuFree(&context);
        return xgpu_error;
    }

    Complex *cuda_matrix_h = context.matrix_h;
    int dumps_integrated = 0;

    tzset();
    extern long timezone;

    while (true) {
        strptime((const char *) the_manager.start_obs_UTC,"%Y-%m-%d-%H:%M:%S",&start_utctime);

        start_time_t = starttime;

        if (start_time_t != current_time_t)
        {
            std::cout << "start_time is " << the_manager.start_obs_UTC << ": "
                      << "decodes to: " << start_time_t << ", "
                      << "current is: " << current_time_t << std::endl;
            std::cout << "start_time is not current (" << start_time_t << ": "
                      << current_time_t << "): restart detected" << std::endl;
            std::cout << "Integrate " << the_manager.integrate << ": "
                      << "Chan to aver " << the_manager.chan_to_aver << std::endl;
            /* there has been a restart therefor the start time in the header is different to the expexted
             * start time*/
            current_time_t = start_time_t;
            incremented_time_t = current_time_t;
        }

        // how much of the second have we done
        int x_done = 0;
        // we only pass on a full second to the FITs builder. There are dumps_per_second correlator integrations
        // however there is nothing stopping the internal integration time of the correlator being much less than that
        // We are now capturing this with the integrate flag.

        // This integration now corresponds to howmany internal cuda invocations make up an integration, dumps per second is how many correlator integrations there are per second

        while (x_done < dumps_per_second) { // how many correlator dumps per second



            buf = NULL;
            static int count = 0; // just a check to see if the buffers are taking too long to drain,
            while (buf == NULL) {
                // the only way this returns is if there is a full buffer to read/or EOD/or overrun
                // it sleeps till those conditions are met
                buf = wait_for_buffer(&ring);
                //PJE: why don't we lock in checking ring.EOD? note that other checks
                //use locks (buffer_EOD applies lock)
                // if (ring.EOD) // this can be set and still there can be data in the ring
                // replace with mutex locking EOD check
                if (get_EOD(ring))
                {
                    std::cout << "NOTICE: EOD on input buffer" << std::endl;
                    if (buffer_EOD(&ring) == 0) {
                        std::cout << "NOTICE:: EOD on input buffer - but ring not yet empty : no reset yet" << std::endl;
                        count++;
                        //PJE: why are we sleeping at all?
                        sleep(1);
                        ///PJE: why is 5 seconds a speical case?
                        if (count > 5) {
                            std::cerr << "warning: waited > 5s for buffer to drain -- forcing reset" << std::endl;
                            reset_ring_buffers(&ring);
                            buf = NULL;
                            /// PJE: why on earth would you use a goto!?
                            // goto SHUTDOWN;
                            //replacing with function
                            return Shutdown(RING_NBUFS,
                                cuda_buffers,
                                context,
                                #ifdef RUN_BEAMER
                                beam_d, data_d,
                                #endif
                                full_matrix_h, baseline_h, xgpu_error
                            );
                        }
                    }
                    else if (buffer_EOD(&ring) == 1)
                    {
                        std::cout << "NOTICE:: EOD on input buffer drained - reset" << std::endl;
                        reset_ring_buffers(&ring);
                        buf = NULL;
                        // goto SHUTDOWN;
                        return Shutdown(RING_NBUFS,
                            cuda_buffers,
                            context,
                            #ifdef RUN_BEAMER
                            beam_d, data_d,
                            #endif
                            full_matrix_h, baseline_h, xgpu_error
                        );
                    }
                }
                else if (ring.overrun)
                {
                    std::cerr << "error: OVERRUN hard reset" << std::endl;
                    get_buffer_status(&ring);
                    reset_ring_buffers(&ring);
                    // goto SHUTDOWN;
                    return Shutdown(RING_NBUFS,
                        cuda_buffers,
                        context,
                        #ifdef RUN_BEAMER
                        beam_d, data_d,
                        #endif
                        full_matrix_h, baseline_h, xgpu_error
                    );
                }

                // get_buffer_status(&ring);
            } // get buffer

            /* --------------------------- Run The Correlator --------------------------- */

            /* We now should have PIPE_LENGTH chunks of NTIME_PIPE samples in the pinned
             memory - we should be free to run the correlator now
             */

             //PJE: Why isn't the following just a function call?
             /// would make it vastly easier to read.

            gettimeofday(&clock1,NULL);
            /*
             * this is the point where we increase our dump time. I am going to offset into the buffer the correct amount for the
             * sub second and process as normal - this should restrict the complication to here only
             *
             */


            xgpu_error = 0 ;

            if ((the_manager.integrate == 0) || dumps_integrated == 0) { // first internal integration (on the GPU)

               xgpu_error = xgpuClearDeviceIntegrationBuffer(&context);

                if(xgpu_error) {
                    std::cerr << "error: xgpuCudaXengine returned error code " << xgpu_error << std::endl;
                    xgpuFree(&context);
                    return xgpu_error;
                }
                dumps_integrated = 0;

            }


            context.array_h = (ComplexInput *) buf;
            // context.input_offset =  0; // there is no offset in this mode as the buffer only contains enough data for NTIME

            xgpu_error = xgpuCudaXengine(&context,SYNCOP_DUMP);

            checkCudaError();

            if(xgpu_error) {
                std::cerr << "error: xgpuCudaXengine returned error code " << xgpu_error << std::endl;
                xgpuFree(&context);
                return xgpu_error;
            }

            hipDeviceSynchronize();

            if (the_manager.integrate != 0) {
                dumps_integrated++; // internal cuda counter
            }
            else {
                dumps_integrated = 0;
            }

            std::cout << "GPU X-Engine done (" << dumps_integrated << ":"
                      << the_manager.integrate << ")" << std::endl;
            gettimeofday(&clock2,NULL);
            elapsed = (clock2.tv_sec - clock1.tv_sec) * 1000.0;      // sec to ms
            elapsed += (clock2.tv_usec - clock1.tv_usec) / 1000.0;   // us to ms

            std::cout << "Correlator/Beamformer took " << elapsed << " milliseconds" << std::endl;

            //

            // Lets copy the cube out into another area (it is only small)
            //
            //
            // Launch the beamformer

#ifdef RUN_BEAMER
            int time_step = 0;
            int steps = 10000;
            int8_t *data = NULL;
            uint8_t *results = NULL;
            data = (int8_t *) buf;
            results = beam_h;

            while(time_step < steps) {
                run_beamer(data_d,data,beam_d,results,step_data_size,step_results_size);
                data =  data + step_data_size;
                results =  results + step_results_size;
                time_step=time_step+TIMESTEPS_PER_CALL;
            }
#endif
            // PJE: what does this criterion ensure?
            if (dumps_integrated == the_manager.integrate) {



                Complex *ptr = full_matrix_h + (x_done * context.matrix_len);
                memcpy(ptr,cuda_matrix_h,(context.matrix_len * sizeof(Complex)));

                xgpu_error = xgpuClearDeviceIntegrationBuffer(&context);

                if(xgpu_error) {
                    std::cerr << "error: xgpuCudaXengine returned error code " << xgpu_error << std::endl;
                    xgpuFree(&context);
                    return xgpu_error;
                }
                dumps_integrated = 0;
                x_done++; // external sub-int count - we must copy out the accumulation and reset the integration buffer

            }
            mark_buffer_empty(&ring); // GPU input buffer mark clear

        } // done 1 seconds worth
        /* -------------------------- Write out the Product -------------------- */


        blockSize=0;
        hdu_num = 0;
        while (hdu_num < dumps_per_second) {

            blockSize += 2880; // header
            blockSize += n_visibilities * (uint64_t) xgpu_info.nfrequency *
                         sizeof(Complex) / the_manager.chan_to_aver; // sizeof a data cube

            int remainder = (blockSize%2880); // pad out to the end of the HDU
            blockSize += 2880 - remainder;
            hdu_num++; // hdu increment
        }

        assert(!(blockSize % 2880));

        if (the_manager.integrate != 0) {
            std::cout << "Integrated " << dumps_integrated << " invocations" << std::endl;
            std::cout << "blockSize " << (int)blockSize << std::endl;
        }

        char *outbuffer = (char *) malloc(blockSize);

        buildFITSBuffer( xgpu_info,
                         full_matrix_h,
                         blockSize,
                         (void *)outbuffer,
                         incremented_time_t,
                         dumps_per_second,
                         &the_manager );


        std::cout << "FITS file built" << std::endl;

        gmtime_r(&incremented_time_t,&current_utctime);

        std::cout << "Buffer time set " << asctime(&current_utctime) << std::endl;

        strftime(file_time,15,"%Y%m%d%H%M%S",&current_utctime);

        sprintf( dump_filename, "/%s_%s_gpubox%02d_00.fits",
                 opt.obsid, file_time, coarse_chan );

        FILE *outf = fopen( dump_filename, "w" );

        if (outf != NULL) {

            fwrite(outbuffer, blockSize, 1, outf);
            fclose(outf);
            std::cout << "Last cube dumped" << std::endl;

        }
        else {
            std::cerr << "error: failed to open dump file " << strerror(errno)
                      << std::endl;
        }
        free(outbuffer);



        incremented_time_t++; // increment time by a second;

        gettimeofday(&clock3,NULL);
        elapsed = (clock3.tv_sec - clock2.tv_sec) * 1000.0;      // sec to ms
        elapsed += (clock3.tv_usec - clock2.tv_usec) / 1000.0;   // us to ms
        std::cout << "Data output (FITS building etc took a further "
                  << elapsed << " milliseconds" << std::endl;

        elapsed = (clock3.tv_sec - clock1.tv_sec) * 1000.0;      // sec to ms
        elapsed += (clock3.tv_usec - clock1.tv_usec) / 1000.0;   // us to ms
        std::cout << "Total processing took " << elapsed
                  << " milliseconds" << std::endl;



    }
    // I cannot think of a reason to use a go to
    // replacing with a function call
    return Shutdown(RING_NBUFS,
        cuda_buffers,
        context,
        #ifdef RUN_BEAMER
        beam_d, data_d,
        #endif
        full_matrix_h, baseline_h, xgpu_error
    );
/*
    / * ------------------------ Do the Book-keeping ---------------------- * /
SHUTDOWN:
    sleep(2);
    for (int i = 0; i < RING_NBUFS; i++) {
        hipHostFree(cuda_buffers[i]);
        hipHostUnregister(cuda_buffers[i]); // device zero only
    }

    xgpuFree(&context);
#ifdef RUN_BEAMER
    hipFree(beam_d); //
    hipFree(data_d); //
#endif
    free(full_matrix_h);
    free(baseline_h);
    return xgpu_error;
*/

}
