#include "hip/hip_runtime.h"
/* --------------------------- header secton ----------------------------*/
#include <iostream>     /* yes we are moving to C++ */
#include <pthread.h>
#include <assert.h>
#include <string.h>
#include <stdio.h>
#include <stdint.h>
#include <sys/socket.h> /* for socket(), bind(), and connect() */
#include <sys/un.h>
#include <sys/shm.h>
#include <sys/time.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <sys/wait.h>
#include <arpa/inet.h>  /* for sockaddr_in and inet_ntoa() */
#include <errno.h>
#include "acquire_data.h"
#include "ringbuffer.h"
#include "buffer_sizes.h"
#include "packet.h"
#include <complex.h>
#include <unistd.h>
#include "fitsio.h"
#include "fourbit.h"
#include <fcntl.h>

/* -------------------------------- Correlator ------------------------------ */
#include "xgpu.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
/* -------------------------------- Correlator Utils ------------------------------ */
#include "corr_utils.h"
/* -------------------------------- Beamformer ------------------------------ */
#include "run_beamer.h"
/* -------------------------------- End Beamformer ------------------------------ */

#define checkCudaError() do {                           \
hipError_t error = hipGetLastError();             \
if (error != hipSuccess) {                         \
fprintf(stderr, "(CUDA) %s", hipGetErrorString(error));  \
fprintf(stderr, " (" __FILE__ ":%d)\n", __LINE__);                \
return XGPU_CUDA_ERROR;                                           \
}                                                   \
} while (0)

/*
 
 IMPORTANT
 
 Data ordering for input vectors is (running from slowest to fastest)
 [time][channel][station][polarization][complexity]
 
 Output matrix has ordering
 [channel][station][station][polarization][polarization][complexity]
 
 We there is a wrinkle in that the station order out of the PFB and thus
 the correlator is *not* the same as the input order.
 
 Please see antenna_mapping.h
 
 
 */

ringbuf_t ring;




/*--------------------------------------------------------------------------*/

void usage()
{
    std::cout << "offline_correlator: a light-weight correlator for the MWA. "
              << "Takes a NCHAN of data from stdin and correlates as per the "
              << "parameters of the linked xGPU library" << std::endl;
    std::cout << "offline_correlator from VCS Tools v" << VERSION_BEAMFORMER
              << std::endl << std::endl;
    std::cout << "offline_correlator: -o <obsid> -s <time_t> -f nchan"
              << std::endl;
    std::cout << "Options:" << std::endl;
    std::cout << " -r <dump_rate> how many correlator dumps per second [1]"
              << std::endl;
    std::cout << " -n <number of channels to average> how many adjacent "
              << "channels to average " << std::endl;
    std::cout << " -i <number of correlator dumps to average> how many "
              << "correlator dumps to average " << std::endl;
    std::cout << "It will take data from stdin. In this case you need to "
              << "give it the start second of the dataset and the associated "
              << "obsid." << std::endl;
}

void *manager(void *context) {
    
    // this just manages the data ingest
    
    
    volatile manager_t *config = (manager_t *) context;
    char *raw_buffer = (char *) malloc(config->ring->bufsize);
    FILE *input = stdin;

    if (raw_buffer == NULL) {
        std::cerr << "error: raw data buffer on start" << std::endl;
        exit(EXIT_FAILURE);
    }
    
    std::cout << "Building lookup... ";
    build_eight_bit_lookup();
    std::cout << "Ready" << std::endl;;
   
    if (config->infile)
    {
        input= fdopen( config->infile, "r" );
    }
 
    while(1)
    {
        // we are getting data from stdin
        // lets just fill up a buffer
        
        int ninputs = config->nstation * config->npol;
        int nchan = config->nfrequency;
        int ntime = config->ntime;
        int ndim = config->ndim;
        int edge = config->edge;
        int nbit = config->nbit;
        int dumps_per_second = config->dumps_per_sec;
        
        
        char *buf = NULL;
        
        while (buf == NULL) {
            buf = get_buffer_to_fill_sync(config->ring); //gets the current buffer
            
        }
        get_buffer_status(config->ring);
        // now need to fill it with 1 seconds worth of input data
        // which is now a variable size as it depends on the number of edge channels that were removed.
        // becuase it is stored on disk as fourbit numbers
        
        size_t nread = 0;
        size_t to_read = (ntime * (nchan-2*edge) * ninputs * ndim * nbit)/8;
        std::cout << "Attempting to read in "<< to_read << " bytes" << std::endl;
        char *raw_buffer_ptr = raw_buffer;
        
        if (nbit == 4) {
            nread = fread(raw_buffer_ptr,1,to_read,input);
            // check fred return status ... just in case
            if (nread != to_read) {
                std::cerr << "error: incomplete read on STDIN (" << nread << " of " << to_read
                          << "). Likely EOD" << std::endl;
                config->ring->EOD = 1;
                break;
            }
            else  {
                
                /* four to eight bit expansion */
                //
                size_t samps = 0;
                size_t timestep = 0;
                size_t chanstep = 0;
                
                
                int16_t *current_raw_ptr = (int16_t *) raw_buffer;
                int8_t *current_out_ptr = (int8_t *) buf;
                size_t samps_per_chan =ninputs*ndim; //NINPUTS*NDIM
                
                
                
                while (timestep < ntime) { // NCHAN*NINPUTS*NDIM*NTIME
                    
                    chanstep = 0;
                    
                    while (chanstep < nchan) {
                        
                        samps = 0;
                        while (samps < samps_per_chan) {
                            if ((chanstep < edge ) || chanstep >= (nchan-edge)) {
                                
                                current_out_ptr[0] = 0;
                                current_out_ptr[1] = 0;
                                current_out_ptr[2] = 0;
                                current_out_ptr[3] = 0;
                            }
                            else {
                                expand_4bit((uint16_t *) current_raw_ptr, (int8_t *) current_out_ptr);
                                current_raw_ptr++; // move 16 bits or 4 samples
                            }
                            current_out_ptr = current_out_ptr + 4; // mover 4 samples
                            samps=samps+4;
                        }
                        chanstep=chanstep+1;
                        
                    }
                    timestep = timestep+1;
                }
            }
            /* done 4 to 8bit expansion -- and droppped in the edges*/
            
        } else {
            nread = fread(buf,1,to_read,stdin);
            // check fred return status ... just in case
            if (nread != to_read)
            {
                std::cerr << "error: incomplete read on STDIN (" << nread << " of " << to_read
                          << "). Likely EOD" << std::endl;
            }
        }
        mark_buffer_filled(config->ring); // this marks the buffer full
        
    }
    if (config->infile) {
	fclose(input);
    }
    free(raw_buffer);

    return NULL;
}

int main(int argc, char **argv) {
    
    XGPUInfo xgpu_info;
    int xgpu_error = 0;
    
    char *buf = 0x0;

    pthread_t buffer_handler;
    extern int buffer_handler_arg;
    
    char *obsid=NULL;
    char *in_file=NULL;    
    
    /* picked up from the inbound header */
    
    struct tm start_utctime;
    struct tm current_utctime;
    /* picked up from the commandline */
    
    time_t starttime = -1;
    
    /* constructed from the tm struct */
    
    time_t start_time_t = 0;
    time_t current_time_t = 0;
    time_t incremented_time_t = 0;
    
    timeval clock1,clock2,clock3;
    double elapsed = 0.0;
    
    
    char file_time[128];
    char dump_filename[128];
    unsigned int npol = 2, nstation = 128, nfrequency = 128, ntime = 10000;
    
    int dumps_per_second = 1; //correlator output dumps per second;
    int chan_to_aver = 1; // number of channels to combine on output
    int dumps_to_aver = 1; // number of correlator dumps to combine on output
    
    int offline = 0;
    
    int edge = 0;
    int nbit = 4;
    int coarse_chan = -1; // only set in the header if this is >= 0
    
    int arg = 0;
    
    while ((arg = getopt(argc, argv, "b:c:d:e:f:hi:n:o:r:s:V")) != -1) {
        
        switch (arg) {
            case 'b':
                nbit = atoi(optarg);
                break;
            case 'c':
                coarse_chan = atoi(optarg);
                break;
            case 'd':
                in_file = strdup(optarg);
                break;
            case 'e':
                edge = atoi(optarg);
                break;
            case 'f':
                // number of channels to correlate per coarse
                nfrequency = atoi(optarg);
                break;
            case 'h':
                usage();
                exit(EXIT_SUCCESS);
            case 'i':
                // correlator dumps to sum
                dumps_to_aver=atoi(optarg);
                break;
            case 'n':
                // number of channels to sum
                chan_to_aver=atoi(optarg);
                break;
            case 'o':
                offline = 1;
                obsid = strdup(optarg);
                break;
            case 'r':
                // correlator dump rate
                dumps_per_second = atoi(optarg);
                break;
            case 's':
                starttime = (time_t) atol(optarg);
                break;
            case 'V':
                std::cout << "offline_correlator from VCS Tools v"
                          << VERSION_BEAMFORMER << std::endl;
                exit(EXIT_SUCCESS);
                break;
        }
    }
    
    if (argc == 1)
    {
        usage();
        exit(EXIT_FAILURE);
    }
    
    manager_t the_manager; // dropped the volatile
    
    the_manager.shutdown=0;
    the_manager.offline=offline;
    the_manager.integrate=dumps_to_aver;
    the_manager.chan_to_aver=chan_to_aver;
    the_manager.edge = edge;
    the_manager.nbit = nbit;
    the_manager.coarse_chan = coarse_chan;
    the_manager.nstation = 128;
    the_manager.nfrequency = nfrequency;
    the_manager.ndim = 2;
    the_manager.npol = 2;
    the_manager.dumps_per_sec = dumps_per_second;
    the_manager.infile = 0;

    if (in_file != NULL) {
        // we have an input file
        if ((the_manager.infile = open(in_file,O_RDONLY)) == -1)
        {
            std::cerr << "error: input (" << in_file << ") file selected but cannot be opened" << std::endl;
            exit(EXIT_FAILURE);
        }
    }
    
    if (starttime < 0)
    {
        usage();
        std::cerr << "error: offline mode selected but no starttime on command line" << std::endl;
        exit(EXIT_FAILURE);
    }
    
    if (obsid < 0)
    {
        usage();
        std::cerr << "error: offline mode selected but no obsid on command line" << std::endl;
        exit(EXIT_FAILURE);
    }
    
    
    
    /*
     First define the input ring buffers, as a thow back to earlier code this
     ringbuffer technology is the same technologu employed by some parts of the media conversion code
     it is not the same buffer technology employed for the output
     a buffer should be the size of an input buffer
     */
    
    /* lets allocate some buffers
     * need pinned memory
     */
    // Get sizing info from library
    
    xgpuInfo(&xgpu_info);
    if (npol != xgpu_info.npol)
    {
        std::cerr << "error: fatal missmatch between XGPU library and requested npol XGPU: "
                  << xgpu_info.npol << ", REQUESTED: " << npol << std::endl;
        exit(EXIT_FAILURE);
    }
    if (nstation != xgpu_info.nstation)
    {
        std::cerr << "error: fatal missmatch between XGPU library and requested nstation XGPU: "
                  << xgpu_info.nstation << ", REQUESTED: " << nstation << std::endl;
        exit(EXIT_FAILURE);
    }
    if (nfrequency != xgpu_info.nfrequency)
    {
        std::cerr << "error: fatal missmatch between XGPU library and requested channels XGPU: "
                  << xgpu_info.nfrequency << ", REQUESTED: " << nfrequency << std::endl;
        exit(EXIT_FAILURE);
    }
    ntime = xgpu_info.ntime;
    
    the_manager.ntime = ntime;
    size_t full_matLength = nfrequency * nstation * nstation * npol * npol;
    size_t full_size = dumps_per_second * full_matLength * sizeof(Complex);
    size_t baseLength = nfrequency;
    
    size_t ring_bufsz = xgpu_info.vecLength * sizeof(ComplexInput);
    
    char **cuda_buffers = (char **) calloc ((RING_NBUFS+1),sizeof(char*));
    
    size_t numbytes =(((ring_bufsz)+4095)/4096)*4096; // page size and page aligned
    
    for (int i = 0; i <= RING_NBUFS; i++)
    {
        std::cout << "allocating buffer " << i << " of " << numbytes << std::endl;
        
        cuda_buffers[i] = (char *) valloc(numbytes);
        
        if (cuda_buffers[i] == NULL) {
            std::cerr << "error: failed to allocate buffer " << i << std::endl;
            exit(EXIT_FAILURE);
        }
        
        if ((i>0) && (i<RING_NBUFS)) { // xgpu_Init will register the first buffer but not the rest
            hipHostRegister(cuda_buffers[i],numbytes,0);
            checkCudaError();
        }
        
    }
    
    std::cout << "assigning buffers" << std::endl;
    if (assign_ring_buffers(RING_NBUFS,ring_bufsz,cuda_buffers,&ring) < 0)
    {
        std::cerr << "Failed to ASSIGN ringbuffer" << std::endl;
        exit(EXIT_FAILURE);
    }
    
    Complex *full_matrix_h = NULL;
    Complex *baseline_h    = NULL;
    
    /*
     * the beamformer results. Format ... 8 bit int
     
     rember this is a total intensity sum.
     
     */
#ifdef RUN_BEAMER
    uint8_t *beam_h = NULL;
    uint8_t *beam_d = NULL;
    int8_t *data_d = NULL;
    size_t timesteps_per_call = TIMESTEPS_PER_CALL;
    // how much data is processed per beamformer call
    size_t step_data_size = xgpu_info.nfrequency * xgpu_info.nstation * xgpu_info.npol * 2 * timesteps_per_call; // complex
    
    size_t step_results_size = xgpu_info.nfrequency*timesteps_per_call*sizeof(uint8_t);
    size_t beam_size = xgpu_info.nfrequency*xgpu_info.ntime*dumps_per_second*sizeof(uint8_t);
    /* FIXME: please check return codes here */
    
    /* ntime is the number of timesamples per GPU call for the correlator */
    /* dumps_per_second is the number of correlator dumps there are every second */
    
    /* full seconds worth of output beam */
    
    beam_h = (uint8_t *) malloc(beam_size);
    
    /* the input data for the beamformer will simple be offset into the buffer */
    /* But we will need to assign the device memory - this only need to be a number of timesteps equal to a pipelength
     */
    
    hipMalloc(&beam_d,step_results_size*sizeof(uint8_t)); //
    hipMalloc(&data_d,step_data_size*sizeof(int8_t)); //
#endif
    
    full_matrix_h = (Complex *) malloc(full_size);
    
    baseline_h = (Complex *) malloc(baseLength * sizeof(Complex));
    
    the_manager.ring = &ring;
    
    the_manager.nfrequency = nfrequency;
    the_manager.nstation = nstation;
    the_manager.npol = npol;
    
    if (pthread_create(&buffer_handler, NULL, manager,
                       (void *) &the_manager)) {
        std::cerr << "error: could not launch manager thread" << std::endl;
        exit(EXIT_FAILURE);
    }
    
    std::cout << "Launched manager thread" << std::endl;
    
    uint64_t blockSize = 0;
    int hdu_num = 0;
    
    // The PRIMARY header + image + padding
    uint64_t n_visibilities = ((uint64_t) xgpu_info.nbaseline *4) ;
    
    while (hdu_num < dumps_per_second) {
        blockSize = blockSize + 2880; // header
        blockSize = blockSize +  (n_visibilities * (uint64_t) xgpu_info.nfrequency * sizeof(Complex));; // sizeof a data cube
        
        int remainder = (blockSize%2880); // pad out to the end of the HDU
        blockSize = blockSize + (2880 - remainder);
        hdu_num++; // hdu increment
    }
    
    assert(!(blockSize%2880));
    
    std::cout << "Correlating " << nstation << " stations, with " << npol
              << " signals, with " << nfrequency << " channels" << std::endl;
    
    // allocate the GPU X-engine memory
    XGPUContext context;
    context.array_h = (ComplexInput *) cuda_buffers[0]; // already asssigned this above
    context.matrix_h = NULL; // we are letting the xgpu library configure tis memory - should make sure it is big enough
    context.array_len = xgpu_info.vecLength;
    context.matrix_len = xgpu_info.matLength;
    
    xgpu_error = xgpuInit(&context,0);
    
    if(xgpu_error)
    {
        std::cerr << "error: xgpuInit returned error code " << xgpu_error << std::endl;
        xgpuFree(&context);
        return xgpu_error;
    }
    
    Complex *cuda_matrix_h = context.matrix_h;
    
    int dumps_integrated = 0;
    
    
    tzset();
    extern long timezone;
    
    while (1) {
        
        
        strptime((const char *) the_manager.start_obs_UTC,"%Y-%m-%d-%H:%M:%S",&start_utctime);
        
        start_time_t = starttime;
        
        if (start_time_t != current_time_t)
        {
            std::cout << "start_time is " << the_manager.start_obs_UTC << ": "
                      << "decodes to: " << start_time_t << ", "
                      << "current is: " << current_time_t << std::endl;
            std::cout << "start_time is not current (" << start_time_t << ": "
                      << current_time_t << "): restart detected" << std::endl;
            std::cout << "Integrate " << the_manager.integrate << ": "
                      << "Chan to aver " << the_manager.chan_to_aver << std::endl;
            /* there has been a restart therefor the start time in the header is different to the expexted
             * start time*/
            current_time_t = start_time_t;
            incremented_time_t = current_time_t;
        }
        
        
        int x_done = 0; // how much of the second have we done
        
        // we only pass on a full second to the FITs builder. There are dumps_per_second correlator integrations
        // however there is nothing stopping the internal integration time of the correlator being much less than that
        // We are now capturing this with the integrate flag.
        
        // This integration now corresponds to howmany internal cuda invocations make up an integration, dumps per second is how many correlator integrations there are per second
        
        while (x_done < dumps_per_second) { // how many correlator dumps per second
            
            
            
            buf = NULL;
            static int count = 0; // just a check to see if the buffers are taking too long to drain,
            while (buf == NULL) {
                
                buf = wait_for_buffer(&ring); // the only way this returns is if there is a full buffer to read/or EOD/or overrun
                
                if (ring.EOD) // this can be set and still there can be data in the ring
                {
                    std::cout << "NOTICE: EOD on input buffer" << std::endl;
                    if (buffer_EOD(&ring)== 0) {
                        std::cout << "NOTICE:: EOD on input buffer - but ring not yet empty : no reset yet" << std::endl;
                        count++;
                        sleep(1);
                        if (count > 5) {
                            std::cerr << "warning: waited > 5s for buffer to drain -- forcing reset" << std::endl;
                            reset_ring_buffers(&ring);
                            buf = NULL;
                            goto SHUTDOWN;
                        }
                    }
                    else if (buffer_EOD(&ring) == 1)
                    {
                        std::cout << "NOTICE:: EOD on input buffer drained - reset" << std::endl;
                        reset_ring_buffers(&ring);
                        buf = NULL;
                        goto SHUTDOWN;
                    }
                }
                else if (ring.overrun)
                {
                    std::cerr << "error: OVERRUN hard reset" << std::endl;
                    get_buffer_status(&ring);
                    reset_ring_buffers(&ring);
                    goto SHUTDOWN;
                }
                
                // get_buffer_status(&ring);
                
                
            } // get buffer
            
            /* --------------------------- Run The Correlator --------------------------- */
            
            /* We now should have PIPE_LENGTH chunks of NTIME_PIPE samples in the pinned
             memory - we should be free to run the correlator now
             */
            
            
            gettimeofday(&clock1,NULL);
            /*
             * this is the point where we increase our dump time. I am going to offset into the buffer the correct amount for the
             * sub second and process as normal - this should restrict the complication to here only
             *
             */
            
            
            xgpu_error = 0 ;
            
            if ((the_manager.integrate == 0) || dumps_integrated == 0) { // first internal integration (on the GPU)
                
               xgpu_error = xgpuClearDeviceIntegrationBuffer(&context);
                
                if(xgpu_error) {
                    std::cerr << "error: xgpuCudaXengine returned error code " << xgpu_error << std::endl;
                    xgpuFree(&context);
                    return xgpu_error;
                }
                dumps_integrated = 0;
                
            }
            
            
            context.array_h = (ComplexInput *) buf;
            // context.input_offset =  0; // there is no offset in this mode as the buffer only contains enough data for NTIME
            
            xgpu_error = xgpuCudaXengine(&context,SYNCOP_DUMP);
            
            checkCudaError();
            
            if(xgpu_error) {
                std::cerr << "error: xgpuCudaXengine returned error code " << xgpu_error << std::endl;
                xgpuFree(&context);
                return xgpu_error;
            }
            
            hipDeviceSynchronize();
            
            if (the_manager.integrate != 0) {
                dumps_integrated++; // internal cuda counter
            }
            else {
                dumps_integrated = 0;
            }
 
            std::cout << "GPU X-Engine done (" << dumps_integrated << ":"
                      << the_manager.integrate << ")" << std::endl;
            gettimeofday(&clock2,NULL);
            elapsed = (clock2.tv_sec - clock1.tv_sec) * 1000.0;      // sec to ms
            elapsed += (clock2.tv_usec - clock1.tv_usec) / 1000.0;   // us to ms
            
            std::cout << "Correlator/Beamformer took " << elapsed << " milliseconds" << std::endl;
            
            //
            
            // Lets copy the cube out into another area (it is only small)
            //
            //
            // Launch the beamformer
            
            
            
           
            
#ifdef RUN_BEAMER
            
            int time_step = 0;
            int steps = 10000;
            int8_t *data = NULL;
            uint8_t *results = NULL;
            data = (int8_t *) buf;
            results = beam_h;
            
            while(time_step < steps) {
                run_beamer(data_d,data,beam_d,results,step_data_size,step_results_size);
                data =  data + step_data_size;
                results =  results + step_results_size;
                time_step=time_step+TIMESTEPS_PER_CALL;
            }
            
#endif
            
            if (dumps_integrated == the_manager.integrate) {
                
              
                
                Complex *ptr = full_matrix_h + (x_done * context.matrix_len);
                memcpy(ptr,cuda_matrix_h,(context.matrix_len * sizeof(Complex)));
                
                xgpu_error = xgpuClearDeviceIntegrationBuffer(&context);
                
                if(xgpu_error) {
                    std::cerr << "error: xgpuCudaXengine returned error code " << xgpu_error << std::endl;
                    xgpuFree(&context);
                    return xgpu_error;
                }
                dumps_integrated = 0;
                x_done++; // external sub-int count - we must copy out the accumulation and reset the integration buffer
                
            }
            mark_buffer_empty(&ring); // GPU input buffer mark clear
            
        } // done 1 seconds worth
        /* -------------------------- Write out the Product -------------------- */
        
        
        blockSize=0;
        hdu_num = 0;
        while (hdu_num < dumps_per_second) {
            
        
        
            blockSize += 2880; // header
            blockSize += n_visibilities * (uint64_t) xgpu_info.nfrequency *
                         sizeof(Complex) / the_manager.chan_to_aver; // sizeof a data cube
            
            int remainder = (blockSize%2880); // pad out to the end of the HDU
            blockSize += 2880 - remainder;
            hdu_num++; // hdu increment
        }
        
        assert(!(blockSize % 2880));
        
        if (the_manager.integrate != 0) {
            std::cout << "Integrated " << dumps_integrated << " invocations" << std::endl;
            std::cout << "blockSize " << (int)blockSize << std::endl;
        }
        
        char *outbuffer = (char *) malloc(blockSize);
        
        buildFITSBuffer( xgpu_info,
                         full_matrix_h,
                         blockSize,
                         (void *)outbuffer,
                         incremented_time_t,
                         dumps_per_second,
                         &the_manager );
        
        
        std::cout << "FITS file built" << std::endl;
        
        gmtime_r(&incremented_time_t,&current_utctime);
        
        std::cout << "Buffer time set " << asctime(&current_utctime) << std::endl;
        
        strftime(file_time,15,"%Y%m%d%H%M%S",&current_utctime);
        
        sprintf( dump_filename, "/%s_%s_gpubox%02d_00.fits",
                 obsid, file_time, coarse_chan );
        
        FILE *outf = fopen( dump_filename, "w" );
        
        if (outf != NULL) {
            
            fwrite(outbuffer, blockSize, 1, outf);
            fclose(outf);
            std::cout << "Last cube dumped" << std::endl;
            
        }
        else {
            
            std::cerr << "error: failed to open dump file " << strerror(errno)
                      << std::endl;
            
        }
        
        free(outbuffer);
        
        
        
        incremented_time_t++; // increment time by a second;
        
        gettimeofday(&clock3,NULL);
        elapsed = (clock3.tv_sec - clock2.tv_sec) * 1000.0;      // sec to ms
        elapsed += (clock3.tv_usec - clock2.tv_usec) / 1000.0;   // us to ms
        std::cout << "Data output (FITS building etc took a further "
                  << elapsed << " milliseconds" << std::endl;
        
        elapsed = (clock3.tv_sec - clock1.tv_sec) * 1000.0;      // sec to ms
        elapsed += (clock3.tv_usec - clock1.tv_usec) / 1000.0;   // us to ms
        std::cout << "Total processing took " << elapsed
                  << " milliseconds" << std::endl;
        
        
        
    }
    /* ------------------------ Do the Book-keeping ---------------------- */
SHUTDOWN:
    sleep(2);
    for (int i = 0; i < RING_NBUFS; i++) {
        hipHostFree(cuda_buffers[i]);
        hipHostUnregister(cuda_buffers[i]); // device zero only
    }
    
    xgpuFree(&context);
#ifdef RUN_BEAMER
    hipFree(beam_d); //
    hipFree(data_d); //
#endif
    free(full_matrix_h);
    free(baseline_h);
    return xgpu_error;
    
}
