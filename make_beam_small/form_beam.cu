#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <hip/hip_runtime.h>

extern "C" {
#include "beam_common.h"
#include "form_beam.h"
#include "mycomplex.h"
}

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    /* Wrapper function for GPU/CUDA error handling. Every CUDA call goes through
       this function. It will return a message giving your the error string,
       file name and line of the error. Aborts on error. */

    if (code != 0)
    {
        fprintf(stderr, "GPUAssert:: %s - %s (%d)\n", hipGetErrorString(code), file, line);
        if (abort)
        {
            exit(code);
        }
    }
}

// define a macro for accessing gpuAssert
#define gpuErrchk(ans) {gpuAssert((ans), __FILE__, __LINE__);}


// define constants to be used in the kernel
#define NSTATION  128
#define NPOL      2
#define NSTOKES   4


__global__ void beamform_kernel( uint8_t *data,
                                 ComplexDouble *W,
                                 ComplexDouble *J,
                                 double invw,
                                 ComplexDouble *Bd,
                                 float *C,
                                 float *I )
/* Layout for input arrays:
 *   data [nsamples] [nchan] [NPFB] [NREC] [NINC] -- see docs
 *   W    [NSTATION] [nchan] [NPOL]               -- weights array
 *   J    [NSTATION] [nchan] [NPOL] [NPOL]        -- jones matrix
 * Layout for output arrays:
 *   Bd   [nsamples] [nchan]   [NPOL]             -- detected beam
 *   C    [nsamples] [NSTOKES] [nchan]            -- coherent full stokes
 *   I    [nsamples] [nchan]                      -- incoherent
 */
{
    // Translate GPU block/thread numbers into meaningful names
    int sample = blockIdx.x;
    int nchan  = blockDim.x;
    int ch     = threadIdx.x;

    // Calculate the indices for the input arrays
    int Di[NSTATION][NPOL];
    int Wi[NSTATION][NPOL];
    int Ji[NSTATION][NPOL][NPOL];

    int ant, pol, pol2, st;
    int pfb, rec, inc;
    for (ant = 0; ant < NSTATION; ant++)
    {
        pfb = ant / 32;
        inc = (ant / 8) % 4;
        for (pol = 0; pol < NPOL; pol++)
        {
            rec = (2*ant+pol) % 16;

            Di[ant][pol] = sample * (NINC*NREC*NPFB*nchan) +
                           ch     * (NINC*NREC*NPFB)       +
                           pfb    * (NINC*NREC)            +
                           rec    * (NINC)                 +
                           inc;

            Wi[ant][pol] = ant * (NPOL*nchan) +
                           ch  * (NPOL)       +
                           pol;

            for (pol2 = 0; pol2 < NPOL; pol2++)
            {
                Ji[ant][pol][pol2] = ant  * (NPOL*NPOL*nchan) +
                                     ch   * (NPOL*NPOL)       +
                                     pol  * (NPOL)            +
                                     pol2;
            }
        }
    }

    // Calculate the indices for the output arrays
    int Bdi[NPOL];
    int Ci[NSTOKES];
    int Ii;

    for (pol = 0; pol < NPOL; pol++)
        Bdi[pol] = sample * (NPOL*nchan) +
                   ch     * (NPOL)       +
                   pol;

    for (st = 0; st < NSTOKES; st++)
        Ci[st] = sample * (nchan*NSTOKES) +
                 st     * (nchan)         +
                 ch;

    Ii = sample*nchan + ch;

    // Calculate the beam and the noise floor
    ComplexDouble B[NPOL];
    ComplexDouble D[NPOL];
    ComplexDouble WD[NPOL];
    ComplexDouble N[NPOL][NPOL];

    for (pol = 0; pol < NPOL; pol++)
    {
        // Initialise beams and noise floor to zero
        Bd[Bdi[pol]] = CMaked( 0.0, 0.0 );
        I[Ii]        = 0.0;
        for (pol2 = 0; pol2 < NPOL; pol2++)
            N[pol][pol2] = CMaked( 0.0, 0.0 );

        for (ant = 0; ant < NSTATION; ant++)
        {
            // Calculate the coherent beam (B = J*W*D)
            B[pol]  = CMaked( 0.0, 0.0 );
            D[pol]  = UCMPLX4_TO_CMPLX_FLT(data[Di[ant][pol]]);
            WD[pol] = CMuld( W[Wi[ant][pol]], D[pol] );

            // (... and along the way, calculate the incoherent beam...)
            I[Ii] += DETECT(D[pol]);

            for (pol2 = 0; pol2 < NPOL; pol2++)
            {
                B[pol] = CAddd( B[pol], CMuld( J[Ji[ant][pol][pol2]],
                                               WD[pol2] ) );
            }

            // Detect the coherent beam
            Bd[Bdi[pol]] = CAddd( Bd[Bdi[pol]], B[pol] );

            // Calculate the noise floor (N = B*B')
            for (pol2 = 0; pol2 < NPOL; pol2++)
            {
                N[pol][pol2] = CAddd( N[pol][pol2],
                                      CMuld( B[pol], CConjd( B[pol2] ) ) );
            }
        }
    }

    // Form the stokes parameters for the coherent beam
    float bnXX = DETECT(Bd[Bdi[0]]) - CReald(N[0][0]);
    float bnYY = DETECT(Bd[Bdi[1]]) - CReald(N[1][1]);
    ComplexDouble bnXY = CSubd( CMuld( Bd[Bdi[0]], CConjd( Bd[Bdi[1]] ) ),
                                N[0][1] );

    // Stokes I, Q, U, V:
    C[Ci[0]] = invw*(bnXX + bnYY);
    C[Ci[1]] = invw*(bnXX - bnYY);
    C[Ci[2]] =  2.0*invw*CReald( bnXY );
    C[Ci[3]] = -2.0*invw*CImagd( bnXY );

    __syncthreads();
}

void cu_form_beam( uint8_t *data, struct make_beam_opts *opts,
                   ComplexDouble ***complex_weights_array,
                   ComplexDouble ****invJi, int file_no, int nstation, int nchan,
                   int npol, int outpol_coh, int outpol_incoh, double invw,
                   ComplexDouble ***detected_beam, float *coh, float *incoh )
/* The CPU version of the beamforming operations, using OpenMP for
 * parallelisation.
 *
 * Inputs:
 *   data    = array of 4bit+4bit complex numbers. For data order, refer to the
 *             documentation.
 *   opts    = passed option parameters, containing meta information about the
 *             obs and the data
 *   W       = complex weights array. [nstation][nchan][npol]
 *   J       = inverse Jones matrix. [nstation][nchan][npol][npol]
 *   file_no = number of file we are processing, starting at 0.
 *   nstation     = 128
 *   nchan        = 128
 *   npol         = 2 (X,Y)
 *   outpol_coh   = 4 (I,Q,U,V)
 *   outpol_incoh = 1 (I)
 *   invw         = the reciprocal of the sum of the antenna weights
 *
 * Outputs:
 *   detected_beam = result of beamforming operation, summed over antennas
 *                   [2*nsamples][nchan][npol]
 *   coh           = result in Stokes parameters (minus noise floor)
 *                   [nsamples][nstokes][nchan]
 *   incoh         = result (just Stokes I)
 *                   [nsamples][nchan]
 *
 * Assumes "coh" and "incoh" contain only zeros.
 */
{
    // Calculate array sizes for host and device
    size_t coh_size   = opts->sample_rate * outpol_coh   * nchan * sizeof(float);
    size_t incoh_size = opts->sample_rate * outpol_incoh * nchan * sizeof(float);
    size_t data_size  = opts->sample_rate * nchan * npol * sizeof(uint8_t);
    size_t Bd_size    = opts->sample_rate * nchan * npol * sizeof(ComplexDouble);
    size_t W_size     = nstation * nchan * npol          * sizeof(ComplexDouble);
    size_t J_size     = nstation * nchan * npol * npol   * sizeof(ComplexDouble);

    // Arrays to be passed to the GPU kernel
    // (We don't need to allocate host memory for data, coh, or incoh -- we
    // assume this is allocated before these pointers were passed into this
    // function)
    ComplexDouble *W, *d_W;
    ComplexDouble *J, *d_J;
    ComplexDouble *Bd, *d_Bd;
    uint8_t *d_data;
    float   *d_coh;
    float   *d_incoh;

    // Allocate host memory
    W  = (ComplexDouble *)malloc( W_size );
    J  = (ComplexDouble *)malloc( J_size );
    // Make Bd point to either the first or second half of detected beam,
    // according to whether this is an odd- or even-numbered file
    if (file_no % 2 == 0)
        Bd = &(detected_beam[0][0][0]);
    else
        Bd = &(detected_beam[opts->sample_rate][0][0]);


    // Allocate device memory
    gpuErrchk(hipMalloc( (void **)&d_W,     W_size ));
    gpuErrchk(hipMalloc( (void **)&d_J,     J_size ));
    gpuErrchk(hipMalloc( (void **)&d_Bd,    Bd_size ));
    gpuErrchk(hipMalloc( (void **)&d_data,  data_size ));
    gpuErrchk(hipMalloc( (void **)&d_coh,   coh_size ));
    gpuErrchk(hipMalloc( (void **)&d_incoh, incoh_size ));

    // Setup input values (= populate W and J)
    int ant, ch, pol, pol2;
    int Wi, Ji;
    for (ant = 0; ant < nstation; ant++)
    for (ch  = 0; ch  < nchan   ; ch++ )
    for (pol = 0; pol < npol    ; pol++)
    {
        Wi = ant * (npol*nchan) +
             ch  * (npol) +
             pol;
        W[Wi] = complex_weights_array[ant][ch][pol];

        for (pol2 = 0; pol2 < npol; pol2++)
        {
            Ji = Wi*npol + pol2;
            J[Ji] = invJi[ant][ch][pol][pol2];
        }
    }

    // Copy the data to the device
    gpuErrchk(hipMemcpy( d_data, data, data_size, hipMemcpyHostToDevice ));
    gpuErrchk(hipMemcpy( d_W,    W,    W_size,    hipMemcpyHostToDevice ));
    gpuErrchk(hipMemcpy( d_J,    J,    J_size,    hipMemcpyHostToDevice ));

    // Call the kernel
    beamform_kernel<<<opts->sample_rate, nchan>>>(
            d_data, d_W, d_J, invw, d_Bd, d_coh, d_incoh );
    hipDeviceSynchronize();

    // Copy the results back into host memory
fprintf(stderr, "*coh   = %p\n", coh );
fprintf(stderr, "*incoh = %p\n", incoh );
fprintf(stderr, "*Bd    = %p\n", Bd );
fprintf(stderr, "*d_coh   = %p\n", d_coh );
fprintf(stderr, "*d_incoh = %p\n", d_incoh );
fprintf(stderr, "*d_Bd    = %p\n", d_Bd );
fprintf(stderr, "coh_size   = %d\n", coh_size);
fprintf(stderr, "incoh_size = %d\n", incoh_size);
fprintf(stderr, "Bd_size    = %d\n", Bd_size);
    gpuErrchk(hipMemcpy( coh,   d_coh,   coh_size,   hipMemcpyDeviceToHost ));
    gpuErrchk(hipMemcpy( incoh, d_incoh, incoh_size, hipMemcpyDeviceToHost ));
    gpuErrchk(hipMemcpy( Bd,    d_Bd,    Bd_size,    hipMemcpyDeviceToHost ));

    // Free memory on host and device
    free( W );
    free( J );
    hipFree( d_W );
    hipFree( d_J );
    hipFree( d_Bd );
    hipFree( d_data );
    hipFree( d_coh );
    hipFree( d_incoh );
}

